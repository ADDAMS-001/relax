#include "hip/hip_runtime.h"
/*-----------------------------------------------------------------------
! Copyright 2013-2016 Sylvain Barbot
!
! This file is part of RELAX
!
! RELAX is free software: you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation, either version 3 of the License, or
! (at your option) any later version.
!
! RELAX is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License
! along with RELAX.  If not, see <http://www.gnu.org/licenses/>.
!
! \author Sagar Masuti 
!----------------------------------------------------------------------*/

/* Main functions called from host are : 
    1) cuinit
    2) custressupdatewrapper
    3) cuviscouseigen
    4) cufrictioneigenstress
    5) cubodyforceswrapper
    6) cudeinit
    7) cusource

  Other functions called from host but not so important/significant
    1) curesetvectors
    2) cutensormemset
    3) copytau
    4) cufieldrep 
    5) cutensorfieldadd
    6) cutensoramp
    7) cucopytraction
    8) cufieldadd
*/


/* This include contains the common macros and definitions */
#include "cuinclude.h"
#include "thrust/extrema.h"
#include <hipfft/hipfft.h>
#include <stdio.h>

/* Switch for enabling/disabling the usage of GPU */
#ifdef USING_CUDA

/* Thrust is used for finding min/max element */
#include <thrust/device_vector.h>

/* Number of points in the filter. Possible values are : 1, 7 and 14 */
#define FILTER_SIZE 1

/*      #define ENABLE_REG_BLOCKING         */ //Dont enable this. 
/*      #define PRINT_DEBUG_INFO        */  
/*      #define STRESS_SHARED_MEM       */ //Dont enable this.

/* Some macros */
#define PI  3.141592653589793115997963468544185161 

#define DEG2RAD 0.01745329251994329547437168059786927

#define MAX_NUM(a,b) (((a) > (b)) ? (a) : (b))

#define MAX3(a,b,c) (MAX_NUM(a, b) > c ? MAX_NUM(a, b) : c)

#define MIN(a, b)  (((a) < (b)) ? (a) : (b))

#define DSIGN(a,b) (((b) > 0) ? (a) : -(a)) 


/* -------------------------------------- global variables ------------------------------------- */ 

            /* Device Variables */
float       *gpV1 = NULL ;          /* Device Pointer. No dereferencing in host. */
float       *gpV2 = NULL ;          /* Device Pointer. No dereferencing in host. */     
float       *gpV3 = NULL ;          /* Device Pointer. No dereferencing in host. */
float       *gpU1 = NULL ;          /* Device Pointer. No dereferencing in host. */
float       *gpU2 = NULL ;          /* Device Pointer. No dereferencing in host. */
float       *gpU3 = NULL ;          /* Device Pointer. No dereferencing in host. */
float       *gpGammadot0 = NULL ;   /* Device Pointer. No dereferencing in host. */
float       *pfDevTract1 = NULL ;   /* Device Pointer. No dereferencing in host. */
float       *pfDevTract2 = NULL ;   /* Device Pointer. No dereferencing in host. */
float       *pfDevTract3 = NULL ;   /* Device Pointer. No dereferencing in host. */
ST_TENSOR   *pstSig = NULL ;        /* Device Pointer. No dereferencing in host. */
ST_LAYER    *pstStruct = NULL ;     /* Device Pointer. No dereferencing in host. */
ST_TENSOR   *pstMoment = NULL ;     /* Device Pointer. No dereferencing in host. */
ST_TENSOR   *pstTau = NULL ;        /* Device Pointer. No dereferencing in host. */
ST_TENSOR_LAYER   *pstPrestress = NULL ;  /* Device Pointer. No dereferencing in host. */
ST_TENSOR   *pstEpsilonik = NULL ;
ST_TENSOR   *pstEpsilonikdot = NULL ;
            /* Host Variables */
int         ihSx1 ;             /* Contains sx1 value*/
int         ihSx2 ;             /* Contains sx2 value*/
int         ihSx3 ;             /* Contains sx3 value*/
int         iLen1 ;             /* Contains the number points in the filter. */
int         iLen2 ;             /* Contains the number points in the filter. */
int         iLen3 ;             /* Contains the number points in the filter. */

            /* Device Constants */
__constant__ double constdKer1 [14] ;
__constant__ double constdKer2 [14] ;
__constant__ double constdKer3 [14] ;

/* --------------------------------------------------------------------------------------------- */



/* -------------------------------------- Funtion declaration ---------------------------------- */ 

int cuOptimalFilter (int *, int *, int *, int, int, int, double, double, double) ;

void cuFreeCudaMemory() ;

int checkMemRequirement(int, int, int) ;

void copygammadot0 (int, int, int, float *, int *) ;

bool cuispresent (void *) ;

__host__ __device__ void cutDot (ST_TENSOR *, double *, double *) ;

__host__ __device__ double cuSum (double *, double *) ;

__host__ __device__ void cuMulSub (double, double *, double *, double *) ;

__host__ __device__ double mycuSinh (double ) ;

__host__ __device__ double cuTensorTrace (ST_TENSOR *) ;

__host__ __device__ void cuIsotrpicStressStrain (ST_TENSOR *, double, double) ;

__host__ __device__ void cuTensorOperate (ST_TENSOR *, void *, char) ;

__host__ __device__ void cuTensorMemset (ST_TENSOR *) ;

__host__ __device__ double cuGauss (double, double) ;

__host__ __device__ double cuOmega (double, double) ;

__host__ __device__ double cuGaussp (double, double ) ;

__host__ __device__ double cuOmegap (double, double) ;

__host__ __device__ void cuTensorDyadProd (ST_TENSOR *, double *, double *) ;

__host__ __device__ double cuTensorNorm (ST_TENSOR *) ;

__host__ __device__ double cuDgGammaDotNot (ST_WEAK *, int, double, double, double, double) ;

__host__ __device__ void cuTensorDeviatoric(ST_TENSOR *, ST_TENSOR *) ;

__host__ __device__ void cuTensorDecompose (ST_TENSOR *, double *, ST_TENSOR *) ;

__host__ __device__ void cuShiftedCoordinates (int, int, int, int, int, int, double, double, 
                                               double, double *, double *, double *) ;

__device__ __host__ void print_tensor (ST_TENSOR *) ;

__device__ void culocalstrain_fir2 (ST_TENSOR *, int, int, int, int, int, int, float *, float *,
                                    float *, int, int, int) ;

__device__ void cuLocalStrain_ani (ST_TENSOR  *, int, int, double, int, int, int, int, int, 
                                   float *, float *, float *, int, int, int) ;

__device__ void cuLocalDivergence_ani (ST_TENSOR *, int, int, double, int, int, int, int, int, 
                                       double *, double *, double *, int, int, int) ;

__device__ void cuLocalDivergence_fir (ST_TENSOR *, int, int, int, int, int, int, double *, 
                                       double *, double *, int, int, int) ;

__global__ void cuTensorAmpKernel (ST_TENSOR *, double *, int, int, int) ;

__global__ void cuTensorFieldKernel (ST_TENSOR *, ST_TENSOR *, float, float, int, int, int) ;

__global__ void cuFieldAddKernel (float *, float *, float, float, int, int, int) ;

__global__ void cuEquivalentTraction (float *, float *, float *, ST_TENSOR *, int, int) ;

__global__ void cuLocalDivergenceKernel (int, int, int, int, int, double, int, int, int, float *, 
                                         float *, float *, ST_TENSOR *) ;

__global__ void cuStressUpdateKernel (int, int, int, double, double, int, int, int, float *, 
                                      float *, float *, ST_TENSOR *) ;

__global__ void cuEquiBodyKernel (ST_TENSOR *, int, int, int, int, int, int, float *, float *, 
                                  float *) ;

__global__ void cuBuildGammadotKernel (int, int, int, double, double, double, 
                                       double, ST_WEAK *, int, float *) ;

__global__ void cuViscousEigenKernel (ST_LAYER *, ST_TENSOR *, ST_TENSOR *, ST_TENSOR_LAYER *,
                                      float *, double, int, int, int, double, double, 
                                      double, float *, float *, bool, bool, bool) ;

__global__ void cuTransientEigenKernel (ST_LAYER *, ST_TENSOR *, ST_TENSOR *, float *,
                                        ST_TENSOR *, ST_TENSOR *, double, int,
                                        int, int, double, double, double, int, float *,
                                        bool) ;

__global__ void cuLocalStressStrainKernel (int, int, int, int, int, double, double, double, int, 
                                           int, int, float *, float *, float *, ST_TENSOR *) ;

__global__ void cuFrictionStress (double, double, double, double, double, double, double, double,
                                  double, double, double, double, double, int, int, int, double, 
                                  double, double, double, double, double, double, double, double, 
                                  double, int, double, float *, ST_TENSOR *, ST_TENSOR *, 
                                  ST_LAYER *) ;

__global__ void cuSourceTractionKernel (int, int, int, double, double, double, double, double, 
                                        double, double, double, double, double, double, double, 
                                        double, double, double, double, double, double, double, 
                                        double, double, double, double, float *, float *, float *) ;

__global__ void cuSourceForceKernel (int, int, int, double, double, double, double, double, double,
                                     double, double, double, double, double, double, double, double,
                                     double, double, double, double, double, double, double, double,
                                     double, double, double, float *, float *, float *) ;

/* --------------------------------------------------------------------------------------------- */


/* ----------------------------------------- Intermediate functions ---------------------------- */

/*
!-----------------------------------------------------------------
  !> StressUpdate
  !! computes the 3-d stress tensor sigma_ij' from the current
  !! deformation field. Strain is the second order tensor
  !!
  !!  \f[ \epsilon_{ij} = \frac{1}{2} ( u_{i,j} + u_{j,i} ) \f]
  !!
  !! The displacement derivatives are approximated numerically by the
  !! application of a differentiator space-domain finite impulse
  !! response filter. Coefficients of the filter can be obtained with
  !! the MATLAB command line
  !!
  !!  \f[ \sigma' = - C' : E \f]
  !!
  !! or in indicial notation
  !!
  !!
  !!  \f[ \sigma_{ij}' = -\lambda'*\delta_{ij}*\epsilon_{kk} - 2*\mu'*\epsilon_{ij}\f]
  !!
  !! where C' is the heterogeneous elastic moduli tensor and lambda'
  !! and mu' are the inhomogeneous lame parameters
  !!
  !!  \f[ C' = C(x) - C_0 \f]
  !!
  !! For isotropic materials
  !!
  !!  \f[ \mu'(x) = \mu(x) - \mu_0 \f]
  !!  \f[ \lambda'(x) = \lambda(x) - \lambda_0 \f]
  !!
  !! Optionally, the surface traction sigma_i3 can be sampled.
  !! 
  !-----------------------------------------------------------------
*/

/**
 *  
 *
 * @param   dLambda[in]     Lame's first parameter
 * @param   dMu[in]         shear modulus or Lame's second parameter. 
 * @param   dDx1[in]        Sampling size in x1(north) direction.
 * @param   dDx2[in]        Sampling size in x2(east) direction.
 * @param   dDx3[in]        Sampling size in x3(down) direction.
 * @param   iSx1[in]        The size of array in x1 direction.
 * @param   iSx2[in]        The size of array in x2 direction.
 * @param   iSx3[in]        The size of array in x3 direction.
 * @param   pstHostSig[in]  Host pointer to the sigma. ** legacy code. Not used. reserved for future **
 * @param   fData?[in]      Host pointer to the data(i.e., v? or u?) ** reserved for future ** 
 * @param   pfInput?[in]    Device pointer to the data( either v? or u?)  
*/

extern "C" void custressupdate_ (double     dLambda,
                                 double     dMu,
                                 double     dDx1,
                                 double     dDx2,
                                 double     dDx3,
                                 int        iSx1,
                                 int        iSx2,
                                 int        iSx3,
                                 ST_TENSOR  *pstHostSig,
                                 float      *fData1,
                                 float      *fData2,
                                 float      *fData3,
                                 float      *pfInput1,
                                 float      *pfInput2,
                                 float      *pfInput3)
{
    int          iInd3 = 0 ;
    int          iInd3p = 0 ;
    int          iInd3m = 0 ;

    double       dPx3 = 0.0;
    hipError_t  cuError = hipSuccess ;
    dim3         dimGrid (iSx2, 1, 1) ;
    dim3         dimBlock (iSx1, 1, 1) ;
    dim3         dimGrid1 (iSx3, iSx2, 1) ;
    dim3         dimBlock1 (iSx1, 1, 1) ;

#ifdef PAPI_PROF        
    char        cTimerName[17] = "stress          " ;
#endif

#ifdef PAPI_PROF        
    papistartprofiling_(cTimerName) ;
#endif

    for (iInd3 = 0 ; iInd3 < iSx3 ; iInd3++)
    {
        if ((iInd3 >= iLen3) && (iInd3 < (iSx3-iLen3)))
        {
            continue ;
        }

        if (iInd3 == 0)
        {
            dPx3 = dDx3 ;
            iInd3p = 1 ;
            iInd3m = 0 ;
        }
        else
        {
            if (iInd3 == iSx3-1)
            {
                dPx3 = dDx3 ;
                iInd3p = iSx3-1 ;
                iInd3m = iSx3-2 ;
            }
            else
            {
                dPx3 = dDx3*2.0 ;
                iInd3m = iInd3-1 ;
                iInd3p = iInd3+1 ;
            }
        }
        cuLocalStressStrainKernel<<<dimGrid, dimBlock>>> (iInd3m, iInd3p, iLen1, iLen2, iInd3, dPx3,
                                                          dLambda, dMu, iSx1, iSx2, iSx3, pfInput1, 
                                                          pfInput2, pfInput3, pstSig) ;

        cuError = hipGetLastError () ;
        CHECK_CUDA_ERROR ("stressupdate kernel launch failed 1\n", STRESS_UPDATE_EXIT_WITH_FREE)

    }
    if (hipDeviceSynchronize() != hipSuccess)
    {
        printf ("sync failed 1\n") ;
    }
    
    cuStressUpdateKernel <<<dimGrid1, dimBlock1>>>(iLen1, iLen2, iLen3, dLambda, dMu,
                                                   iSx1, iSx2, iSx3, pfInput1, pfInput2, pfInput3,
                                                   pstSig) ;
    cuError = hipGetLastError () ;
    CHECK_CUDA_ERROR ("stressupdate kernel launch failed\n", STRESS_UPDATE_EXIT_WITH_FREE)

    if (hipDeviceSynchronize() != hipSuccess)
    {
        printf ("sync failed 2\n") ;
    }

#ifdef PAPI_PROF        
    papiendprofiling_(cTimerName) ;
#endif

#ifdef PRINT_DEBUG_INFO
    printf ("custressupdate: exited with no errors\n") ;
#endif

    return ;


STRESS_UPDATE_EXIT_WITH_FREE:
    cuFreeCudaMemory () ;
    return ;
}

/**
  !-----------------------------------------------------------------
  !> subroutine EquivalentBodyForce
  !! computes and updates the equivalent body-force
  !!
  !!         f = - div.( C : E^i )
  !!
  !! and the equivalent surface traction
  !!
  !!         t = n . C : E^i
  !!
  !! with n = (0,0,-1). In indicial notations
  !!
  !!         f_i = - (C_ijkl E^i_kl),j
  !!
  !! and
  !!
  !!         t_1 = n_j C_ijkl E^i_kl
  !!
  !! where f is the equivalent body-force, t is the equivalent surface
  !! traction, C is the elastic moduli tensor and E^i is the moment
  !! density tensor tensor.
  !!
  !! Divergence is computed with a mixed numerical scheme including
  !! centered finite-difference (in the vertical direction) and
  !! finite impulse response differentiator filter for derivatives
  !! estimates. see function 'stress' for further explanations.
  !-----------------------------------------------------------------
 *
 * @param   pstSig      Device pointer to the sigma. 
 * @param       dDx1[in]        Sampling size in x1(north) direction.
 * @param       dDx2[in]        Sampling size in x2(east) direction.
 * @param       dDx3[in]        Sampling size in x3(down) direction.
 * @param       iSx1[in]        The size of array in x1 direction.
 * @param       iSx2[in]        The size of array in x2 direction.
 * @param       iSx3[in]        The size of array in x3 direction.
 * @param       fData?[in]      Host pointer to the data(i.e., v? or u?) ** reserved for future ** 
 * @param       pfT?[in]        Host pointer to the data(t?)  
*/

extern "C" void cuequivalentbodyforces_ (ST_TENSOR  *pstSig,
                                         double     dDx1,
                                         double     dDx2,
                                         double     dDx3,
                                         int        iSx1,
                                         int        iSx2,
                                         int        iSx3,
                                         float      *fData1,
                                         float      *fData2,
                                         float      *fData3,
                                         float      *pfT1,
                                         float      *pfT2,
                                         float      *pfT3)
{
    int          iInd3 = 0 ;
    int          iInd3p = 0 ;
    int          iInd3m = 0 ;

    double       dPx3 = 0.0;
    hipError_t  cuError = hipSuccess ;

    dim3         dimGrid (iSx2, 1, 1) ;
    dim3         dimBlock (iSx1, 1, 1) ;
    dim3         dimGrid1 (iSx3, iSx2, 1) ;
    dim3         dimBlock1 (iSx1, 1, 1) ;

    cuEquivalentTraction <<<dimGrid, dimBlock>>> (pfDevTract1, pfDevTract2, pfDevTract3,
                                                  pstSig, iSx1, iSx2 ) ;

    cuError = hipGetLastError () ;
    CHECK_CUDA_ERROR ("cuequivalentbodyforces_ : Failed in launch of cuEquivalentTraction\n", 
                      BODY_FORCES_EXIT_WITH_FREE) ;

    if (hipSuccess != hipDeviceSynchronize ())
    {
        printf ("cuequivalentbodyforces_ : Failed in sync 1\n") ;
        goto BODY_FORCES_EXIT_WITH_FREE ;
    }

    for (iInd3 = 0 ; iInd3 < iSx3 ; iInd3++)
    {
        if ((iInd3 >= iLen3) && (iInd3 < (iSx3-iLen3)))
        {
            continue ;
        }

        if (iInd3 == 0)
        {
            dPx3 = dDx3 ;
            iInd3p = 1 ;
            iInd3m = 0 ;
        }
        else
        {
            if (iInd3 == iSx3-1)
            {
                dPx3 = dDx3 ;
                iInd3p = iSx3-1 ;
                iInd3m = iSx3-2 ;
            }
            else
            {
                dPx3 = dDx3*2.0 ;
                iInd3m = iInd3-1 ;
                iInd3p = iInd3+1 ;
            }
        }
        cuLocalDivergenceKernel<<<dimGrid, dimBlock>>> (iInd3m, iInd3p, iLen1, iLen2, iInd3, dPx3,
                                                        iSx1, iSx2, iSx3, gpV1, gpV2, gpV3, pstSig);


        cuError = hipGetLastError () ;
        CHECK_CUDA_ERROR ("cuequivalentbodyforces_ : cuLocalDivergenceKernel kernel launch failed\n",
                          BODY_FORCES_EXIT_WITH_FREE)

    }

    dimGrid1.x = iSx3 - 2 * iLen3 ;

    cuEquiBodyKernel <<<dimGrid1, dimBlock1>>> (pstSig, iLen1, iLen2, iLen3, 
                                                iSx1, iSx2, iSx3, gpV1, gpV2, gpV3) ;
    cuError = hipGetLastError () ;
    CHECK_CUDA_ERROR ("cuequivalentbodyforces_ : cuEquiBodyKernel kernel launch failed\n", 
                      BODY_FORCES_EXIT_WITH_FREE)

    if (hipSuccess != hipDeviceSynchronize ())
    {
        printf ("cuequivalentbodyforces_ : Failed in sync 2\n") ;
        goto BODY_FORCES_EXIT_WITH_FREE ;
    }


#ifdef PRINT_DEBUG_INFO
    printf ("cubodyforces: exited with no errors\n") ;
#endif


    return ;


BODY_FORCES_EXIT_WITH_FREE :
    cuFreeCudaMemory () ;
}

/* -------------------------------------- Intermediate functions end ----------------------------------- */



/* -------------------------------- extern functions called from fortran ------------------------------- */
/**
 * This function allocates and initializes various memory required. 
 *
 * @param       iSx1[in]        The size of array in x1 direction.
 * @param       iSx2[in]        The size of array in x2 direction.
 * @param       iSx3[in]        The size of array in x3 direction.
 * @param       dDx1[in]        Sampling size in x1(north) direction.
 * @param       dDx2[in]        Sampling size in x2(east) direction.
 * @param       dDx3[in]        Sampling size in x3(down) direction.
 * @param   iRet[in,out]    Return code for any errors in allocation or initialization.
 **/

extern "C" void cuinit_ (int    iSx1,
                         int    iSx2,
                         int    iSx3,
                         double dDx1,
                         double dDx2,
                         double dDx3,
                         int    *iRet)
{
    hipError_t cuError = hipSuccess ;
    int         iSize = 0 ;
    int         iSize2 = 0 ;
#ifdef PRINT_DEBUG_INFO
    size_t iFreeMem ;
    size_t iTotalMem ;
#endif
    int         iDev ;

    hipDeviceProp_t deviceProp;
    cuError = hipGetDevice (&iDev) ;
    hipGetDeviceProperties(&deviceProp, iDev);
    //printf("Device %d: \"%s\"\n", iDev, deviceProp.name);
    *iRet = 1 ;

    ihSx1 = iSx1 ;
    ihSx2 = iSx2 ;
    ihSx3 = iSx3 ;
    
    if (-1 == checkMemRequirement(iSx1,iSx2,iSx3))
    {
        printf ("********************** ERROR ******************\n") ;
        printf ("Memory required to run on GPU is insufficient\n");
        printf ("Either try reducing the grid size or run on CPU only\n") ;
        printf ("********************** ERROR ******************\n\n") ;
        return ;
    }

    iSize = sizeof (float) * (iSx1 + 2) * iSx2 * iSx3 ;

    cuError = hipMalloc((void**)&gpV1, iSize) ;
    if (hipSuccess != cuError)
    {
        printf ("cuinit : Failed to allocate memory 1\n") ;
        goto CUINIT_FAILURE ;
    }

    cuError = hipMalloc((void**)&gpV2, iSize) ;
    if (hipSuccess != cuError)
    {
        printf ("cuinit : Failed to allocate memory 2\n") ;
        goto CUINIT_FAILURE ;
    }

    cuError = hipMalloc((void**)&gpV3, iSize) ;
    if (hipSuccess != cuError)
    {
        printf ("cuinit : Failed to allocate memory 3\n") ;
        goto CUINIT_FAILURE ;
    }

    cuError = hipMalloc((void**)&gpU1, iSize) ;
    if (hipSuccess != cuError)
    {
        printf ("cuinit : Failed to allocate memory 4\n") ;
        goto CUINIT_FAILURE ;
    }

    cuError = hipMalloc((void**)&gpU2, iSize) ;
    if (hipSuccess != cuError)
    {
        printf ("cuinit : Failed to allocate memory 5\n") ;
        goto CUINIT_FAILURE ;
    }

    cuError = hipMalloc((void**)&gpU3, iSize) ;
    if (hipSuccess != cuError)
    {
        printf ("cuinit : Failed to allocate memory 6\n") ;
        goto CUINIT_FAILURE ;
    }

    iSize2 = sizeof (ST_TENSOR) * iSx1 * iSx2 * (iSx3/2) ;
    cuError = hipMalloc((void**)&pstSig, iSize2) ;
    if (hipSuccess != cuError)
    {
        printf ("cuinit : Failed to allocate memory 7\n") ;
        goto CUINIT_FAILURE ;
    }

    cuError = hipMemset (pstSig, 0, iSize2) ;
    if (hipSuccess != cuError)
    {
        printf ("cuinit : Failed in memset\n") ;
        goto CUINIT_FAILURE ;
    }

    iSize = sizeof (float) * (iSx1+2) * iSx2 ;
    cuError = hipMalloc ((void **)&pfDevTract1, iSize) ;
    if (hipSuccess != cuError)
    {
        printf ("cuinit : Failed to allocate memory 8\n") ;
        goto CUINIT_FAILURE ;
    }

    cuError = hipMalloc ((void **)&pfDevTract2, iSize) ;
    if (hipSuccess != cuError)
    {
        printf ("cuinit : Failed to allocate memory 9\n") ;
        goto CUINIT_FAILURE ;
    }
    cuError = hipMalloc ((void **)&pfDevTract3, iSize) ;
    if (hipSuccess != cuError)
    {
        printf ("cuinit : Failed to allocate memory 10\n") ;
        goto CUINIT_FAILURE ;
    }

    iSize = sizeof (ST_LAYER) * iSx3/2 ;
    cuError = hipMalloc ((void **)&pstStruct, iSize) ;
    if (hipSuccess != cuError)
    {
        printf ("cuinit : Failed to allocate memory 11\n") ;
        goto CUINIT_FAILURE ;
    }
    cuError = hipMalloc ((void **)&pstMoment, iSize2) ;
    if (hipSuccess != cuError)
    {
        printf ("cuinit : Failed to allocate memory 12\n") ;
        goto CUINIT_FAILURE ;
    }
    cuError = hipMalloc((void**)&pstTau, iSize2) ;
    if (hipSuccess != cuError)
    {
        printf ("cuinit : Failed to allocate memory 13\n") ;
        goto CUINIT_FAILURE ;
    }

    iSize = sizeof (ST_TENSOR_LAYER) * iSx3/2 ;
    cuError = hipMalloc ((void **)&pstPrestress, iSize) ;
    if (hipSuccess != cuError)
    {
        printf ("cuinit : Failed to allocate memory 14\n") ;
        goto CUINIT_FAILURE ;
    }

//    if (pstFlags->istransient)
    {
        cuError = hipMalloc ((void **)&pstEpsilonik, iSize2) ;
        if (hipSuccess != cuError)
        {
            printf ("cuinit : Failed to allocate memory 15\n") ;
            goto CUINIT_FAILURE ;
        }
        cuError = hipMalloc ((void **)&pstEpsilonikdot, iSize2) ;
        if (hipSuccess != cuError)
        {
            printf ("cuinit : Failed to allocate memory 16\n") ;
            goto CUINIT_FAILURE ;
        }
    }

    //memset u1, u2, u3
    iSize = (sizeof (float) * (ihSx1+2) * ihSx2 * ihSx3) ;
    cuError = hipMemset (gpU1, 0, iSize) ;
    CHECK_CUDA_ERROR ("cuinit_ : Memset failed 1\n", CUINIT_FAILURE)
    cuError = hipMemset (gpU2, 0, iSize) ;
    CHECK_CUDA_ERROR ("cuinit_ : Memset failed 2\n", CUINIT_FAILURE)
    cuError = hipMemset (gpU3, 0, iSize) ;
    CHECK_CUDA_ERROR ("cuinit_ : Memset failed 3\n", CUINIT_FAILURE)

    *iRet = cuOptimalFilter (&iLen1, &iLen2, &iLen3,
                             iSx1, iSx2, iSx3, dDx1, dDx2, dDx3) ;
    if (0 != *iRet)
    {
        printf ("custressupdate_ : Something went wrong with optimal filter\n");
        goto CUINIT_FAILURE ;
    }

    createPlanForFFT (iSx1, iSx2, iSx3) ;
    *iRet = 0 ;

#ifdef PRINT_DEBUG_INFO
    hipMemGetInfo(&iFreeMem, &iTotalMem);
    printf ("cuinit: Memory available after allocation is : %lu MB\n", iFreeMem/(1024*1024));
    printf ("cuinit: Total memory available is : %lu MB\n",iTotalMem/(1024*1024));
    printf ("cuinit: exited with no errors\n") ;
#endif

    return ;

CUINIT_FAILURE:
    cuFreeCudaMemory () ;
}

extern "C" void cuinflags_ (int istransient, 
                            )
{
    
}
/**
 * This is called from the host code. For more information check custressupdate function.
 * @param   eType[in]       Variable to indicate data to be used(i.e., u? and v?) 
 * @param   dLambda[in]     Lame's first parameter
 * @param   dMu[in]         shear modulus or Lame's second parameter. 
 * @param   dDx1[in]        Sampling size in x1(north) direction.
 * @param   dDx2[in]        Sampling size in x2(east) direction.
 * @param   dDx3[in]        Sampling size in x3(down) direction.
 * @param   iSx1[in]        The size of array in x1 direction.
 * @param   iSx2[in]        The size of array in x2 direction.
 * @param   iSx3[in]        The size of array in x3 direction.
 * @param   fData?[in]      Host pointer to the data(i.e., v? or u?) ** reserved for future ** 
 * @param   pstHostSig[in]  Host pointer to the sigma. ** legacy code. Not used. reserved for future **
 *
 **/

extern "C" void custressupdatewrapper_ (E_TYPE     eType,
                                        double     dLambda,
                                        double     dMu,
                                        double     dDx1,
                                        double     dDx2,
                                        double     dDx3,
                                        int        iSx1,
                                        int        iSx2,
                                        int        iSx3,
                                        float      *fData1,
                                        float      *fData2,
                                        float      *fData3,
                                        ST_TENSOR  *pstHostSig)
{

    switch (eType)
    {
        case E_TYPE_U :
        {
            custressupdate_ (dLambda, dMu, dDx1, dDx2, dDx3, iSx1, iSx2, iSx3,
                             pstHostSig, fData1, fData2, fData3, gpU1, gpU2, gpU3) ;
        }
        break ;
        case E_TYPE_V:
        {
            custressupdate_ (dLambda, dMu, dDx1, dDx2, dDx3, iSx1, iSx2, iSx3,
                             pstHostSig, fData1, fData2, fData3, gpV1, gpV2, gpV3) ;
        }
        break ;
        case E_INVALID_TYPE:
        {
            printf ("custressupdatewrapper_: Invalid input\n") ;
        }
    }

}

/**
 * This is called from the host code. For more information check cuequivalentbodyforces_ function.
 * @param       eType[in]       Variable to indicate data to be used(i.e., pstSig/pstMoment) 
 * @param       dDx1[in]        Sampling size in x1(north) direction.
 * @param       dDx2[in]        Sampling size in x2(east) direction.
 * @param       dDx3[in]        Sampling size in x3(down) direction.
 * @param       iSx1[in]        The size of array in x1 direction.
 * @param       iSx2[in]        The size of array in x2 direction.
 * @param       iSx3[in]        The size of array in x3 direction.
 * @param       fData?[in]      Host pointer to the data(i.e., v? or u?) ** reserved for future ** 
 * @param       pfT?[in]        Host pointer to the data(t?)   
 *
 **/


extern "C" void cubodyforceswrapper_ (E_TENSOR_TYPE  eType,
                                      double         dDx1,
                                      double         dDx2,
                                      double         dDx3,
                                      int            iSx1,
                                      int            iSx2,
                                      int            iSx3,
                                      float          *fData1,
                                      float          *fData2,
                                      float          *fData3,
                                      float          *pfT1,
                                      float          *pfT2,
                                      float          *pfT3)
{
    switch (eType)
    {
        case E_TENSOR_SIG:
        {
            cuequivalentbodyforces_ (pstSig, dDx1, dDx2, dDx3, iSx1, iSx2, iSx3,
                                     fData1, fData2, fData3, pfT1, pfT2, pfT3) ;
        }
        break ;
        case E_TENSOR_MOMENT:
        {
            cuequivalentbodyforces_ (pstMoment, dDx1, dDx2, dDx3, iSx1, iSx2, iSx3,
                                     fData1, fData2, fData3, pfT1, pfT2, pfT3) ;
        }
        break ;
        default:
        {
            printf ("cubodyforceswrapper_ : Invalid input\n") ;
        }
    }
}


extern "C" void cusource_ (double  dMu,
                           double  dS,
                           double  dX,
                           double  dY,
                           double  dZ,
                           double  dL,
                           double  dW,
                           double  dStrike,
                           double  dDip,
                           double  dRake,
                           double  dBeta,
                           int     iSx1,
                           int     iSx2,
                           int     iSx3,
                           double  dDx1,
                           double  dDx2,
                           double  dDx3,
                           float   *pfData1,
                           float   *pfData2,
                           float   *pfData3,
                           float   *pfTract1,
                           float   *pfTract2,
                           float   *pfTract3)
{
    double  dcStrike ;
    double  dsStrike ;
    double  dcDip ;
    double  dsDip ;

    double  dCr ;
    double  dSr ;
    double  dX2r ;
    double  dScale ;

    double  dXr ;
    double  dYr ;
    double  dZr ;
    double  dWp ;
    double  dLp ;

    dim3    dimGrid (iSx2, 1, 1) ;
    dim3    dimBlock (iSx1, 1, 1) ;
    dim3    dimGrid1 (iSx3/2, iSx2, 1) ;
    dim3    dimBlock1 (iSx1, 1, 1) ;

    hipError_t  cuError = hipSuccess ;

#ifdef PAPI_PROF        
    char    cTimerName[17] = "source          " ;
#endif

#ifdef PAPI_PROF        
    papistartprofiling_(cTimerName) ;
#endif

    dcStrike = cos (dStrike) ;
    dsStrike = sin (dStrike) ;
    dcDip = cos (dDip) ;
    dsDip = sin (dDip) ;
    dCr = cos (dRake) ;
    dSr = sin (dRake) ;
    dScale = -1.0 * (dMu * dS) ;


    dWp = dW * (1.0 + 2.0 * dBeta) / 2.0 ;
    dLp = dL * (1.0 + 2.0 * dBeta) / 2.0 ;

    dX2r = (dcStrike * dX) - (dsStrike * dY) ;
    dXr = (dcDip * dX2r) - (dsDip * dZ) ;
    dYr = (dsStrike * dX) + (dcStrike * dY) ;
    dZr = (dsDip * dX2r) + (dcDip * dZ) ;


    cuSourceTractionKernel <<<dimGrid, dimBlock>>> (iSx1, iSx2, iSx3, dDx1, dDx2, dDx3, dcStrike, 
                                                    dsStrike,
                                                    dcDip, dsDip, dCr, dSr, dScale, dWp, dLp, dX2r,
                                                    dXr, dYr, dZr, dX, dY, dW, dL, dBeta, dMu, dS,
                                                    pfDevTract1, pfDevTract2, pfDevTract3) ;

    cuError = hipGetLastError () ;
    CHECK_CUDA_ERROR ("cusource_ : Failed in launch of cusourcetractionkernel\n", CUSOURCE_FREE_EXIT) ;

    if (hipSuccess != hipDeviceSynchronize())
    {
        printf ("cusource_: Synch failed\n") ;
        goto CUSOURCE_FREE_EXIT ;
    }
        
    cuSourceForceKernel <<<dimGrid1, dimBlock1>>> (iSx1, iSx2, iSx3, dDx1, dDx2, dDx3, dcStrike, 
                                                   dsStrike, dcDip, dsDip, dCr, dSr, dScale, dWp,
                                                   dLp, dX2r, dXr, dYr, dZr, dX, dY, dZ, dDip, dW,
                                                   dL, dBeta, dMu, dS, gpV1, gpV2, gpV3) ;
    cuError = hipGetLastError () ;
    CHECK_CUDA_ERROR ("cusource_ : Failed in launch of cuSourceForceKernel\n", CUSOURCE_FREE_EXIT) ;

    if (hipSuccess != hipDeviceSynchronize())
    {
        printf ("cusource_: Synch failed\n") ;
        goto CUSOURCE_FREE_EXIT ;
    }


#ifdef PAPI_PROF        
    papiendprofiling_(cTimerName) ;
#endif

#ifdef PRINT_DEBUG_INFO
    printf ("cusources : exited with no errors\n") ;
#endif

    return ;

CUSOURCE_FREE_EXIT:
    cuFreeCudaMemory () ;
}

extern "C" void cutransienteigen_ (ST_LAYER          *pStruct,
                                   ST_TENSOR         *pstEpsilonik,
                                   ST_TENSOR         *pstEpsilonikdot,
                                   double            dMu,
                                   int               iSx1,
                                   int               iSx2,
                                   int               iSx3,
                                   double            dDx1,
                                   double            dDx2,
                                   double            dDx3,
                                   int               bMaxwell,
                                   double            *dMaxwell,
                                   int               bdgammadot0,
                                   float             *dgammadot0)
{
    hipError_t  cuError = hipSuccess ;
    int          iSize = 0 ;
    float        *devMinArray = NULL ;
    dim3         dimGrid (iSx3, iSx2, 1) ;
    dim3         dimBlock (iSx1, 1, 1) ;
    int          isdgammadot0 = false ;
    int          iRet = 0;
    double       dTemp = 0.0 ;

#ifdef PAPI_PROF
    char cTimerName[17] = "Transienteigen  " ;
    papistartprofiling_ (cTimerName) ;
#endif

    iSize = sizeof (ST_LAYER) * iSx3 ;
    if (bMaxwell)
    {
        cuError = hipMalloc((void **) &devMinArray, sizeof (float) * iSx1 * iSx2 * iSx3) ;
        if (hipSuccess != cuError)
        {
            printf ("cutransienteigen_: Failed to allocate 0\n") ;
            cuFreeCudaMemory () ;
        }
    }

    cuError = hipMemcpy (pstStruct, pStruct, iSize, hipMemcpyHostToDevice) ;
    if (hipSuccess != cuError)
    {
        printf ("cutransienteigen_: memcpy failed 1\n") ;
        cuFreeCudaMemory () ;
    }

    /* if dgammadot0 is present then we need to add that to gammadot0 */
    if (bdgammadot0)
    {
        isdgammadot0 = true;
        copygammadot0 (iSx1, iSx2, iSx3, dgammadot0, &iRet) ;
    }

    cuTransientEigenKernel <<<dimGrid, dimBlock>>> (pstStruct, pstSig, pstMoment,
                                                    gpGammadot0, pstEpsilonik,
                                                    pstEpsilonikdot, dMu,
                                                    iSx1, iSx2, iSx3, dDx1, dDx2,
                                                    dDx3, bMaxwell, devMinArray, isdgammadot0) ;

    cuError = hipGetLastError () ;
    if ((hipSuccess != cuError) && (hipSuccess != hipDeviceSynchronize()))
    {
        printf ("cutransienteigen_: transient kernel failure \n") ;
        cuFreeCudaMemory () ;
    }

    if (bMaxwell)
    {
        thrust::device_ptr<float> dev(devMinArray);
        thrust::device_ptr<float> min = thrust::min_element(dev, dev+(iSx1 * iSx2 * iSx3)) ;

        cuError = hipGetLastError () ;
        if (hipSuccess != cuError)
        {
            printf ("cutransienteigen_: Thrust min element failure \n") ;
            cuFreeCudaMemory () ;
        }

        dTemp  =  *min ;
        *dMaxwell = MIN (*dMaxwell, dTemp) ;
        hipFree (devMinArray) ;
    }

#ifdef PAPI_PROF
    papiendprofiling_ (cTimerName) ;
#endif
    return ;

}

extern "C" void cutransienteigenwrapper_ (E_TENSOR_TYPE eType,
                                          ST_LAYER   *pStruct,
                                          double     dMu,
                                          int        iSx1,
                                          int        iSx2,
                                          int        iSx3,
                                          double     dDx1,
                                          double     dDx2,
                                          double     dDx3,
                                          int        bMaxwell,
                                          double     *dMaxwell,
                                          int        bgammadot,
                                          float      *dGammadot0 = NULL)
{

    switch (eType)
    {
        case E_TENSOR_IK :
        {
            cutransienteigen_ (pStruct, pstEpsilonik, pstEpsilonikdot, dMu,
                               iSx1, iSx2, iSx3, dDx1, dDx2, dDx3,
                               bMaxwell, dMaxwell, bgammadot, dGammadot0) ;
        }
        break ;
        case E_TENSOR_IKDOT:
        {
            cutransienteigen_ (pStruct, pstEpsilonikdot, pstEpsilonikdot, dMu,
                               iSx1, iSx2, iSx3, dDx1, dDx2, dDx3,
                               bMaxwell, dMaxwell, bgammadot, dGammadot0) ;
        }
        break ;
        case E_INVALID_TYPE:
        {
            printf ("custressupdatewrapper_: Invalid input\n") ;
        }
    }
}
extern "C" void cuviscouseigen_ (ST_LAYER          *pStruct,
                                 ST_TENSOR         *pSig,
                                 ST_TENSOR         *pMoment,
                                 ST_TENSOR_LAYER   *pPrestress, 
                                 double            dMu,
                                 int               iSx1,
                                 int               iSx2,
                                 int               iSx3,
                                 double            dDx1,
                                 double            dDx2,
                                 double            dDx3,
                                 float             *dMaxwell,
                                 float             *dgammadot0,
                                 float             *pGamma)
{
    hipError_t  cuError = hipSuccess ;
    int          iSize = 0 ;
    int          iSize1 = 0 ;
    float        *devMinArray = NULL ;
    dim3         dimGrid (iSx3, iSx2, 1) ;
    dim3         dimBlock (iSx1, 1, 1) ;
    bool         isdgammadot0 = false ;
    bool         bMaxwell = false;
    bool         bGamma = false;
    int          iRet = 0; 

#ifdef PAPI_PROF
    char cTimerName[17] = "Eigenstress     " ;
    papistartprofiling_ (cTimerName) ;
#endif

    iSize = sizeof (ST_LAYER) * iSx3 ;
    iSize1 = sizeof (float) * (iSx1 + 2) * iSx2 * iSx3 * 2 ;

    /* Check if maxwell time is present then we need to do a reduction */
    if (cuispresent(dMaxwell))
    {
        bMaxwell = true;
        cuError = hipMalloc((void **) &devMinArray, sizeof (float) * iSx1 * iSx2 * iSx3) ;
        CHECK_CUDA_ERROR ("cuviscouseigen_ : Failed to allocate 0\n", VISCOUS_FREE_EXIT) ;
    }

    cuError = hipMemcpy (pstStruct, pStruct, iSize, hipMemcpyHostToDevice) ;
    CHECK_CUDA_ERROR ("cuviscouseigen_ : memcpy failed 1\n", VISCOUS_FREE_EXIT) ;
        
    iSize = sizeof (ST_TENSOR_LAYER) * iSx3 ;
    cuError = hipMemcpy (pstPrestress, pPrestress, iSize, hipMemcpyHostToDevice) ;
    CHECK_CUDA_ERROR ("cuviscouseigen_ : memcpy failed 2\n", VISCOUS_FREE_EXIT) ;

    if (cuispresent(pGamma))
    {
        bGamma = true ;
        cuError = hipMemset (gpV1, 0, iSize1) ;
        CHECK_CUDA_ERROR ("cuviscouseigen_ : memset failed 1\n", VISCOUS_FREE_EXIT) ;
    }

    /* if dgammadot0 is present then we need to add that to gammadot0 */
    if (cuispresent(dgammadot0))
    {
        isdgammadot0 = true;
        copygammadot0 (iSx1, iSx2, iSx3, dgammadot0, &iRet) ;
    }

    cuViscousEigenKernel <<<dimGrid, dimBlock>>> (pstStruct, pstSig, pstMoment, 
                                                  pstPrestress, gpGammadot0, dMu, 
                                                  iSx1, iSx2, iSx3, dDx1, dDx2, 
                                                  dDx3, devMinArray, gpV1, bMaxwell,
                                                  bGamma, isdgammadot0) ;


    cuError = hipGetLastError () ;
    CHECK_CUDA_ERROR ("cuviscouseigen_ : Kernel launch failed\n", VISCOUS_FREE_EXIT) ;
    if (hipSuccess != hipDeviceSynchronize())
    {
        printf ("cuviscouseigen_: sync failed \n") ;
    }

    if (bMaxwell)
    {
        thrust::device_ptr<float> dev(devMinArray);
        thrust::device_ptr<float> min = thrust::min_element(dev, dev+(iSx1 * iSx2 * iSx3)) ;

        cuError = hipGetLastError () ;
        CHECK_CUDA_ERROR ("cuviscouseigen_ : Thrust min element failure \n", VISCOUS_FREE_EXIT) ;

        *dMaxwell =  (float)*min ;
        hipFree (devMinArray) ;
    }


#ifdef PAPI_PROF
    papiendprofiling_ (cTimerName) ;
#endif
    return ;

VISCOUS_FREE_EXIT:
    cuFreeCudaMemory () ;
}

extern "C" void cutensorfieldadd_ (E_TENSOR_FIELD  eField,
                                   int             iSx1,
                                   int             iSx2,
                                   int             iSx3,
                                   float           fC1,
                                   float           fC2)
{
    dim3 dimGrid (iSx3, iSx2, 1) ;
    dim3 dimBlock (iSx1, 1, 1) ;
#ifdef PAPI_PROF
    char cTimerName[17] = "tensorfieldadd  " ;
    papistartprofiling_ (cTimerName) ;
#endif

    switch (eField)
    {
        case E_TENSOR_TAU_TAU :
        {
            cuTensorFieldKernel <<<dimGrid, dimBlock>>> (pstTau, pstTau, fC1, fC2, iSx1, iSx2, iSx3) ;
            if (hipSuccess != hipDeviceSynchronize())
            {
                printf ("Failed to sync 1\n") ;
            }
        }
        break ;
        case E_TENSOR_SIG_TAU :
        {
            cuTensorFieldKernel <<<dimGrid, dimBlock>>> (pstSig, pstTau, fC1, fC2, iSx1, iSx2, iSx3) ;
            if (hipSuccess != hipDeviceSynchronize())
            {
                printf ("Failed to sync 2\n") ;
            }
        }
        break ;
        case E_TENSOR_MOM_MOM :
        {
            cuTensorFieldKernel <<<dimGrid, dimBlock>>> (pstMoment, pstMoment, fC1, fC2, iSx1, 
                                                         iSx2, iSx3) ;
            if (hipSuccess != hipDeviceSynchronize())
            {
                printf ("Failed to sync 3\n") ;
            }
        }
        break ;
        case E_TENSOR_SIG_MOM :
        {
            cuTensorFieldKernel <<<dimGrid, dimBlock>>> (pstSig, pstMoment, fC1, fC2, iSx1, iSx2, iSx3) ;
            if (hipSuccess != hipDeviceSynchronize())
            {
                printf ("Failed to sync 4\n") ;
            }
        }
        break ;
        case E_TENSOR_TAU_MOM :
        {
            cuTensorFieldKernel <<<dimGrid, dimBlock>>> (pstTau, pstMoment, fC1, fC2, iSx1, iSx2, iSx3) ;
            if (hipSuccess != hipDeviceSynchronize())
            {
                printf ("Failed to sync 5\n") ;
            }
        }
        break ;
        case E_TENSOR_IK_IKDOT:
        {
            cuTensorFieldKernel <<<dimGrid, dimBlock>>> (pstEpsilonik, pstEpsilonikdot, fC1, fC2, iSx1, iSx2, iSx3) ;
            if (hipSuccess != hipDeviceSynchronize())
            {
                printf ("Failed to sync 6\n") ;
            }
        }
        break ;
        case E_TENSOR_IKDOT_IK:
        {
            cuTensorFieldKernel <<<dimGrid, dimBlock>>> (pstEpsilonikdot, pstEpsilonik, fC1, fC2, iSx1, iSx2, iSx3) ;
            if (hipSuccess != hipDeviceSynchronize())
            {
                printf ("Failed to sync 6\n") ;
            }
        }
        break ;
        case E_TENSOR_IKDOT_IKDOT:
        {
            cuTensorFieldKernel <<<dimGrid, dimBlock>>> (pstEpsilonikdot, pstEpsilonikdot, fC1, fC2, iSx1, iSx2, iSx3) ;
            if (hipSuccess != hipDeviceSynchronize())
            {
                printf ("Failed to sync 7\n") ;
            }
        }
        break ; 
        default :
        {
            printf ("cutensorfieldadd_ : The enum sent to this function is wrong %d\n",eField) ;
            return ;
        }
    }

    if (hipSuccess != hipGetLastError())
    {
        printf ("cutensorfieldadd_ : Something went wrong in tensor kernel\n") ;
        cuFreeCudaMemory () ;
    }
    if (hipSuccess != hipDeviceSynchronize())
    {
        printf ("cutensorfieldadd_ : Failed to sync\n") ;
        cuFreeCudaMemory () ;
    }

#ifdef PAPI_PROF
    papiendprofiling_ (cTimerName) ;
#endif

    return ;
}

extern "C" void cucopytraction_ (float *pTract,
                                 int    iSx1,
                                 int    iSx2,
                                 int   *iRet)
{
    *iRet = 0 ;
    if (hipSuccess != hipMemcpy (pTract, pfDevTract3, sizeof (float) * (iSx1+2) * iSx2, 
                                   hipMemcpyDeviceToHost))
    {
        printf ("Error in memcpy\n") ;
        *iRet = 1 ;
    }
}

extern "C" void cudeinit_ ()
{
    destroyPlanForFFT() ;
    cuFreeCudaMemory () ;
}

extern "C" void cutensoramp_ (E_TENSOR_AMP_TYPE  eType,
                              int                iSx1,
                              int                iSx2,
                              int                iSx3,
                              double             *dAmp)
{
    hipError_t     cuError = hipSuccess ;
    dim3        dimGrid (iSx3, iSx2, 1) ;
    dim3        dimBlock (iSx1, 1, 1) ;
    double      *pdTemp = NULL ;

    cuError = hipMalloc((void **) &pdTemp, sizeof (double) * iSx1 * iSx2 * iSx3) ;
    if (cuError != hipSuccess)
    {
        printf ("cutensoramp_ : Failed to allocate 0\n") ;
    }


    switch (eType)
    {
        case E_TENSOR_AMP_MOMENT :
        {
            cuTensorAmpKernel <<<dimGrid, dimBlock>>> (pstMoment, pdTemp, iSx1, iSx2, iSx3) ;
        }
        break ;
        case E_TENSOR_AMP_TAU :
        {
            cuTensorAmpKernel <<<dimGrid, dimBlock>>> (pstTau, pdTemp, iSx1, iSx2, iSx3) ;
        }
        break ;
        default :
        {
            printf ("The enum sent to this function is wrong\n") ;
            return ;
        }
    }
    if (hipSuccess != hipGetLastError())
    {
        printf ("cutensorfieldadd_ : Something went wrong in tensor kernel\n") ;
        cuFreeCudaMemory () ;
    }
    if (hipSuccess != hipDeviceSynchronize())
    {
        printf ("SYnch failed\n") ;
    }

    thrust::device_ptr<double> dev(pdTemp);
    double dSum = thrust::reduce(dev, dev+(iSx1 * iSx2 * iSx3), (double) 0.0, thrust::plus<double>());
    cuError = hipGetLastError () ;
    CHECK_CUDA_ERROR ("cutensoramp_ : Thrust reduce failure \n", TENSOR_EXIT) ;

    *dAmp = dSum ;

    hipFree (pdTemp) ;

    return ;

TENSOR_EXIT:
    hipFree (pdTemp) ;
    cuFreeCudaMemory () ;
}

extern "C" void cufieldadd_ (E_TYPE eType,
                             float *pfVal1,
                             float *pfVal2,
                             float *pfVal3,
                             float *pfVal4,
                             float *pfVal5,
                             float *pfVal6,
                             int   iSx1,
                             int   iSx2,
                             int   iSx3,
                             float fC1,
                             float fC2)
{
    dim3 dimGrid (iSx3, iSx2, 1) ;
    dim3 dimBlock (iSx1/2, 1, 1) ;
    hipError_t cuError = hipSuccess ;

    switch (eType)
    {
        case E_TYPE_U :
        {
            cuFieldAddKernel <<<dimGrid, dimBlock>>> (gpU1, gpV1, fC1, fC2, iSx1, iSx2, iSx3) ;
            cuError = hipGetLastError () ;
            CHECK_CUDA_ERROR ("cufieldadd_: Kernel launch failure 1\n", FIELD_ADD_EXIT) ;

            cuFieldAddKernel <<<dimGrid, dimBlock>>> (gpU2, gpV2, fC1, fC2, iSx1, iSx2, iSx3) ;
            cuError = hipGetLastError () ;
            CHECK_CUDA_ERROR ("cufieldadd_: Kernel launch failure 2\n", FIELD_ADD_EXIT) ;

            cuFieldAddKernel <<<dimGrid, dimBlock>>> (gpU3, gpV3, fC1, fC2, iSx1, iSx2, iSx3) ;
            cuError = hipGetLastError () ;
            CHECK_CUDA_ERROR ("cufieldadd_: Kernel launch failure 3\n", FIELD_ADD_EXIT) ;

            if (hipSuccess != hipDeviceSynchronize())
            {
                printf ("cufieldadd: failed to synchronize\n") ;
            }
        }
        break ;
        case E_TYPE_V :
        {
            cuFieldAddKernel <<<dimGrid, dimBlock>>> (gpV1, gpU1, fC1, fC2, iSx1, iSx2, iSx3) ;
            cuError = hipGetLastError () ;
            CHECK_CUDA_ERROR ("cufieldadd_: Kernel launch failure 4\n", FIELD_ADD_EXIT) ;

            cuFieldAddKernel <<<dimGrid, dimBlock>>> (gpV2, gpU2, fC1, fC2, iSx1, iSx2, iSx3) ;
            cuError = hipGetLastError () ;
            CHECK_CUDA_ERROR ("cufieldadd_: Kernel launch failure 5\n", FIELD_ADD_EXIT) ;

            cuFieldAddKernel <<<dimGrid, dimBlock>>> (gpV3, gpU3, fC1, fC2, iSx1, iSx2, iSx3) ;
            cuError = hipGetLastError () ;
            CHECK_CUDA_ERROR ("cufieldadd_: Kernel launch failure 6\n", FIELD_ADD_EXIT) ;

            if (hipSuccess != hipDeviceSynchronize())
            {
                printf ("cufieldadd: failed to synchronize\n") ;
            }
        }
        break ;
        default :
        {
            printf ("Invalid input\n") ;
        }
    }
    return ;

FIELD_ADD_EXIT:
    printf ("Something wrong in cufieldadd\n") ;
    cuFreeCudaMemory () ;
}

extern "C" void cufieldrep_ (int iSx1,
                             int iSx2,
                             int iSx3)
{
    float *pfdata = NULL ;
    cufieldadd_ (E_TYPE_U, pfdata, pfdata, pfdata, pfdata, pfdata, pfdata, iSx1, iSx2, iSx3, 0.0, 1.0) ;
        
    return ;
}

extern "C" void curesetvectors_ ()
{
    int iSize = 0 ;
    hipError_t cuError = hipSuccess ;

    iSize = (sizeof (float) * (ihSx1+2) * ihSx2 * ihSx3) ;
    cuError = hipMemset (gpV1, 0, iSize) ;
    CHECK_CUDA_ERROR ("curesetvectors_ : Memset failed 1\n", CURESET_FAILURE)
    cuError = hipMemset (gpV2, 0, iSize) ;
    CHECK_CUDA_ERROR ("curesetvectors_ : Memset failed 2\n", CURESET_FAILURE)
    cuError = hipMemset (gpV3, 0, iSize) ;
    CHECK_CUDA_ERROR ("curesetvectors_ : Memset failed 3\n", CURESET_FAILURE)

    iSize = (sizeof (float) * (ihSx1+2) * ihSx2) ;
    cuError = hipMemset (pfDevTract1, 0, iSize) ;
    CHECK_CUDA_ERROR ("curesetvectors_ : Memset failed 4\n", CURESET_FAILURE)
    cuError = hipMemset (pfDevTract2, 0, iSize) ;
    CHECK_CUDA_ERROR ("curesetvectors_ : Memset failed 5\n", CURESET_FAILURE)
    cuError = hipMemset (pfDevTract3, 0, iSize) ;
    CHECK_CUDA_ERROR ("curesetvectors_ : Memset failed 6\n", CURESET_FAILURE)

    return  ;

CURESET_FAILURE:
    cuFreeCudaMemory () ;

}

bool cuispresent (void *pVar = NULL)
{
    int ipresent=0;
    __util_MOD_ispresent(pVar, &ipresent) ;
    return (ipresent == 1) ? true : false ;
}

void copygammadot0 (int        iSx1,
                    int        iSx2,
                    int        iSx3,
                    float      *fGammadot0,
                    int        *iRet)
{
    int         iSize = 0 ;
    hipError_t cuError = hipSuccess ;

    *iRet = 1 ; 
    iSize = sizeof (float) * iSx1 * iSx2 * iSx3 ;
    //allocate 
    if (NULL == gpGammadot0)
    { 
        cuError = hipMalloc((void**)&gpGammadot0, iSize) ;
        if (hipSuccess != cuError)
        {
            printf ("copygammadot0_ : Failed to allocate memory 1\n") ;
            goto COPY_GAMMA_DOT_0; 
        }
    }

    if (hipSuccess != hipMemcpy (gpGammadot0, fGammadot0, iSize, 
                                   hipMemcpyHostToDevice))
    {
            printf ("copygammadot0_ : failed in memcpy 1\n") ;
            goto COPY_GAMMA_DOT_0;
    }

    *iRet = 0 ; 
    return ;

COPY_GAMMA_DOT_0:
    cuFreeCudaMemory () ;

}

extern "C" void copytau_ (ST_TENSOR  *pTemp,
                          int        iSx1,
                          int        iSx2,
                          int        iSx3,
                          int        iForward)
{
    if (1 == iForward)
    {
        if (hipSuccess != hipMemcpy (pstTau, pTemp, sizeof(ST_TENSOR)*iSx1*iSx2*iSx3, 
                                       hipMemcpyHostToDevice))
        {
                printf ("copytau_ : failed in memcpy 1\n") ;
        }
    }
    else
    {
        if (hipSuccess != hipMemcpy (pTemp, pstTau, sizeof(ST_TENSOR)*iSx1*iSx2*iSx3, 
                                       hipMemcpyDeviceToHost))
        {
            printf ("copytau_ : failed in memcpy 2\n") ;
        }
    }
}

extern "C" void cutensormemset_ (E_TENSOR_TYPE eType)
{
    hipError_t cuError = hipSuccess ;

    switch (eType)
    {
        case E_TENSOR_MOMENT:
        {
            cuError = hipMemset (pstMoment, 0, sizeof (ST_TENSOR) * ihSx1 * ihSx2 * ihSx3/2) ;
        }
        break ;
        case E_TENSOR_SIG:
        {
            cuError = hipMemset (pstSig, 0, sizeof (ST_TENSOR) * ihSx1 * ihSx2 * ihSx3/2) ;
        }
        break ;
        case E_TENSOR_TAU:
        {
            cuError = hipMemset (pstTau, 0, sizeof (ST_TENSOR) * ihSx1 * ihSx2 * ihSx3/2) ;
        }
        break ;
        case E_INVALID_TENSOR_TYPE:
        {   
            printf ("Invalid input\n") ;
        }
        break ;
    }
    if (hipSuccess != cuError)
    {
        printf ("cutensormemset: Error in cudamemset\n") ;
    }
}

extern "C" void cuexportspatial_ (float *pTemp1,
                                  float *pTemp2,
                                  float *pTemp3,
                                  int   iIndex3)
{
    hipError_t cuError = hipSuccess ;
    int iIndex = 0 ;
    iIndex = iIndex3*ihSx2*(ihSx1+2) ;

    cuError = hipMemcpy (pTemp1, &gpU1[iIndex], (ihSx1+2)*ihSx2*sizeof (float), hipMemcpyDeviceToHost) ;
    CHECK_CUDA_ERROR ("cuexportspatial_ : Error in memcpy 1\n", CUEXPORT_SPATIAL)

    cuError = hipMemcpy (pTemp2, &gpU2[iIndex], (ihSx1+2)*ihSx2*sizeof (float), hipMemcpyDeviceToHost) ;
    CHECK_CUDA_ERROR ("cuexportspatial_ : Error in memcpy 2\n", CUEXPORT_SPATIAL)

    cuError = hipMemcpy (pTemp3, &gpU3[iIndex], (ihSx1+2)*ihSx2*sizeof (float), hipMemcpyDeviceToHost) ;
    CHECK_CUDA_ERROR ("cuexportspatial_ : Error in memcpy 3\n", CUEXPORT_SPATIAL)

    return ;

CUEXPORT_SPATIAL :
    printf ("Error\n") ;
}
extern "C" void cuexportpoints_ (double         *pfU1,
                                 double         *pfU2,
                                 double         *pfU3,
                                 ST_TENSOR      *pstTemp,
                                 int            iInd1,
                                 int            iInd2,
                                 int            iInd3)
{
    int iIndex1 = 0 ;
    int iIndex2 = 0 ;
    hipError_t cuError = hipSuccess  ;
    float temp1 = 0.0 ;

    iIndex1 = (((iInd3 * ihSx2) + iInd2) * (ihSx1 + 2)) + iInd1 ;
    iIndex2 = (((iInd3 * ihSx2) + iInd2) * ihSx1) + iInd1 ;

    cuError = hipMemcpy (&temp1, &gpU1[iIndex1], sizeof (float), hipMemcpyDeviceToHost) ;
    CHECK_CUDA_ERROR ("cuexportpoints_ : Error in memcpy 1\n", CUEXPORT_EXIT)
    *pfU1 = (double) temp1 ;

    cuError = hipMemcpy (&temp1, &gpU2[iIndex1], sizeof (float), hipMemcpyDeviceToHost) ;
    CHECK_CUDA_ERROR ("cuexportpoints_ : Error in memcpy 2\n", CUEXPORT_EXIT)
    *pfU2 = (double) temp1 ;

    cuError = hipMemcpy (&temp1, &gpU3[iIndex1], sizeof (float), hipMemcpyDeviceToHost) ;
    CHECK_CUDA_ERROR ("cuexportpoints_ : Error in memcpy 3\n", CUEXPORT_EXIT)
    *pfU3 = (double) temp1 ;

    cuError = hipMemcpy (pstTemp, &pstSig[iIndex2], sizeof (ST_TENSOR), hipMemcpyDeviceToHost) ;
    CHECK_CUDA_ERROR ("cuexportpoints_ : Error in memcpy 4\n", CUEXPORT_EXIT)

    return ;

CUEXPORT_EXIT:
    printf ("Error\n") ;

}
extern "C" void cubuildgammadot_ (int     iSx1,
                                  int     iSx2,
                                  int     iSx3,
                                  double  dDx1,
                                  double  dDx2,
                                  double  dDx3,
                                  int     iNz,
                                  ST_WEAK *pDuctile,
                                  float   *pDgammadot0)
{
    hipError_t  cuError = hipSuccess ;
    int          iSize = 0 ;
    int          iSize1 = 0 ;
    dim3         dimGrid (iSx3, iSx2, 1) ;
    dim3         dimBlock (iSx1, 1, 1) ;
    double       dBeta = 0 ;
    float        *pfDgammadot0 = NULL ;
    ST_WEAK      *pstDuctile = NULL ;
    
    iSize = sizeof (ST_WEAK) * iNz ;
    iSize1 = sizeof (float) * iSx1 * iSx2 * iSx3 ; 

    cuError = hipMalloc((void **) &pfDgammadot0, iSize1) ;
    if (hipSuccess != cuError)
    {
        printf ("cubuildgammadot_ : Couldnt allocate memory 1\n") ;
        cuFreeCudaMemory() ;
    }

    cuError = hipMalloc((void **) &pstDuctile, iSize) ;
    if (hipSuccess != cuError)
    {
        printf ("cubuildgammadot_ : Couldnt allocate memory 2\n") ;
        cuFreeCudaMemory() ;
    }

    cuError = hipMemcpy (pstDuctile, pDuctile, iSize, hipMemcpyHostToDevice) ;
    if (hipSuccess != cuError)
    {
        printf ("cubuildgammadot_ : Couldnt copy memory 3\n") ;
        cuFreeCudaMemory() ;
    }

    //call kernel
    cuBuildGammadotKernel <<<dimGrid, dimBlock>>> (iSx1, iSx2, iSx3, dDx1, dDx2,
                                                   dDx3, dBeta, pstDuctile, iNz, 
                                                   pfDgammadot0) ;

    cuError = hipGetLastError () ;
    if (hipSuccess != cuError)
    {
        printf ("cubuildgammadot0_ : Kernel launch failed\n") ;
    }

    if (hipSuccess != hipDeviceSynchronize())
    {
        printf ("cubuildgammadot0_ : sync failed \n") ;
    }

    //copy back

    cuError = hipMemcpy (pDgammadot0, pfDgammadot0, iSize1, hipMemcpyDeviceToHost) ;
    if (hipSuccess != cuError)
    {
        printf ("cubuildgammadot_ : Couldnt copy memory 4\n") ;
        cuFreeCudaMemory() ;
    }

    hipFree(pfDgammadot0);
    hipFree(pstDuctile);

    return ; 
}

extern "C" void cufrictioneigenstress_ (double     dX,
                                        double     dY,
                                        double     dZ,
                                        double     dL,
                                        double     dW,
                                        double     dStrike,
                                        double     dDip,
                                        double     dRake,
                                        double     dBeta,
                                        double     dMu,
                                        ST_LAYER   *pStruct,
                                        int        iSx1,
                                        int        iSx2,
                                        int        iSx3,
                                        double     dDx1,
                                        double     dDx2,
                                        double     dDx3,
                                        int        bPresent,
                                        float      *dMaxwell,
                                        ST_TENSOR  *pMoment,
                                        ST_TENSOR  *pSig)
{
    double      dScaling ;
    double      dCstrike ;
    double      dSstrike ;
    double      dCdip ;
    double      dSdip ;
    double      dCr ;
    double      dSr ;
    double      dLp ;
    double      dWp ;
    double      dX2r ;
    double      dXr ;
    double      dYr ;
    double      dZr ;
    float       *devMinArray ;
    dim3        dimGrid (iSx3, iSx2, 1) ;
    dim3        dimBlock (iSx1, 1, 1) ;
    hipError_t cuError ;
    int         iSize ;

    cuError = hipSuccess ;
    dScaling = sqrt(PI2) * dDx1 ;

    dCstrike = cos(dStrike) ;
    dSstrike = sin(dStrike) ;
    dCdip = cos(dDip) ;
    dSdip = sin(dDip) ;
    dCr = cos(dRake) ;
    dSr = sin(dRake) ;

    dWp = dW*(1.0 + 2.0 * dBeta)/2.0 ;
    dLp = dL*(1.0 + 2.0 * dBeta)/2.0 ;

    dX2r = dCstrike * dX  - dSstrike * dY ;
    dXr = dCdip * dX2r - dSdip * dZ ;
    dYr = dSstrike * dX + dCstrike * dY ;
    dZr = dSdip * dX2r + dCdip * dZ ;

    if (1 == bPresent)
    {
        cuError = hipMalloc((void **) &devMinArray, sizeof (float) * iSx1 * iSx2 * iSx3) ;
        CHECK_CUDA_ERROR ("cufrictioneigenstress_ : Failed to allocate 0\n", FRICTION_FREE_EXIT) ;
    }
    iSize = sizeof (ST_LAYER) * iSx3 ;
    cuError = hipMemcpy (pstStruct, pStruct, iSize, hipMemcpyHostToDevice) ;
    CHECK_CUDA_ERROR ("cufrictioneigenstress_ : memcpy failed 1\n", FRICTION_FREE_EXIT) ;

    cuFrictionStress <<<dimGrid, dimBlock>>> (dScaling, dCstrike, dSstrike, dCdip, dSdip, dCr, dSr,
                                              dWp, dLp, dX2r, dXr, dYr, dZr, iSx1, iSx2, iSx3, dDx1,
                                              dDx2, dDx3, dX, dY, dZ, dL, dW, dRake, dMu, bPresent,  
                                              dBeta, devMinArray, pstMoment, pstSig, pstStruct) ;

    cuError = hipGetLastError () ;
    CHECK_CUDA_ERROR ("cufrictioneigenstress_: Kernel launch failed\n", FRICTION_FREE_EXIT) ;
    if (hipSuccess != hipDeviceSynchronize())
    {
        printf ("cufrictioneigenstress_ : sync failed \n") ;
    }

    if (1 == bPresent)
    {
        thrust::device_ptr<float> dev(devMinArray);
        thrust::device_ptr<float> min = thrust::min_element(dev, dev+(iSx1 * iSx2 * iSx3)) ;

        cuError = hipGetLastError () ;
        CHECK_CUDA_ERROR ("cufrictioneigenstress_ : Thrust min element failure \n", FRICTION_FREE_EXIT) ;

        *dMaxwell =  *min ;
        hipFree (devMinArray) ;
    }

FRICTION_FREE_EXIT:
    return ;
}
/* -------------------------------------- utility functions ----------------------------------- */

hipError_t copyFilter1(double dDx,
                        int    iWhich) 
{
    hipError_t cuError = hipSuccess ;
    double ker ;
    
    memcpy (&ker, CONST_FIR_1, sizeof (double)) ;
    ker /= dDx ;

    switch (iWhich)
    {
        case 1 :
        {
            cuError = hipMemcpyToSymbol(HIP_SYMBOL(constdKer1), &ker, sizeof(ker)) ;
        }
        break ;
        case 2 : 
        {
            cuError = hipMemcpyToSymbol(HIP_SYMBOL(constdKer2), &ker, sizeof(ker)) ;
        }
        break ;
        case 3 : 
        {
            cuError = hipMemcpyToSymbol(HIP_SYMBOL(constdKer3), &ker, sizeof(ker)) ;
        }
        break ;
    }
    return cuError ; 
}

hipError_t copyFilter7(double dDx,
                        int    iWhich) 
{
    hipError_t cuError = hipSuccess ;
    double ker[7] ;
    int     i ;
    
    memcpy (ker, CONST_FIR_7, 7 * sizeof (double)) ;
    for (i = 0 ; i < 7 ; i++)
    {
        ker[i] /= dDx ;
    }

    switch (iWhich)
    {
        case 1 :
        {
            cuError = hipMemcpyToSymbol(HIP_SYMBOL(constdKer1), ker, sizeof(ker)) ;
        }
        break ;
        case 2 :
        {
            cuError = hipMemcpyToSymbol(HIP_SYMBOL(constdKer2), ker, sizeof(ker)) ;
        }
        break ;
        case 3 :
        {
            cuError = hipMemcpyToSymbol(HIP_SYMBOL(constdKer3), ker, sizeof(ker)) ;
        }
        break ;
    }
return cuError ;
}

hipError_t copyFilter14(double dDx,
                         int    iWhich)
{
    hipError_t cuError = hipSuccess ;
    double ker[14] ;
    int     i ;
    
    memcpy (ker, CONST_FIR_14, 14 * sizeof (double)) ;
    for (i = 0 ; i < 14 ; i++)
    {               
        ker[i] /= dDx ; 
    }               

    switch (iWhich)
    {
        case 1 :
        {
            cuError = hipMemcpyToSymbol(HIP_SYMBOL(constdKer1), ker, sizeof(ker)) ;
        }
        break ;
        case 2 :
        {
            cuError = hipMemcpyToSymbol(HIP_SYMBOL(constdKer2), ker, sizeof(ker)) ;
        }
        break ;
        case 3 :
        {
            cuError = hipMemcpyToSymbol(HIP_SYMBOL(constdKer3), ker, sizeof(ker)) ;
        }
        break ;
    }
    if (hipSuccess != cuError)
    {
        printf ("copyFilter14 : %s",hipGetErrorString(cuError)) ;
    }
    return cuError ;
}

int cuOptimalFilter (int       *iLen1,
                     int       *iLen2,
                     int       *iLen3,
                     int       iSx1,
                     int       iSx2,
                     int       iSx3, 
                     double    dDx1,
                     double    dDx2,
                     double    dDx3)
{
    hipError_t  cuError = hipSuccess ;
    int          iRet = 0 ;

    if ((iSx1 > 1) && (iSx1 < 5))
    {
        *iLen1 = 1 ;
        cuError = copyFilter1 (dDx1, 1) ;
    }
    else if ((iSx1 > 4) && (iSx1 < 15))
    {
         *iLen1 = 7 ;
         cuError = copyFilter7 (dDx1, 1) ;
    }
    else if (iSx1 > 15)
    {
         *iLen1 = FILTER_SIZE ;
         switch (*iLen1)
         {
             case 1 : 
                 cuError = copyFilter1 (dDx1, 1) ;
             break ;
             case 7 :
                 cuError = copyFilter7 (dDx1, 1) ;
             break ;
             case 14 : 
                 cuError = copyFilter14 (dDx1, 1) ;
             break ;
         }
    }

    if ((iSx2 > 1) && (iSx2 < 5))
    {
        *iLen2 = 1 ;
        cuError = copyFilter1 (dDx2, 2) ;
    }
    else if ((iSx2 > 4) && (iSx2 < 15))
    {
        *iLen2 = 7 ;
        cuError = copyFilter7 (dDx2, 2) ;
    }
    else if (iSx2 > 15)
    {
         *iLen2 = FILTER_SIZE ;
         switch (*iLen2)
         {
             case 1 :
                 cuError = copyFilter1 (dDx2, 2) ;
             break ;
             case 7 :
                 cuError = copyFilter7 (dDx2, 2) ;
             break ;
             case 14 :
                 cuError = copyFilter14 (dDx2, 2) ;
             break ;
         }      
    }

    if ((iSx3 > 1) && (iSx3 < 5))
    {
        *iLen3 = 1 ;   
        cuError = copyFilter1 (dDx3, 3) ;
    }
    else if ((iSx3 > 4) && (iSx3 < 15))
    {
         *iLen3 = 7 ;
         cuError = copyFilter7 (dDx3, 3) ;
    }
    else if (iSx3 > 15)
    {
         *iLen3 = FILTER_SIZE ;
         switch (*iLen3)
         {
             case 1 :
                 cuError = copyFilter1 (dDx3, 3) ;
             break ;
             case 7 :
                 cuError = copyFilter7 (dDx3, 3) ;
             break ;
             case 14 :
                 cuError = copyFilter14 (dDx3, 3) ;
             break ;
         }
    }

    if (cuError != hipSuccess)
    {
        printf ("cuOptimalFilter: Failed in memcpy 1\n");
        iRet = 1 ;
    }

#ifdef PRINT_DEBUG_INFO
    printf ("cuOptimalFilter: exited with no errors\n") ;
#endif

    return iRet ;
}


void cuFreeCudaMemory()
{
    CUDA_FREE_MEM(pstStruct) ;
    CUDA_FREE_MEM(pstSig) ;
    CUDA_FREE_MEM(pstMoment) ;
    CUDA_FREE_MEM(pstTau) ;
    CUDA_FREE_MEM(pstEpsilonik) ;
    CUDA_FREE_MEM(pstEpsilonikdot) ;
    CUDA_FREE_MEM(pstPrestress) ;
    CUDA_FREE_MEM (gpV1) ;
    CUDA_FREE_MEM (gpV2) ;
    CUDA_FREE_MEM (gpV3) ;
    CUDA_FREE_MEM (gpU1) ;
    CUDA_FREE_MEM (gpU2) ;
    CUDA_FREE_MEM (gpU3) ;
    CUDA_FREE_MEM (pfDevTract1) ;
    CUDA_FREE_MEM (pfDevTract2) ;
    CUDA_FREE_MEM (pfDevTract3) ;
    CUDA_FREE_MEM (gpGammadot0) ;
}

int checkMemRequirement(int iSx1,
                        int iSx2,
                        int iSx3)
{
    int         liReq = 0 ;
    long int    iTemp = 0 ;
    size_t      iTotalMem = 0 ;
    size_t      iFreeMem = 0 ;

    /* Ui's, Vi's and fft's */
    iTemp=((iSx1+2)*iSx2*iSx3*sizeof(float)*8)/(1024*1024) ;
    liReq+=iTemp ;

    /* sig, moment and tau */
    iTemp=((iSx1*iSx2*iSx3/2)*sizeof(ST_TENSOR)*3)/(1024*1024) ;
    liReq+=iTemp ;  

    iTemp=((iSx1*iSx2*iSx3/2)*sizeof(ST_TENSOR)*2)/(1024*1024) ;
    liReq+=iTemp ;  
    
    /* Ti's */
    iTemp=((iSx1+2)*iSx2*sizeof(float)*3)/(1024*1024) ;
    liReq+=iTemp ;
    
    iTemp=(iSx3/2)*sizeof(ST_LAYER)/(1024*1024) ;
    liReq+=iTemp ;

    /* dMinArray */ 
    iTemp=((iSx1+2)*iSx2*iSx3*sizeof(float))/(1024*1024) ;

    hipMemGetInfo(&iFreeMem, &iTotalMem);
    iTotalMem/=(1024*1024) ;    
    
    if ((liReq+iTemp) > iTotalMem)
    {
        printf ("\nTotal memory required is : %d MB\n", (int)(liReq+iTemp)) ;
        printf ("Total available is is : %lu MB \n", iTotalMem) ;
        return -1 ;
    }
    
    return 0;    
}


/* ------------------------------------------- utility end -------------------------------------- */


/* ------------------------------------------- Kernels ------------------------------------------ */

__global__ void cuStressUpdateKernel (int        iLen1,
                                      int        iLen2,
                                      int        iLen3,
                                      double     dLambda,
                                      double     dMu,
                                      int        iSx1,
                                      int        iSx2,
                                      int        iSx3,
                                      float      *gpV1,
                                      float      *gpV2,
                                      float      *gpV3,
                                      ST_TENSOR  *pstSig)
{
    unsigned int iInd1 ;
    unsigned int iInd2 ;
    unsigned int iInd3 ;
    unsigned int iIdx ;
    int  iIndex = 0 ;
    int  iInd1m = 0 ;
    int  iInd2m = 0 ;
    int  iInd3m = 0 ;
    int  iInd1p = 0 ;
    int  iInd2p = 0 ;
    int  iInd3p = 0 ;
    int  iTemp = 0 ;
    
    int  iOffp = 0 ;
    int  iOffm = 0 ;
    ST_TENSOR  stT = {0};

#ifdef STRESS_SHARED_MEM
    __shared__ float shdV11 [256] ;
    __shared__ float shdV21 [256] ;
    __shared__ float shdV31 [256] ;
#endif

    iInd1 = threadIdx.x ;
    iInd2 = blockIdx.y ;
    iInd3 = blockIdx.x ;

#ifdef STRESS_SHARED_MEM
    iIdx = (((iInd3 * iSx2) + iInd2)  * iSx1) + iInd1 ;

    if (0 == iInd1)
    {
        for (iIndex = 0; iIndex < iSx1 ; iIndex++)
        {
            shdV11[iIndex] = gpV1[iIdx] ;
            shdV21[iIndex] = gpV2[iIdx] ;
            shdV31[iIndex] = gpV3[iIdx] ;
            iIdx++ ;    
        }
    
    }
    __syncthreads() ;
#endif



    if ((iInd1 < iSx1) && (iInd2 < iSx2) && ((iInd3 > (iLen3-1)) && (iInd3 < (iSx3-iLen3))))
    {
        iIdx = (((iInd3 * iSx2) + iInd2)  * iSx1) + iInd1 ;

#ifdef STRESS_SHARED_MEM
        iTemp = iSx1 - 1 ;
        for (iIndex = 0; iIndex < iLen1; iIndex++)
        {
            iInd1m = ((iTemp+iInd1-iIndex) % (iSx1)) ;
            iInd1p = ((iInd1+iIndex) % (iSx1)) + 1 ;
            iInd1p = (iInd1p % iSx1) ;
            
            stT.s11 += ((shdV11[iInd1p] - shdV11[iInd1m]) * constdKer1[iIndex]) ;
            stT.s12 += ((shdV21[iInd1p] - shdV21[iInd1m]) * constdKer1[iIndex]) ;
            stT.s13 += ((shdV31[iInd1p] - shdV31[iInd1m]) * constdKer1[iIndex]) ;
        }
#else
        iTemp = iSx1 - 1 ;
        for (iIndex = 0; iIndex < iLen1; iIndex++)
        {
            iInd1m = ((iTemp+iInd1-iIndex) % (iSx1)) ;
            iInd1p = ((iInd1+iIndex) % (iSx1)) + 1 ;
            iInd1p = (iInd1p % iSx1) ;

            iOffp = (((iInd3 * iSx2) + iInd2) * (iSx1 + 2)) + iInd1p ;
            iOffm = (((iInd3 * iSx2) + iInd2) * (iSx1 + 2)) + iInd1m ;

            stT.s11 += ((gpV1[iOffp] - gpV1[iOffm]) * constdKer1[iIndex]) ;
            stT.s12 += ((gpV2[iOffp] - gpV2[iOffm]) * constdKer1[iIndex]) ;
            stT.s13 += ((gpV3[iOffp] - gpV3[iOffm]) * constdKer1[iIndex]) ;
        }
#endif
        iTemp = iSx2 - 1 ;

        for (iIndex = 0; iIndex < iLen2; iIndex++)
        {
            iInd2m = ((iTemp+iInd2-iIndex) % (iSx2)) ;
            iInd2p = ((iInd2+iIndex) % (iSx2)) + 1 ;
            iInd2p = (iInd2p % iSx2) ;

            iOffp = (((iInd3 * iSx2) + iInd2p) * (iSx1 + 2)) + iInd1 ;
            iOffm = (((iInd3 * iSx2) + iInd2m) * (iSx1 + 2)) + iInd1 ;

            stT.s12 += ((gpV1[iOffp] - gpV1[iOffm]) * constdKer2[iIndex]) ;
            stT.s22 += ((gpV2[iOffp] - gpV2[iOffm]) * constdKer2[iIndex]) ;
            stT.s23 += ((gpV3[iOffp] - gpV3[iOffm]) * constdKer2[iIndex]) ;
        }

        for (iIndex = 1; iIndex <= iLen3; iIndex++)
        {
            iInd3m = iInd3 - iIndex ;
            iInd3p = iInd3 + iIndex ;

            iOffp = (((iInd3p * iSx2) + iInd2) * (iSx1 + 2)) + iInd1 ;
            iOffm = (((iInd3m * iSx2) + iInd2) * (iSx1 + 2)) + iInd1 ;

            stT.s13 += ((gpV1[iOffp] - gpV1[iOffm]) * constdKer3[iIndex-1]) ;
            stT.s23 += ((gpV2[iOffp] - gpV2[iOffm]) * constdKer3[iIndex-1]) ;
            stT.s33 += ((gpV3[iOffp] - gpV3[iOffm]) * constdKer3[iIndex-1]) ;
        }

        stT.s12 /= 2.0 ;
        stT.s13 /= 2.0 ;
        stT.s23 /= 2.0 ;

        cuIsotrpicStressStrain (&stT, dLambda, dMu) ;
        cuTensorOperate (&pstSig[iIdx], (void *)&stT, '+') ;

    }

}


__global__ void cuLocalStressStrainKernel (int        iInd3m,
                                           int        iInd3p, 
                                           int        iLen1, 
                                           int        iLen2, 
                                           int        iInd3, 
                                           double     dPx3, 
                                           double     dLambda,
                                           double     dMu, 
                                           int        iSx1, 
                                           int        iSx2, 
                                           int        iSx3,
                                           float      *gpV1, 
                                           float      *gpV2, 
                                           float      *gpV3,
                                           ST_TENSOR  *pstSig) 
{
    ST_TENSOR stT ;
    int iInd1 = 0 ;
    int iInd2 = 0 ;
    int iIdx = 0 ;

    iInd2 = blockIdx.x ;
    iInd1 = threadIdx.x ;

    cuTensorMemset(&stT) ;
            
    iIdx = (((iInd3 * iSx2) + iInd2)  * (iSx1)) + iInd1 ;                        
    
    cuLocalStrain_ani (&stT, iInd3m, iInd3p, dPx3, iLen1, iLen2,
               iInd1, iInd2, iInd3, gpV1, gpV2, gpV3, iSx1, iSx2, iSx3) ;
        
            
    cuIsotrpicStressStrain (&stT, dLambda, dMu) ;

    cuTensorOperate (&pstSig[iIdx], (void *)&stT, '+') ;

}


__global__ void cuEquiBodyKernel (ST_TENSOR   *pstT,
                                  int         iLen1,
                                  int         iLen2,
                                  int         iLen3,
                                  int         iSx1,
                                  int         iSx2,
                                  int         iSx3,
                                  float       *fData1,
                                  float       *fData2,
                                  float       *fData3)
{
    double  f1 ;
    double  f2 ;
    double  f3 ;
    int     iInd1 = 0 ;
    int     iInd2 = 0 ;
    int     iInd3 = 0 ;
    int     iIdx = 0 ;
    int     iIndex = 0 ;
    int     iIndm = 0 ;
    int     iIndp = 0 ;
    int     iTemp = 0 ;
    int     iOffp = 0 ;
    int     iOffm = 0 ;

#ifdef ENABLE_REG_BLOCKING
    float   fTen_1_s1[14] ;
    float   fTen_1_s2[14] ;
    float   fTen_1_s3[14] ;
    float   fTen_2_s1[14] ;
    float   fTen_2_s2[14] ;
    float   fTen_2_s3[14] ;
#endif

    ST_TENSOR   stTemp1 ;
    ST_TENSOR   stTemp2 ;

    iInd1 = threadIdx.x ;
    iInd2 = blockIdx.y ;
    iInd3 = blockIdx.x  + iLen3 ;

    if ((iInd1 < iSx1) && (iInd2 < iSx2) && ((iInd3 > (iLen3-1)) && (iInd3 < (iSx3-iLen3))))
    {
#ifdef ENABLE_REG_BLOCKING
        iTemp = iSx1 - 1 ;
        for (iIndex = 0; iIndex < iLen1; iIndex++)
        {
            iIndm = ((iTemp+iInd1-iIndex) % (iSx1)) ;
            iIndp = (((iInd1+iIndex) % (iSx1)) + 1) % iSx1 ;

            iOffm = (((iInd3 * iSx2) + iInd2) * (iSx1)) + iIndm ;
            iOffp = (((iInd3 * iSx2) + iInd2) * (iSx1)) + iIndp ;

            stTemp1 = pstT[iOffp] ;
            stTemp2 = pstT[iOffm] ;
            fTen_1_s1[iIndex] = stTemp1.s11 ;
            fTen_1_s2[iIndex] = stTemp1.s12 ;
            fTen_1_s3[iIndex] = stTemp1.s13 ;

            fTen_2_s1[iIndex] = stTemp2.s11 ;
            fTen_2_s2[iIndex] = stTemp2.s12 ;
            fTen_2_s3[iIndex] = stTemp2.s13 ;
        }

#pragma unroll 
        for (iIndex = 0; iIndex < iLen1; iIndex++)
        {
            f1 += ((fTen_1_s1[iIndex] - fTen_2_s1[iIndex]) * constdKer1[iIndex]) ;
            f2 += ((fTen_1_s2[iIndex] - fTen_2_s2[iIndex]) * constdKer1[iIndex]) ;
            f3 += ((fTen_1_s3[iIndex] - fTen_2_s3[iIndex]) * constdKer1[iIndex]) ;
        }

#else
        iTemp = iSx1 - 1 ;
        for (iIndex = 0; iIndex < iLen1; iIndex++)
        {
            iIndm = ((iTemp+iInd1-iIndex) % (iSx1)) ;
            iIndp = (((iInd1+iIndex) % (iSx1)) + 1) % iSx1 ;

            iOffm = (((iInd3 * iSx2) + iInd2) * (iSx1)) + iIndm ;
            iOffp = (((iInd3 * iSx2) + iInd2) * (iSx1)) + iIndp ;

            stTemp1 = pstT[iOffp] ;
            stTemp2 = pstT[iOffm] ;

            f1 += ((stTemp1.s11 - stTemp2.s11) * constdKer1[iIndex]) ;
            f2 += ((stTemp1.s12 - stTemp2.s12) * constdKer1[iIndex]) ;
            f3 += ((stTemp1.s13 - stTemp2.s13) * constdKer1[iIndex]) ;
        }
        iTemp = iSx2 - 1 ;

        for (iIndex = 0; iIndex < iLen2; iIndex++)
        {
            iIndm = ((iTemp+iInd2-iIndex) % (iSx2)) ;
            iIndp = (((iInd2+iIndex) % (iSx2)) + 1) % iSx2 ;

            iOffm = (((iInd3 * iSx2) + iIndm) * iSx1) + iInd1 ;
            iOffp = (((iInd3 * iSx2) + iIndp) * iSx1) + iInd1 ;

            stTemp1 = pstT[iOffp] ;
            stTemp2 = pstT[iOffm] ;

            f1 += ((stTemp1.s12 - stTemp2.s12) * constdKer2[iIndex]) ;
            f2 += ((stTemp1.s22 - stTemp2.s22) * constdKer2[iIndex]) ;
            f3 += ((stTemp1.s23 - stTemp2.s23) * constdKer2[iIndex]) ;
        }

        for (iIndex = 1; iIndex <= iLen3; iIndex++)
        {
            iIndm = iInd3 - iIndex ;
            iIndp = iInd3 + iIndex ;

            iOffm = (((iIndm * iSx2) + iInd2) * iSx1) + iInd1 ;
            iOffp = (((iIndp * iSx2) + iInd2) * iSx1) + iInd1 ;

            stTemp1 = pstT[iOffp] ;
            stTemp2 = pstT[iOffm] ;

            f1 += ((stTemp1.s13 - stTemp2.s13) * constdKer3[iIndex-1]) ;
            f2 += ((stTemp1.s23 - stTemp2.s23) * constdKer3[iIndex-1]) ;
            f3 += ((stTemp1.s33 - stTemp2.s33) * constdKer3[iIndex-1]) ;
        }
#endif

        iIdx = (((iInd3 * iSx2) + iInd2) * (iSx1 + 2)) + iInd1 ;

        fData1[iIdx] -= f1 ;
        fData2[iIdx] -= f2 ;
        fData3[iIdx] -= f3 ;
    }
}


__global__ void cuLocalDivergenceKernel (int          iInd3m,
                                         int          iInd3p,
                                         int          iLen1,
                                         int          iLen2,
                                         int          iInd3,
                                         double       dPx3,
                                         int          iSx1,
                                         int          iSx2,
                                         int          iSx3,
                                         float        *gpV1,
                                         float        *gpV2,
                                         float        *gpV3,
                                         ST_TENSOR    *pstSig)
{
    int     iInd1 = 0 ;
    int     iInd2 = 0 ;
    int     iIdx = 0 ;
    double  f1 = 0.0 ;
    double  f2 = 0.0 ;
    double  f3 = 0.0 ;

    iInd2 = blockIdx.x ;
    iInd1 = threadIdx.x ;

    if ((iInd1 < iSx1) && (iInd2 < iSx2))
    {
        cuLocalDivergence_ani (pstSig, iInd3m, iInd3p, dPx3, iLen1, iLen2, iInd1, iInd2, iInd3,
                               &f1, &f2, &f3, iSx1, iSx2, iSx3) ;

        iIdx = (((iInd3 * iSx2) + iInd2) * (iSx1 + 2)) + iInd1 ;

        gpV1[iIdx] -= f1 ;
        gpV2[iIdx] -= f2 ;
        gpV3[iIdx] -= f3 ;
    }
}

__global__ void cuEquivalentTraction (float      *pfTraction1,
                                      float      *pfTraction2,
                                      float      *pfTraction3,
                                      ST_TENSOR  *pstSig,
                                      int        iSx1,
                                      int        iSx2)
{
    unsigned int iIdx1 = 0 ;
    unsigned int iIdx2 = 0 ;
    unsigned int iIdx = 0 ;

    iIdx1 = threadIdx.x ;
    iIdx2 = blockIdx.x ;

    if ((iIdx1 < iSx1) && (iIdx2 < iSx2))
    {
        iIdx = (iIdx2 * (iSx1 + 2)) + iIdx1 ;

        pfTraction1[iIdx] += pstSig[iIdx].s13 ;
        pfTraction2[iIdx] += pstSig[iIdx].s23 ;
        pfTraction3[iIdx] += pstSig[iIdx].s33 ;
    }
}

__global__ void cuSourceForceKernel (int     iSx1,
                                     int     iSx2,
                                     int     iSx3,
                                     double  dDx1,
                                     double  dDx2,
                                     double  dDx3,
                                     double  dcStrike,
                                     double  dsStrike,
                                     double  dcDip,
                                     double  dsDip,
                                     double  dCr,
                                     double  dSr,
                                     double  dScale,
                                     double  dWp,
                                     double  dLp,
                                     double  dX2r,
                                     double  dXr,
                                     double  dYr,
                                     double  dZr,
                                     double  dX,
                                     double  dY,
                                     double  dZ,
                                     double  dDip,
                                     double  dW,
                                     double  dL,
                                     double  dBeta,
                                     double  dMu,
                                     double  dS,
                                     float   *pfData1,
                                     float   *pfData2,
                                     float   *pfData3)
{
    int     iInd1 = 0 ;
    int     iInd2 = 0 ;
    int     iInd3 = 0 ;
    int     iIdx = 0 ;
    int     iTemp = 0 ;

    double  dX1 ;
    double  dX2 ;
    double  dX3 ;

    double  dX1s ;
    double  dX2s ;
    double  dX3s ;

    double  dX1i ;
    double  dX3i ;

    double  dSource ;
    double  dTemp1 ;
    double  dTemp2 ;
    double  dTemp3 ;

    double  dDblcp ;
    double  dCplei ;
//      double          dDipci ;
    double  dImage ;
    double  dDipcs ;

    iInd1 = threadIdx.x ;
    iInd2 = blockIdx.y ;
    iInd3 = blockIdx.x ;

    if ((iInd1 < iSx1) && (iInd2 < iSx2) && (iInd3 < iSx3/2))
    {
        cuShiftedCoordinates (1, 1, iInd3, iSx1, iSx2, iSx3,
                              dDx1, dDx2, dDx3, &dX1, &dX2, &dX3) ;
        iTemp = ((abs(dX3 - dZ) > dLp) && (abs(dX3 + dZ) > dLp)) ;
        if (0 == iTemp)
        {
            cuShiftedCoordinates (iInd1, iInd2, iInd3, iSx1, iSx2, iSx3,
                                  dDx1, dDx2, dDx3, &dX1, &dX2, &dX3) ;

            iTemp = ((abs(dX1 - dX) > MAX_NUM(dWp, dLp)) || (abs(dX2 - dY) > MAX_NUM(dWp, dLp))) ;
            if (0 == iTemp)
            {
                dX2r = (dcStrike * dX1) - (dsStrike * dX2) ;
                dX1s = (dcDip * dX2r) - (dsDip * dX3) ;
                dX1i = (dcDip * dX2r) + (dsDip * dX3) ;

                iTemp = ((abs(dX1s - dXr) > (7.01 * dDx1)) && (abs(dX1i - dXr) > (7.01 * dDx1))) ;
                if (0 == iTemp)
                {
                    dX2s = (dsStrike * dX1) + (dcStrike * dX2) ;
                    dX3s = (dsDip * dX2r) + (dcDip * dX3) ;
                    dX3i = (-dsDip * dX2r) + (dcDip * dX3) ;

                    dTemp1 = cuGauss (dX1s - dXr, dDx1) ;
                    dTemp2 = cuOmega ((dX2s - dYr)/dW, dBeta) ;
                    dTemp3 = cuOmega ((dX3s - dZr)/dL, dBeta) ;

                    dSource = dScale * cuGaussp (dX1s-dXr, dDx1) * dTemp2 * dTemp3 ;
                    dDblcp = (dScale / dW) * dTemp1 * cuOmegap ((dX2s-dYr)/dW, dBeta) * dTemp3 ;
                    dDipcs = (dScale / dL) * dTemp1 * dTemp2 * cuOmegap ((dX3s-dZr)/dL, dBeta) ;

                    dTemp1 = cuGauss (dX1i - dXr, dDx1) ;
                    dTemp3 = cuOmega ((dX3i + dZr)/dL, dBeta) ;
                    dImage = dScale * cuGaussp (dX1i-dXr, dDx1) * dTemp2 * dTemp3 ;
                    dCplei = (dScale / dW) * dTemp1 * cuOmegap ((dX2s-dYr)/dW, dBeta) * dTemp3 ;
//                                    dDipci = (dScale / dL) * dTemp1 * dTemp2 * cuOmegap ((dX3i+dZr)/dL, dBeta) ;

                    iIdx = ((iInd3 * iSx2) + iInd2) * (iSx1 + 2) + iInd1 ;
                    if ((2.01 * DEG2RAD) > dDip)
                    {
                        pfData1[iIdx] += ((dCr * dsStrike * (dSource + dImage)) +
                                          (dCr * dcDip * dcStrike * (dDblcp + dCplei))) ;
                        pfData2[iIdx] += ((dCr * dcStrike * (dSource + dImage)) -
                                          (dCr * dcDip * dsStrike * (dDblcp + dCplei))) ;
                        pfData3[iIdx] -= (dCr * dsDip * dDblcp) ;
                    }
                    else
                    {
                        pfData1[iIdx] += ((dCr * dsStrike * dSource) +
                                          (dCr * dcDip * dcStrike * dDblcp)) ;
                        pfData2[iIdx] += ((dCr * dcStrike * (dSource)) -
                                          (dCr * dcDip * dsStrike * (dDblcp))) ;
                        pfData3[iIdx] -= (dCr * dsDip * dDblcp) ;
                    }

                    pfData1[iIdx] += ((dcDip * dSr * dcStrike * dDipcs) +
                                      (dsDip * dSr * dcStrike * dSource)) ;
                    pfData2[iIdx] -= ((dcDip * dSr * dsStrike * dDipcs) +
                                      (dsDip * dSr * dsStrike * dSource)) ;
                    pfData3[iIdx] += ((dcDip * dSr * dSource) - (dsDip * dSr * dDipcs)) ;
                }
            }
        }
    }
}

__global__ void cuSourceTractionKernel (int     iSx1,
                                        int     iSx2,
                                        int     iSx3,
                                        double  dDx1,
                                        double  dDx2,
                                        double  dDx3,
                                        double  dcStrike,
                                        double  dsStrike,
                                        double  dcDip,
                                        double  dsDip,
                                        double  dCr,
                                        double  dSr,
                                        double  dScale,
                                        double  dWp,
                                        double  dLp,
                                        double  dX2r,
                                        double  dXr,
                                        double  dYr,
                                        double  dZr,
                                        double  dX,
                                        double  dY,
                                        double  dW,
                                        double  dL,
                                        double  dBeta,
                                        double  dMu,
                                        double  dS,
                                        float   *pfTract1,
                                        float   *pfTract2,
                                        float   *pfTract3)
{
    int     iInd1 = 0 ;
    int     iInd2 = 0 ;
    int     iInd3 = 0 ;
    int     iIdx = 0 ;
    int     iTemp = 0 ;

    double  dX1 ;
    double  dX2 ;
    double  dX3 ;

    double  dX1s ;
    double  dX2s ;
    double  dX3s ;

    double  dX1i ;
    double  dX3i ;

    double  dSource = 0.0 ;
    double  dTemp1 ;
    double  dTemp2 ;
    double  dTemp3 ;

    double  dN[3] = {0.0} ;
    double  dB[3] = {0.0} ;
    double  dBmod[3] = {0.0} ;


    ST_TENSOR  dM ;

    iInd1 = threadIdx.x ;
    iInd2 = blockIdx.x ;
    iInd3 = 0 ;

    if((iInd1 < iSx1) && (iInd2 < iSx2))
    {
        cuShiftedCoordinates (iInd1, iInd2, iInd3, iSx1, iSx2, iSx3,
                              dDx1, dDx2, dDx3, &dX1, &dX2, &dX3) ;

        iTemp = ((abs(dX1 - dX) > MAX_NUM(dWp, dLp)) || (abs(dX2 - dY) > MAX_NUM(dWp, dLp))) ;
        if (0 == iTemp)
        {
            dX2r = (dcStrike * dX1) - (dsStrike * dX2) ;
            dX1s = (dcDip * dX2r) - (dsDip * dX3) ;
            dX1i = (dcDip * dX2r) + (dsDip * dX3) ;

            iTemp = ((abs(dX1s - dXr) > (7.01 * dDx1)) && (abs(dX1i - dXr) > (7.01 * dDx1))) ;
            if (0 == iTemp)
            {
                dX2s = (dsStrike * dX1) - (dcStrike * dX2) ;
                dX3s = (dsDip * dX2r) - (dcDip * dX3) ;
                dX3i = (-dsDip * dX2r) + (dcDip * dX3) ;

                dTemp1 = cuGauss (dX1s - dXr, dDx1) ;
                dTemp2 = cuOmega ((dX2s - dYr)/dW, dBeta) ;
                dTemp3 = cuOmega ((dX3s - dZr)/dL, dBeta) ;
                dSource += (dTemp1 * dTemp2 * dTemp3) ;

                dTemp1 = cuGauss (dX1i - dXr, dDx1) ;
                dTemp3 = cuOmega ((dX3i + dZr)/dL, dBeta) ;
                dSource += (dTemp1 * dTemp2 * dTemp3) ;

                dN[0] = dcDip * dcStrike * dSource ;
                dN[1] = -dcDip * dsStrike * dSource ;
                dN[2] = -dsDip * dSource ;


                dB[0] = dsStrike * dCr ;
                dB[1] = dcStrike * dCr ;

                dB[0] += (dcStrike * dsDip * dSr) ;
                dB[1] -= (dsStrike * dsDip * dSr) ;
                dB[2] = dcDip * dSr ;

                dBmod[0] = dMu * dS * dB[0] ;
                dBmod[1] = dMu * dS * dB[1] ;
                dBmod[2] = dMu * dS * dB[2] ;

                //dyadic product
                cuTensorDyadProd (&dM, dN, dBmod) ;

                iIdx = (iInd2 * (iSx1 + 2)) + iInd1 ;

                pfTract1[iIdx] += (float)dM.s13 ;
                pfTract2[iIdx] += (float)dM.s23 ;
                pfTract3[iIdx] += (float)dM.s33 ;
            }
        }
    }
}

__global__ void cuTransientEigenKernel (ST_LAYER          *pstStruct,
                                        ST_TENSOR         *pstSig,
                                        ST_TENSOR         *pstMoment,
                                        float             *gpGammadot0,
                                        ST_TENSOR         *pstEpsilonik,
                                        ST_TENSOR         *pstEpsilonikdot,
                                        double            dMu,
                                        int               iSx1,
                                        int               iSx2,
                                        int               iSx3,
                                        double            dDx1,
                                        double            dDx2,
                                        double            dDx3,
                                        int               bMaxwell,
                                        float             *dMinArray,
                                        bool              isdgammadot0)
{
    int        iInd1 = 0 ;
    int        iInd2 = 0 ;
    int        iInd3 = 0 ;
    int        iIdx = 0 ;
    double     dPower = 0.0 ;
    double     dMuk = 0.0 ;
    double     dGammaDot0 = 0.0 ;
    double     dGammaDot = 0.0 ;
    double     dNq = 0.0 ;
    ST_TENSOR  stS = {0} ;
    ST_TENSOR  stQ = {0} ;
    ST_TENSOR  stTemp = {0} ;
    double     dTemp = 0.0 ;
    int        iCond = 0 ;

    iInd3 = blockIdx.x ;
    iInd2 = blockIdx.y ;
    iInd1 = threadIdx.x ;

    if ((iInd1 < iSx1) && (iInd2 < iSx2) && (iInd3 < iSx3))
    {
        iIdx = (((iInd3 * iSx2) + iInd2) * iSx1) + iInd1 ;
        if (bMaxwell)
        {
            dMinArray[iIdx] = 1.0e+30 ;
        }
        dPower = pstStruct[iInd3].stressexponent - 1 ;
        dMuk = pstStruct[iInd3].Gk;

        dGammaDot0 = pstStruct[iInd3].gammadot0 ;

        if (isdgammadot0) 
        {
            dGammaDot0 += gpGammadot0[iIdx] ;
        }

        iCond = (1.0e-20 > dGammaDot0) ? 1 : 0 ;
        if (0 == iCond)
        {
            cuTensorDeviatoric (&pstSig[iIdx], &stS) ;

            dTemp = 0.0 ;            
            cuTensorOperate (&stTemp, (void *)&dTemp, '*') ;

            dTemp = 2 * dMuk ;
            cuTensorOperate (&stTemp, (void *)(&pstEpsilonik[iIdx]), '=') ;

            // 2Gk*epsilonik
            cuTensorOperate (&stTemp, (void *)&dTemp, '*') ;

            cuTensorOperate (&stQ, (void *)&stS, '=') ;
            //  Q = (sigma - 2Gk*epsilonik)
            cuTensorOperate (&stQ, (void *)&stTemp, '-') ;

            // q = || Q ||
            dNq = cuTensorNorm(&stQ) ;
             
            // powerlaw viscosity
            dTemp = dNq/dMu ;
            dGammaDot = dGammaDot0 * pow (dTemp, dPower) ;
            
            cuTensorOperate (&pstEpsilonikdot[iIdx], (void *)&stQ, '=') ;

            dTemp = dGammaDot / dMu ;
            cuTensorOperate (&pstEpsilonikdot[iIdx], (void *)&dTemp, '*') ;

            dTemp = 2 * dGammaDot ;
            cuTensorOperate (&stQ, (void *)&dTemp, '*') ;

            // update moment density forcing
            cuTensorOperate (&pstMoment[iIdx], (void *)&stQ, '+') ;

            if (bMaxwell)
            {
                dTemp = 1 / dGammaDot ;
                if (0 != dTemp)
                {
                        dMinArray[iIdx] = (float)dTemp ;
                }
            }
        }
        else
        {
            dTemp=0.0;
            cuTensorOperate (&pstEpsilonikdot[iIdx], (void *)&dTemp, '*') ;
        }
    }
}

__global__ void cuViscousEigenKernel (ST_LAYER          *pstStruct,
                                      ST_TENSOR         *pstSig,
                                      ST_TENSOR         *pstMoment,
                                      ST_TENSOR_LAYER   *pstPrestress,
                                      float             *gpGammadot0,
                                      double            dMu,
                                      int               iSx1,
                                      int               iSx2,
                                      int               iSx3,
                                      double            dDx1,
                                      double            dDx2,
                                      double            dDx3,
                                      float             *dMinArray,
                                      float             *pGamma,
                                      bool              bPresent,
                                      bool              bGammaPresent,
                                      bool              isdgammadot0)
{
    int        iInd1 = 0 ;
    int        iInd2 = 0 ;
    int        iInd3 = 0 ;
    int        iIdx = 0 ;
    double     dPower = 0.0 ;
    double     dCohesion = 0.0 ;
    double     dGammaDot0 = 0.0 ;
    double     dGammaDot = 0.0 ;
    double     dGammaDotp = 0.0 ;
    double     dTau = 0.0 ;
    double     dTauc = 0.0 ;
    double     dTaup = 0.0 ;
    ST_TENSOR  stS = {0} ;
    ST_TENSOR  stP = {0} ;
    ST_TENSOR  stSP = {0} ;
    double     dTemp = 0.0 ;
    int        iCond = 0 ;

    iInd3 = blockIdx.x ;
    iInd2 = blockIdx.y ;
    iInd1 = threadIdx.x ;

    if ((iInd1 < iSx1) && (iInd2 < iSx2) && (iInd3 < iSx3))
    {
        iIdx = (((iInd3 * iSx2) + iInd2) * iSx1) + iInd1 ;
        if (bPresent)
        {
                dMinArray[iIdx] = 1.0e+30 ;
        }

        dPower = pstStruct[iInd3].stressexponent ;
        dCohesion = pstStruct[iInd3].cohesion ;

        cuTensorDeviatoric (&pstPrestress[iInd3].t, &stP) ;
        dTaup = cuTensorNorm(&stP) ;

        dGammaDot0 = pstStruct[iInd3].gammadot0 ;

        if (isdgammadot0) 
        {
            dGammaDot0 += gpGammadot0[iIdx] ;
        }

        iCond = (1.0e-20 > dGammaDot0) ? 1 : 0 ;

        if (0 == iCond)
        {
            cuTensorDeviatoric (&pstSig[iIdx], &stS) ;

            dTemp=0.0;
            cuTensorOperate (&stSP, (void *)&dTemp, '*') ;
            cuTensorOperate (&stSP, (void *)&stS, '+') ;
            cuTensorOperate (&stSP, (void *)&stP, '+') ;

            dTau = cuTensorNorm(&stSP) ;

            dTauc = MAX_NUM(0, dTau - dCohesion) ;

            iCond = (dTauc <= 1.0e-20) ? 1 : 0 ;
            if (0 == iCond)
            {
                dTemp = dTauc/dMu ;
                dGammaDot = dGammaDot0 * pow (dTemp, dPower-1) ;
                dTemp = dTaup/dMu ;
                dGammaDotp = dGammaDot0 * pow (dTaup/dMu, dPower-1) ;

                dTemp = 2 * dGammaDot ;
                cuTensorOperate (&stS, (void *)&dTemp, '*') ;
                
                dTemp = 2 * dGammaDotp ;
                cuTensorOperate (&stP, (void *)&dTemp, '*') ;

                cuTensorOperate (&pstMoment[iIdx], (void *)&stS, '+') ;
                cuTensorOperate (&pstMoment[iIdx], (void *)&stP, '-') ;

                /*if (1 == bGammaPresent)
                {
                        iIdx = (((iInd3 * iSx2) + iInd2) * (iSx1+2)) + iInd1 ;
                        pGamma[iIdx] = (float) dGammaDot ;
                }*/
                if (bPresent)
                {
                    dTemp = 1 / dGammaDot ;
                    iIdx = (((iInd3 * iSx2) + iInd2) * iSx1) + iInd1 ;
                    if (0 != dTemp)
                    {
                            dMinArray[iIdx] = (float)dTemp ;
                    }
                }
            }
        }
    }
}

__global__ void cuTensorFieldKernel (ST_TENSOR  *pstTens1,
                                     ST_TENSOR  *pstTens2,
                                     float      fC1,
                                     float      fC2,
                                     int        iSx1,
                                     int        iSx2,
                                     int        iSx3)
{
    int        iInd1 = 0 ;
    int        iInd2 = 0 ;
    int        iInd3 = 0 ;
    int        iIdx = 0 ;
    ST_TENSOR  stTemp = {0} ;
    double     dC1 = fC1 ;
    double     dC2 = fC2 ;

    iInd3 = blockIdx.x ;
    iInd2 = blockIdx.y ;
    iInd1 = threadIdx.x ;

    if ((iInd1 < iSx1) && (iInd2 < iSx2) && (iInd3 < iSx3))
    {
        iIdx = (((iInd3 * iSx2) + iInd2) * iSx1) + iInd1 ;
        cuTensorOperate (&stTemp, &pstTens2[iIdx], '+') ;
        cuTensorOperate (&pstTens1[iIdx], (void *)&dC1, '*') ;
        cuTensorOperate (&stTemp, (void *)&dC2, '*') ;
        cuTensorOperate (&pstTens1[iIdx], &stTemp, '+') ;
    }
}

__global__ void cuTensorAmpKernel (ST_TENSOR    *pstTensor,
                                   double       *pdArray,
                                   int           iSx1,
                                   int           iSx2,
                                   int           iSx3)
{
    int     iInd1 = 0 ;
    int     iInd2 = 0 ;
    int     iInd3 = 0 ;
    int     iIdx = 0 ;
    double  dTemp = 0.0 ;

    iInd3 = blockIdx.x ;
    iInd2 = blockIdx.y ;
    iInd1 = threadIdx.x ;

    if ((iInd1 < iSx1) && (iInd2 < iSx2) && (iInd3 < iSx3))
    {
        iIdx = (((iInd3 * iSx2) + iInd2) * iSx1) + iInd1 ;
        dTemp = cuTensorNorm(&pstTensor[iIdx]) ;
        pdArray[iIdx] = dTemp ;
    }
}

__global__ void cuBuildGammadotKernel (int           iSx1,
                                       int           iSx2,
                                       int           iSx3,
                                       double        dDx1,
                                       double        dDx2,
                                       double        dDx3,
                                       double        dBeta, 
                                       ST_WEAK       *pstDuctile,
                                       int           iNz,
                                       float         *pfDgammadot0)
{
    int     iInd1 = 0 ;
    int     iInd2 = 0 ;
    int     iInd3 = 0 ;
    int     iIdx = 0 ;
    
    double     dX1 ;
    double     dX2 ;
    double     dX3 ;
    double     dDum ;


    iInd3 = blockIdx.x ;
    iInd2 = blockIdx.y ;
    iInd1 = threadIdx.x ;

    if ((iInd1 < iSx1) && (iInd2 < iSx2) && (iInd3 < iSx3))
    {
        dX3 = (iInd3) * dDx3 ;
        cuShiftedCoordinates (iInd1, iInd2, iInd3, iSx1, iSx2, iSx3,
                              dDx1, dDx2, dDx3, &dX1, &dX2, &dDum) ;
    
        iIdx = (((iInd3 * iSx2) + iInd2) * iSx1) + iInd1 ;
        pfDgammadot0[iIdx] = cuDgGammaDotNot(pstDuctile, iNz, dX1, dX2, dX3, dBeta) ; 
    }
}

__global__ void cuFieldAddKernel (float  *pfData1,
                                  float  *pfData2,
                                  float  fC1,
                                  float  fC2,
                                  int    iSx1,
                                  int    iSx2,
                                  int    iSx3)
{
    int     iInd1 = 0 ;
    int     iInd2 = 0 ;
    int     iInd3 = 0 ;
    int     iIdx = 0 ;
    float   fTemp = 0 ;
    float2  *fData1 ;
    float2  *fData2 ;

    iInd3 = blockIdx.x ;
    iInd2 = blockIdx.y ;
    iInd1 = threadIdx.x ;

    if ((iInd1 < iSx1) && (iInd2 < iSx2) && (iInd3 < iSx3))
    {
        iIdx = (((iInd3 * iSx2) + iInd2) * iSx1) + 2 * iInd1 ;
        fData1 = (float2 *) &pfData1[iIdx] ;
        fData2 = (float2 *) &pfData2[iIdx] ;

        fTemp = (float) (fC2 * (*fData2).x) ;
        (*fData1).x = (float) (fC1 * (*fData1).x) ;
        (*fData1).x += fTemp ;

        fTemp = (float) (fC2 * (*fData2).y) ;
        (*fData1).y = (float) (fC1 * (*fData1).y) ;
        (*fData1).y += fTemp ;
    }
}

__global__ void cuFrictionStress (double     dScaling,
                                  double     dcStrike,
                                  double     dsStrike,
                                  double     dcDip,
                                  double     dsDip,
                                  double     dCr,
                                  double     dSr,
                                  double     dWp,
                                  double     dLp,
                                  double     dX2r,
                                  double     dXr,
                                  double     dYr,
                                  double     dZr,
                                  int        iSx1,
                                  int        iSx2,
                                  int        iSx3,
                                  double     dDx1,
                                  double     dDx2,
                                  double     dDx3,
                                  double     dX,
                                  double     dY,
                                  double     dZ,
                                  double     dL,
                                  double     dW,
                                  double     dRake,
                                  double     dMu,
                                  int        bPresent,
                                  double     dBeta,
                                  float      *dMinArray,
                                  ST_TENSOR  *pMoment,
                                  ST_TENSOR  *pSig,
                                  ST_LAYER   *pStruct)
{
    double     dN[3] ;
    double     dnTemp[3] ;
    double     dR[3] ;
    double     dT[3] ;
    double     dTs[3] ;

    int        iTemp ;
    int        iIdx ;
    int        iIdx1 ;
    int        iIdx2 ;
    int        iIdx3 ;

    double     dX1s ;
    double     dX2s ;
    double     dX3s ;

    double     dX1i ;
    double     dX3i ;

    double     dTemp1 ;
    double     dTemp2 ;
    double     dTemp3 ;

    double     dX1 ;
    double     dX2 ;
    double     dDum ;

    // make these shared 
    double     dVo ;
    double     dTauc ;
    double     dFriction ;
    double     dCohesion ;
    double     dX3 ;

    double     dSource ;
    double     dImage ;
    double     dImpulse ;

    double     dTau ;
    double     dTaun ;
    double     dTaus ;
    ST_TENSOR  stTen ;
    double     dZero = 0.0 ;

    double     dGammaDot ;

    dN[0] = dcDip * dcStrike ;
    dN[1] = -dcDip * dsStrike ;
    dN[2] = -dsDip ;

dR[0] = dsStrike * dCr + dcStrike * dsDip * dSr ;
    dR[1] = dcStrike * dCr - dsStrike * dsDip * dSr ;
    dR[2] = dcDip * dSr ;

    iIdx3 = blockIdx.x ;
    iIdx2 = blockIdx.y ;
    iIdx1 = threadIdx.x ;

    if ((iIdx1 < iSx1) && (iIdx2 < iSx2) && (iIdx3 < iSx3))
    {
        iIdx = (((iIdx3 * iSx2) + iIdx2) * iSx1) + iIdx1 ;
        if (1 == bPresent)
        {
            dMinArray[iIdx] = 1.0e+30 ;
        }

        // These should go in shared memory.
        dX3 = (iIdx3) * dDx3 ;
        iTemp = (fabs(dX3 - dZ) > dLp) && (fabs(dX3 + dZ) > dLp) ;
        if (0 == iTemp)
        {
           dVo = pStruct[iIdx3].gammadot0 ;
           dTauc = pStruct[iIdx3].stressexponent ;
           dFriction = pStruct[iIdx3].friction ;
           dCohesion = pStruct[iIdx3].cohesion ;

           cuShiftedCoordinates (iIdx1, iIdx2, iIdx3, iSx1, iSx2, iSx3,
                                 dDx1, dDx2, dDx3, &dX1, &dX2, &dDum) ;
           iTemp = ((fabs(dX1 - dX) > MAX_NUM(dWp, dLp)) || (fabs(dX2 - dY) > MAX_NUM(dWp, dLp))) ;
           if (0 == iTemp)
           {
                dX2r = (dcStrike * dX1) - (dsStrike * dX2) ;
                dX1s = (dcDip * dX2r) - (dsDip * dX3) ;
                dX1i = (dcDip * dX2r) + (dsDip * dX3) ;

                iTemp = ((fabs(dX1s - dXr) > (7.01 * dDx1)) && (fabs(dX1i - dXr) > (7.01 * dDx1))) ;
                if (0 == iTemp)
                {
                    dX2s = (dsStrike * dX1) + (dcStrike * dX2) ;
                    dX3s = (dsDip * dX2r) + (dcDip * dX3) ;
                    dX3i = (-dsDip * dX2r) + (dcDip * dX3) ;

                    dTemp1 = cuGauss (dX1s - dXr, dDx1) ;
                    dTemp2 = cuOmega ((dX2s - dYr)/dW, dBeta) ;
                    dTemp3 = cuOmega ((dX3s - dZr)/dL, dBeta) ;

                    dSource = dTemp1 * dTemp2 * dTemp3 ;
                    dTemp1 = cuGauss ((dX1i - dXr), dDx1) ;
                    dTemp3 = cuOmega ((dX3i + dZr)/dL, dBeta) ;
                    dImage = dTemp1 * dTemp2 * dTemp3 ;

                    dImpulse = dSource + dImage ;

                    stTen = pSig[iIdx] ;

                    cutDot (&stTen, dN, dT) ;
                    dTaun = cuSum (dT, dN) ;
                    cuMulSub (dTaun, dT, dN, dTs) ;

                                    //replace sqrt
                    dTemp2 = cuSum (dTs, dTs) ;

                    dTaus = sqrt (dTemp2) ;

                    dTemp1 = dTaus + (dFriction * dTaun) - dCohesion ;
                    dTau = MAX_NUM(dZero, dTemp1) ;

                    dTemp1 = cuSum (dTs, dR) ;
                    iTemp = (dTemp1 < 0.0) && (fabs(dRake) < (PI2 * 1.5)) ;
                    if (0 == iTemp)
                    {
                        dTs[0] = dTs[0] / dTaus ;
                        dTs[1] = dTs[1] / dTaus ;
                        dTs[2] = dTs[2] / dTaus ;
                        dGammaDot = dVo * 2.0 * mycuSinh (dTau/dTauc) ;

                        dTemp3 = MIN (dL, dW) ;
                        dTemp1 = ((dTau / dMu / dGammaDot) *
                        (dTemp3 / sqrt (dDx1 * dDx2))) ;

                        if (1 == bPresent)
                        {
                            dMinArray[iIdx] = (float) dTemp1 ;
                        }
                        /*if (1 == bPresent)
                        {
                        iIdx = (((iIdx3 * iSx2) + iIdx2) * (iSx1 + 2)) + iIdx1 ;
                        gpV1[iIdx] += (dGammaDot * dImpulse * dScaling) ;
                        }*/

                        dnTemp[0] = dN[0] * (2.0 * dMu * dImpulse * dGammaDot) ;
                        dnTemp[1] = dN[1] * (2.0 * dMu * dImpulse * dGammaDot) ;
                        dnTemp[2] = dN[2] * (2.0 * dMu * dImpulse * dGammaDot) ;

                        cuTensorDyadProd (&stTen, dTs, dnTemp) ;

                        cuTensorOperate (&pMoment[iIdx], (void *)&stTen, '+') ;
                    }
                }
            }
        }
    }
}

/* --------------------------------- __global__ functions end ---------------------------------- */


/* --------------------------------- host/device functions ------------------------------------- */

__device__ __host__ void print_tensor (ST_TENSOR    *pstT)
{
    printf ("%e,%e,%e,%e,%e,%e\n",pstT->s11, pstT->s12, pstT->s13, pstT->s22, pstT->s23, pstT->s33) ;
}


__host__ __device__ double cuTensorTrace (ST_TENSOR    *pstT)
{
    return (double)(pstT->s11 + pstT->s22 + pstT->s33) ;
}


__host__ __device__ void cuIsotrpicStressStrain (ST_TENSOR    *pstT,
                                                 double       dLambda,
                                                 double       dMu)
{
    double dEpskk ; 
    double dTemp = 0.0 ;
    
    dTemp = 2.0 * dMu ; 
    //dEpskk = cuTensorTrace (pstT) ;
    dEpskk = (double)(pstT->s11 + pstT->s22 + pstT->s33) ;
    cuTensorOperate (pstT, &dTemp, '*') ;
    
    pstT->s11 += (dLambda * dEpskk) ;
    pstT->s22 += (dLambda * dEpskk) ;
    pstT->s33 += (dLambda * dEpskk) ; 
}


__device__ void culocalstrain_fir2 (ST_TENSOR  *pstT,
                                    int        iLen1,
                                    int        iLen2,
                                    int        iLen3,
                                    int        iInd1,
                                    int        iInd2,
                                    int        iInd3,
                                    float      *gpV1,
                                    float      *gpV2,
                                    float      *gpV3,
                                    int        iSx1,
                                    int        iSx2,
                                    int        iSx3)
{
    int     iIndex = 0 ;
    int     iInd1m = 0 ;
    int     iInd2m = 0 ;
    int     iInd3m = 0 ;
    int     iInd1p = 0 ;
    int     iInd2p = 0 ;
    int     iInd3p = 0 ;
    int     iTemp = 0 ;

    int     iOffp = 0 ;
    int     iOffm = 0 ;

    iTemp = iSx1 - 1 ;
    for (iIndex = 0; iIndex < iLen1; iIndex++)
    {
        iInd1m = ((iTemp+iInd1-iIndex) % (iSx1)) ;
        iInd1p = ((iInd1+iIndex) % (iSx1)) + 1 ;
        iInd1p = (iInd1p % iSx1) ;

        iOffp = (((iInd3 * iSx2) + iInd2) * (iSx1 + 2)) + iInd1p ;
        iOffm = (((iInd3 * iSx2) + iInd2) * (iSx1 + 2)) + iInd1m ;

        pstT->s11 += ((gpV1[iOffp] - gpV1[iOffm]) * constdKer1[iIndex]) ;
        pstT->s12 += ((gpV2[iOffp] - gpV2[iOffm]) * constdKer1[iIndex]) ;
        pstT->s13 += ((gpV3[iOffp] - gpV3[iOffm]) * constdKer1[iIndex]) ;
    }
    
    iTemp = iSx2 - 1 ;

    for (iIndex = 0; iIndex < iLen2; iIndex++)
    {
        iInd2m = ((iTemp+iInd2-iIndex) % (iSx2)) ;
        iInd2p = ((iInd2+iIndex) % (iSx2)) + 1 ;
        iInd2p = (iInd2p % iSx2) ;

        iOffp = (((iInd3 * iSx2) + iInd2p) * (iSx1 + 2)) + iInd1 ;
        iOffm = (((iInd3 * iSx2) + iInd2m) * (iSx1 + 2)) + iInd1 ;

        pstT->s12 += ((gpV1[iOffp] - gpV1[iOffm]) * constdKer2[iIndex]) ;
        pstT->s22 += ((gpV2[iOffp] - gpV2[iOffm]) * constdKer2[iIndex]) ;
        pstT->s23 += ((gpV3[iOffp] - gpV3[iOffm]) * constdKer2[iIndex]) ;
    }

    for (iIndex = 1; iIndex <= iLen3; iIndex++)
    {
        iInd3m = iInd3 - iIndex ; 
        iInd3p = iInd3 + iIndex ; 
    
        iOffp = (((iInd3p * iSx2) + iInd2) * (iSx1 + 2)) + iInd1 ;
        iOffm = (((iInd3m * iSx2) + iInd2) * (iSx1 + 2)) + iInd1 ; 

        pstT->s13 += ((gpV1[iOffp] - gpV1[iOffm]) * constdKer3[iIndex-1]) ;
        pstT->s23 += ((gpV2[iOffp] - gpV2[iOffm]) * constdKer3[iIndex-1]) ;
        pstT->s33 += ((gpV3[iOffp] - gpV3[iOffm]) * constdKer3[iIndex-1]) ;
    }

    pstT->s12 /= 2.0 ;
    pstT->s13 /= 2.0 ;
    pstT->s23 /= 2.0 ;
}

__device__ void cuLocalStrain_ani (ST_TENSOR  *pstT, 
                                   int        iInd3m, 
                                   int        iInd3p, 
                                   double     dPx3,
                                   int        iLen1, 
                                   int        iLen2, 
                                   int        iInd1, 
                                   int        iInd2, 
                                   int        iInd3, 
                                   float      *gpV1,
                                   float      *gpV2,
                                   float      *gpV3, 
                                   int        iSx1, 
                                   int        iSx2, 
                                   int        iSx3)
{
    int     iIndex = 0 ;
    int     iInd1m = 0 ;
    int     iInd2m = 0 ;
    int     iInd1p = 0 ;
    int     iInd2p = 0 ;
    int     iTemp = 0 ;
    int     iOffp = 0 ;
    int     iOffm = 0 ;

    iTemp = iSx1 - 1 ; 

    for (iIndex = 0; iIndex < iLen1; iIndex++)
    {
        iInd1m = ((iTemp+iInd1-iIndex) % (iSx1)) ;  
        iInd1p = ((iInd1+iIndex) % (iSx1)) + 1 ;
        iInd1p = (iInd1p % iSx1) ;
        
        iOffp = (((iInd3 * iSx2) + iInd2) * (iSx1 + 2)) + iInd1p ;
        iOffm = (((iInd3 * iSx2) + iInd2) * (iSx1 + 2)) + iInd1m ;
                
        pstT->s11 += ((gpV1[iOffp] - gpV1[iOffm]) * constdKer1[iIndex]) ;   
        pstT->s12 += ((gpV2[iOffp] - gpV2[iOffm]) * constdKer1[iIndex]) ;   
        pstT->s13 += ((gpV3[iOffp] - gpV3[iOffm]) * constdKer1[iIndex]) ;   
    }
    iTemp = iSx2 - 1 ; 
    
    for (iIndex = 0; iIndex < iLen2; iIndex++)
    {
        iInd2m = ((iTemp+iInd2-iIndex) % (iSx2)) ; 
        iInd2p = ((iInd2+iIndex) % (iSx2)) + 1 ;
        iInd2p = (iInd2p % iSx2) ;

        iOffp = (((iInd3 * iSx2) + iInd2p) * (iSx1 + 2)) + iInd1 ;
        iOffm = (((iInd3 * iSx2) + iInd2m) * (iSx1 + 2)) + iInd1 ;

        pstT->s12 += ((gpV1[iOffp] - gpV1[iOffm]) * constdKer2[iIndex]) ;
        pstT->s22 += ((gpV2[iOffp] - gpV2[iOffm]) * constdKer2[iIndex]) ;
        pstT->s23 += ((gpV3[iOffp] - gpV3[iOffm]) * constdKer2[iIndex]) ;
    }

    iOffp = (((iInd3p * iSx2) + iInd2) * (iSx1 + 2)) + iInd1 ;
    iOffm = (((iInd3m * iSx2) + iInd2) * (iSx1 + 2)) + iInd1 ;

    pstT->s13 += ((gpV1[iOffp] - gpV1[iOffm]) / dPx3) ;
    pstT->s23 += ((gpV2[iOffp] - gpV2[iOffm]) / dPx3) ;
    pstT->s33 = ((gpV3[iOffp] - gpV3[iOffm]) / dPx3) ; 

    pstT->s12 /= 2.0 ;
    pstT->s13 /= 2.0 ;
    pstT->s23 /= 2.0 ; 
}

__device__ void cuLocalDivergence_ani (ST_TENSOR  *pstT,
                                       int        iInd3m,
                                       int        iInd3p,
                                       double     dPx3,
                                       int        iLen1,  
                                       int        iLen2,  
                                       int        iInd1,  
                                       int        iInd2,
                                       int        iInd3, 
                                       double     *pF1,
                                       double     *pF2,
                                       double     *pF3,
                                       int        iSx1,                   
                                       int        iSx2,   
                                       int        iSx3)   
{
    int     iIndex = 0 ;
    int     iInd1m = 0 ;
    int     iInd2m = 0 ;
    int     iInd1p = 0 ;
    int     iInd2p = 0 ;
    int     iTemp = 0 ;
    int     iOffp = 0 ;
    int     iOffm = 0 ;

    iTemp = iSx1 - 1 ;

    for (iIndex = 0; iIndex < iLen1; iIndex++)
    {
        iInd1m = ((iTemp+iInd1-iIndex) % (iSx1)) ;
        iInd1p = ((iInd1+iIndex) % (iSx1)) + 1 ;
        iInd1p = (iInd1p % iSx1) ;

        iOffp = (((iInd3 * iSx2) + iInd2) * (iSx1)) + iInd1p ;
        iOffm = (((iInd3 * iSx2) + iInd2) * (iSx1)) + iInd1m ;

        *pF1 += ((pstT[iOffp].s11 - pstT[iOffm].s11) * constdKer1[iIndex]) ;
        *pF2 += ((pstT[iOffp].s12 - pstT[iOffm].s12) * constdKer1[iIndex]) ;
        *pF3 += ((pstT[iOffp].s13 - pstT[iOffm].s13) * constdKer1[iIndex]) ;
    }
    iTemp = iSx2 - 1 ;

    for (iIndex = 0; iIndex < iLen2; iIndex++)
    {
        iInd2m = ((iTemp+iInd2-iIndex) % (iSx2)) ;
        iInd2p = ((iInd2+iIndex) % (iSx2)) + 1 ;
        iInd2p = (iInd2p % iSx2) ;

        iOffp = (((iInd3 * iSx2) + iInd2p) * iSx1) + iInd1 ;
        iOffm = (((iInd3 * iSx2) + iInd2m) * iSx1) + iInd1 ;

        *pF1 += ((pstT[iOffp].s12 - pstT[iOffm].s12) * constdKer2[iIndex]) ;
        *pF2 += ((pstT[iOffp].s22 - pstT[iOffm].s22) * constdKer2[iIndex]) ;
        *pF3 += ((pstT[iOffp].s23 - pstT[iOffm].s23) * constdKer2[iIndex]) ;
    }

    iOffp = (((iInd3p * iSx2) + iInd2) * iSx1) + iInd1 ;
    iOffm = (((iInd3m * iSx2) + iInd2) * iSx1) + iInd1 ;

    *pF1 += ((pstT[iOffp].s13 - pstT[iOffm].s13) / dPx3) ;
    *pF2 += ((pstT[iOffp].s23 - pstT[iOffm].s23) / dPx3) ;
    *pF3 += ((pstT[iOffp].s33 - pstT[iOffm].s33) / dPx3) ;
}

__device__ void cuLocalDivergence_fir (ST_TENSOR  *pstT,
                                       int        iLen1,
                                       int        iLen2,
                                       int        iLen3,
                                       int        iInd1,
                                       int        iInd2,
                                       int        iInd3,
                                       double     *pF1,
                                       double     *pF2,
                                       double     *pF3,
                                       int        iSx1,
                                       int        iSx2,
                                       int        iSx3)
{
    int     iIndex = 0 ;
    int     iIndm = 0 ;
    int     iIndp = 0 ;
    int     iTemp = 0 ;
    int     iOffp = 0 ;
    int     iOffm = 0 ;

    iTemp = iSx1 - 1 ;

    for (iIndex = 0; iIndex < iLen1; iIndex++)
    {
        iIndm = ((iTemp+iInd1-iIndex) % (iSx1)) ;
        iIndp = ((iInd1+iIndex) % (iSx1)) + 1 ;
        iIndp = (iIndp % iSx1) ;

        iOffp = (((iInd3 * iSx2) + iInd2) * (iSx1)) + iIndp ;
        iOffm = (((iInd3 * iSx2) + iInd2) * (iSx1)) + iIndm ;

        *pF1 += ((pstT[iOffp].s11 - pstT[iOffm].s11) * constdKer1[iIndex]) ;
        *pF2 += ((pstT[iOffp].s12 - pstT[iOffm].s12) * constdKer1[iIndex]) ;
        *pF3 += ((pstT[iOffp].s13 - pstT[iOffm].s13) * constdKer1[iIndex]) ;
    }
    iTemp = iSx2 - 1 ;

    for (iIndex = 0; iIndex < iLen2; iIndex++)
    {
        iIndm = ((iTemp+iInd2-iIndex) % (iSx2)) ;
        iIndp = ((iInd2+iIndex) % (iSx2)) + 1 ;
        iIndp = (iIndp % iSx2) ;

        iOffp = (((iInd3 * iSx2) + iIndp) * iSx1) + iInd1 ;
        iOffm = (((iInd3 * iSx2) + iIndm) * iSx1) + iInd1 ;

        *pF1 += ((pstT[iOffp].s12 - pstT[iOffm].s12) * constdKer2[iIndex]) ;
        *pF2 += ((pstT[iOffp].s22 - pstT[iOffm].s22) * constdKer2[iIndex]) ;
        *pF3 += ((pstT[iOffp].s23 - pstT[iOffm].s23) * constdKer2[iIndex]) ;
    }

    for (iIndex = 1; iIndex <= iLen3; iIndex++)
    {
        iIndm = iInd3 - iIndex ;
        iIndp = iInd3 + iIndex ;

        iOffp = (((iIndp * iSx2) + iInd2) * iSx1) + iInd1 ;
        iOffm = (((iIndm * iSx2) + iInd2) * iSx1) + iInd1 ;

        *pF1 += ((pstT[iOffp].s13 - pstT[iOffm].s13) * constdKer3[iIndex-1]) ;
        *pF2 += ((pstT[iOffp].s23 - pstT[iOffm].s23) * constdKer3[iIndex-1]) ;
        *pF3 += ((pstT[iOffp].s33 - pstT[iOffm].s33) * constdKer3[iIndex-1]) ;
    }
}

__host__ __device__ void cuTensorMemset (ST_TENSOR *pstT)
{
    pstT->s11 = 0 ;
    pstT->s12 = 0 ;
    pstT->s13 = 0 ;
    pstT->s22 = 0 ;
    pstT->s23 = 0 ;
    pstT->s33 = 0 ;
}

__host__ __device__ void cuTensorOperate (ST_TENSOR  *pstT,
                                          void       *pTemp,
                                          char       cOp)
{
    double      *dTemp ;
    ST_TENSOR   *pstTemp ;

    switch (cOp)
    {
        case '+':
        {
            pstTemp = (ST_TENSOR *) pTemp ;
            pstT->s11 += pstTemp->s11 ;
            pstT->s12 += pstTemp->s12 ;
            pstT->s13 += pstTemp->s13 ;
            pstT->s22 += pstTemp->s22 ;
            pstT->s23 += pstTemp->s23 ;
            pstT->s33 += pstTemp->s33 ;
        }
        break ;

        case '-':
        {
            pstTemp = (ST_TENSOR *) pTemp ;
            pstT->s11 -= pstTemp->s11 ;
            pstT->s12 -= pstTemp->s12 ;
            pstT->s13 -= pstTemp->s13 ;
            pstT->s22 -= pstTemp->s22 ;
            pstT->s23 -= pstTemp->s23 ;
            pstT->s33 -= pstTemp->s33 ;
        }
        break ;
        case '*':
        {
            dTemp = (double *) pTemp ;
            pstT->s11 *= (*dTemp) ;
            pstT->s12 *= (*dTemp) ;
            pstT->s13 *= (*dTemp) ;
            pstT->s22 *= (*dTemp) ;
            pstT->s23 *= (*dTemp) ;
            pstT->s33 *= (*dTemp) ;
        }
        break ;
        case '=':
        {
            pstTemp = (ST_TENSOR *) pTemp ;
            pstT->s11 = pstTemp->s11 ;
            pstT->s12 = pstTemp->s12 ;
            pstT->s13 = pstTemp->s13 ;
            pstT->s22 = pstTemp->s22 ;
            pstT->s23 = pstTemp->s23 ;
            pstT->s33 = pstTemp->s33 ;
        }
        break ;
    }
}

__host__ __device__ double cuGauss (double dX, 
                                    double dSigma)
{
    double dTemp = 0.0 ;
    
    dTemp = (exp (-0.5 * (dX / dSigma) * (dX / dSigma))) / (sqrt (PI2) * dSigma) ;
    
    return dTemp ;
} 

__host__ __device__ double cuOmega (double  dX,
                                    double  dBeta)
{
    double dTemp = 0.0 ; 
    double dInter = 0.0 ; 
    
    dInter = (1.0 - (2.0 * dBeta)) / (2.0 * (1.0 - dBeta)) ;
  
    if (abs (dX) <= dInter)
    {
        dTemp = 1.0 ;
    }
    else
    {
        if (abs (dX) < (1.0 / (2.0 * (1.0 - dBeta))))
        {
            dInter = cos (PI * ((1.0 - dBeta) * abs (dX) - 0.5 + dBeta) / (2.0 * dBeta)) ;
            dTemp = dInter * dInter ;
            
        }   
        else 
        {
            dTemp = 0.0 ;
        }
    }

    return dTemp ;
}

__host__ __device__ double cuGaussp (double dX,
                                     double dSigma)
{
    double dTemp = 0.0 ;

    dTemp = -dX * (exp (-0.5 * (dX / dSigma) * (dX / dSigma))) / (sqrt (PI2) * (dSigma * dSigma * dSigma)) ;
    return dTemp ;
}

__host__ __device__ double cuOmegap (double      dX,
                                     double      dBeta)
{
    double dTemp = 0.0 ;
    double dInter = 0.0 ;

    dInter = (1.0 - (2.0 * dBeta)) / (2.0 * (1.0 - dBeta)) ;

    if (abs (dX) > dInter)
    {
        if (abs (dX) < (1.0 / (2.0 * (1.0 - dBeta))))
        {
            dTemp = -DSIGN(1.0, dX) * PI * (1.0 - dBeta) / (2.0 * dBeta) * 
            sin (PI * ((1.0 - dBeta) * abs (dX) - 0.5 + dBeta) / dBeta) ;   
        }
    }

    return dTemp ;
}

__host__ __device__ void cuTensorDyadProd (ST_TENSOR *pstT,
                                           double    *pdA,
                                           double    *pdB)
{
    pstT->s11 = pdA[0] * pdB[0] ;
    pstT->s12 = ((pdA[0] * pdB[1]) + (pdA[1] * pdB[0])) / 2.0 ;
    pstT->s13 = ((pdA[0] * pdB[2]) + (pdA[2] * pdB[0])) / 2.0 ;
    pstT->s22 = pdA[1] * pdB[1] ;
    pstT->s23 = ((pdA[1] * pdB[2]) + (pdA[2] * pdB[1])) / 2.0 ; 
    pstT->s33 = pdA[2] * pdB[2] ;

    return ;
}

__host__ __device__ void cuShiftedCoordinates (int     iInd1, 
                                               int     iInd2,
                                               int     iInd3,
                                               int     iSx1,
                                               int     iSx2,
                                               int     iSx3,
                                               double  dDx1, 
                                               double  dDx2, 
                                               double  dDx3, 
                                               double  *pdX1,
                                               double  *pdX2,
                                               double  *pdX3)
{
    if (iInd1 <= iSx1/2)
    {
        *pdX1 = iInd1 * dDx1 ;
    }
    else
    {
        *pdX1 = (iInd1 - iSx1) * dDx1 ;  
    }

    if (iInd2 <= iSx2/2)
    {
        *pdX2 = iInd2 * dDx2 ; 
    }
    else
    {
        *pdX2 = (iInd2 - iSx2) * dDx2 ;
    }

    if (iInd3 <= iSx3/2)
    {
        *pdX3 = iInd3 * dDx3 ; 
    }
    else
    {
        *pdX3 = (iInd3 - iSx3) * dDx3 ;
    }
}

__host__ __device__ double cuTensorNorm (ST_TENSOR *pstTemp)
{
    double dTemp = 0.0 ;
    double dVal = 2.0 ;
    double dSqr = 0.0 ;

/* Took me 2 days to find this bug (Nightmare) */
    double s11 = (double) pstTemp->s11 ;
    double s12 = (double) pstTemp->s12 ;
    double s13 = (double) pstTemp->s13 ;
    double s22 = (double) pstTemp->s22 ;
    double s23 = (double) pstTemp->s23 ;
    double s33 = (double) pstTemp->s33 ;

    dTemp = ((s11 * s11) + 2.0 * (s12 * s12) + 2.0 * (s13 * s13) + (s22 * s22) + 2.0 * 
            (s23 * s23) + (s33 * s33)) / dVal;

    dSqr = sqrt (dTemp) ;

    return dSqr ;
}


__host__ __device__ void cuTensorDecompose (ST_TENSOR   *pstT,
                                            double      *dGamma,
                                            ST_TENSOR   *pstR)
{
    *dGamma = (float)cuTensorNorm (pstT) ;

    pstR->s11 = pstT->s11 / (*dGamma) ;
    pstR->s12 = pstT->s12 / (*dGamma) ;
    pstR->s13 = pstT->s13 / (*dGamma) ;
    pstR->s22 = pstT->s22 / (*dGamma) ;
    pstR->s23 = pstT->s23 / (*dGamma) ;
    pstR->s33 = pstT->s33 / (*dGamma) ;
}

__host__ __device__ void cuTensorDeviatoric(ST_TENSOR *pstSrc,
                                            ST_TENSOR *pstDest)
{
    float dDiag ;

    dDiag = cuTensorTrace (pstSrc) / 3.0 ;

    pstDest->s11 = pstSrc->s11 - dDiag ;
    pstDest->s12 = pstSrc->s12 ;
    pstDest->s13 = pstSrc->s13 ;
    pstDest->s22 = pstSrc->s22 - dDiag ;
    pstDest->s23 = pstSrc->s23 ;
    pstDest->s33 = pstSrc->s33 - dDiag ;
}

__host__ __device__ double cuDgGammaDotNot (ST_WEAK     *pstZones,
                                            int         iN,
                                            double      dX1,
                                            double      dX2,
                                            double      dX3,
                                            double      dBeta)
{
    double   dDg ;
    double   dX ;
    double   dY ;
    double   dZ ;
    double   dL ;
    double   dW ;
    double   dD ;
    double   dStrike ;
    double   dDip ;
    double   dLM ;
    double   dX1s ;
    double   dX2s ;
    double   dX3s ;


    double   dcStrike ;
    double   dsStrike ;
    double   dcDip ;
    double   dsDip ;

    double   dX2r ;
    double   dXr ;
    double   dYr ;
    double   dZr ;

    double   dWp ;
    double   dLp ;
    double   dDp ;

    double   dDgGammaDot0 = 0.0 ;
    int      iInd = 0 ;

    for (iInd = 0; iInd < iN; iInd++)
    {
        dDg = pstZones[iInd].dgammadot0 ;

        dX = pstZones[iInd].x ;
        dY = pstZones[iInd].y ;
        dZ = pstZones[iInd].z ;

        // ask  
        dL = pstZones[iInd].width ;
        dW = pstZones[iInd].length ;
        dD = pstZones[iInd].thickness ;
        dStrike = pstZones[iInd].strike ;
        dDip = pstZones[iInd].dip ;

        dWp = dW * (1.0 + 2.0 * dBeta) / 2.0 ;
        dLp = dL * (1.0 + 2.0 * dBeta) / 2.0 ;
        dDp = dD * (1.0 + 2.0 * dBeta) / 2.0 ;

        dLM = MAX3(dWp, dLp, dDp) ;

        if ((abs (dX3-dZ) > dLM) || (abs(dX1-dX) > dLM) || (abs(dX2-dY) > dLM))
        {
            continue ;
        }

        dcStrike = cos (dStrike) ;
        dsStrike = sin (dStrike) ;
        dcDip = cos (dDip) ;
        dsDip = sin (dDip) ;

        dX2r = (dcStrike * dX) - (dsStrike * dY) ;
        dXr = (dcDip * dX2r) - (dsDip * dZ) ;
        dYr = (dsStrike * dX) + (dcStrike * dY) ;
        dZr = (dsDip * dX2r) + (dcDip * dZ) ;

        dX2r = (dcStrike * dX1) - (dsStrike * dX2) ;
        dX1s = (dcDip * dX2r) - (dsDip * dX3) ;
        dX2s = (dsStrike * dX1) + (dcStrike * dX2) ;
        dX3s = (dsDip * dX2r) + (dcDip * dX3) ;

        dDgGammaDot0 += (cuOmega((dX1s-dXr)/dD, dBeta) * cuOmega((dX2s-dYr)/dW, dBeta) *
                         cuOmega((dX3s-dZr)/dL, dBeta) * dDg) ;
    }

    return dDgGammaDot0 ;
}

__host__ __device__ void cutDot (ST_TENSOR  *pstTemp,
                                 double     *dN,
                                 double     *dT)
{
    dT[0] = pstTemp->s11 * dN[0] + pstTemp->s12 * dN[1] + pstTemp->s13 * dN[2] ;
    dT[1] = pstTemp->s12 * dN[0] + pstTemp->s22 * dN[1] + pstTemp->s23 * dN[2] ;
    dT[2] = pstTemp->s13 * dN[0] + pstTemp->s23 * dN[1] + pstTemp->s33 * dN[2] ;
}


__host__ __device__ double cuSum (double *dN,
                                  double *dT)
{
    return (dN[0] * dT[0] + dN[1] * dT[1] + dN[2] * dT[2]) ;
}

__host__ __device__ void cuMulSub (double  dTaun,
                                   double  *dT,
                                   double  *dN,
                                   double  *dTs)
{
    dTs[0] = dT[0] - (dTaun * dN[0]) ;
    dTs[1] = dT[1] - (dTaun * dN[1]) ;
    dTs[2] = dT[2] - (dTaun * dN[2]) ;
}

__host__ __device__ double mycuSinh (double dX)
{
    double dTemp ;

    if (fabs(dX) > 11.0)
    {
        dTemp = (dX > 0 ? 1.0 : -1.0) * sinh (11.0) ;
    }
    else
    {
        dTemp = sinh(dX) ;
    }

    return dTemp ;
}

/* ---------------------------------- host/device functions end -------------------------------------- */

#endif

/* EOF */
