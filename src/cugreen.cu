#include "hip/hip_runtime.h"
/*-----------------------------------------------------------------------
! Copyright 2013 Sylvain Barbot
!
! This file is part of RELAX
!
! RELAX is free software: you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation, either version 3 of the License, or
! (at your option) any later version.
!
! RELAX is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License
! along with RELAX.  If not, see <http://www.gnu.org/licenses/>.
!
! \author Sagar Masuti 
!----------------------------------------------------------------------*/


#include <stdio.h>
//#define PAPI_PROF
#include "cuinclude.h"
#include <hipfft/hipfft.h>

#ifdef USING_CUDA

//#define ENABLE_FFTW3

#ifdef ENABLE_FFTW3
extern "C"
{
	extern void __fourier_MOD_fft3 (float *data,int *sx1,int *sx2,int *sx3,double *dx1,double *dx2, double *dx3,int *direction) ;
	extern void __fourier_MOD_fft2 (float *data,int *sx1,int *sx2,double *dx1,double *dx2, int *direction) ;
}
#endif

/* --------------------------- Forward declaration -------------------------------- */

/* ------------------------------- Global Functions ------------------------------- */ 

__global__ void scaleAndSub (float *, float *, float, int, int, int) ;

__global__ void cuAddKernel (float *, float *, float *, float2 *, float2 *, float2 *, int, int, int, 
                             int, int) ;

__global__ void scale1D (float *, float *, float *, float) ;

__global__ void cuElasticResKernel (float *, float *, float *, int, int, int, double, double, double, 
                                    double, double) ;

__global__ void cerrutiKernel (double2, double2, double2, float2 *, float2 *, float2 *, int, double, 
                               double, double, double, double, double) ;

__global__ void cuSurfaceReduction(double, double, double, double, double, double, double, float,
                                   float, float, int, int, int, float, float, float) ;

__global__ void cuSurfaceKernel (double, double, double, double, double, double, double, float [], 
                                 float [], float [], int, int, int, float *, float *, float *) ;

__global__ void cuCerrutiModKernel (double, double, double, float [], float [], float [], int, int,                                     int, double, double, double, float *, float *, float *) ;

/* ------------------------------- Global Functions ------------------------------- */ 

/* ------------------------------- Device Functions ------------------------------- */ 

__host__ __device__ void cuWaveNumber (int, int, int, int, int, int, double, double, double, 
                                       double *, double*, double *) ;

__device__ void cerrutiSolution (double, double2, double2, double2, double, double, float2 *, 
                                 float2 *, float2 *, double, double, double, double, int) ;

__device__ void cuCerrutiModDevice (double, double2, double2, double2, double, double, float2 *, 
                                    float2 *, float2 *, double, double, double) ;

__device__ void surfaceCal (double, double, double, double, double, double, double, float, float, 
                            float, int, int, int, float, float, float, unsigned int, unsigned int, 
                            unsigned int, unsigned int, unsigned int) ;

/* ------------------------------- Device Functions ------------------------------- */ 

/* ------------------------------- Host Functions --------------------------------- */ 
static hipError_t scaleAll (float, float, int, int, int) ;

static int calfft (float *, float *, float *, int, int, double, double, float *, float *, float *) ;

static void OneDfft (hipfftHandle, float *, float *, float *, int, double) ;

static hipError_t doSurfaceTraction (double, double, double, int, int, int, double, double, 
                                      double) ;

static void cerruti (float *, float *, float *, int, int, int, double, double, double, double, double,                     double, double, float *, float *, float *) ;

static void cuCerrutiModified (double, double, double, double, double, double, float [], float [], 
                               float [], int, int, int, float *, float *, float *) ;

static int allocAndCopy (float *, float *, float *, int, float *, float *, float *, int, float, float, int, int,
                         int) ;

static int  inverseFFT (float *, float *, float *, int, int, int, double, double, double, float *, 
                        float *, float *) ;


/* ------------------------------- Host Functions --------------------------------- */ 
/* -------------------------------------------------------------------------------- */


/* ------------------------------- util functions --------------------------------- */

__device__ double2 comp_double_add (double2, double2) ;
__device__ double2 comp_double_mult (double2, double2) ;
__device__ double2 comp_double_sub (double2, double2) ;
__host__ __device__ double2 make_comp_double (double, double) ;
__device__ double2 comp_double_div (double2, double2) ;
__device__ float comp_float_real (float2) ;
__device__ float comp_float_img (float2) ;
__device__ float2 make_comp_float (float, float) ;
__device__ float2 comp_float_add (float2, float2) ;
__device__ float2 comp_float_sub (float2, float2) ;
__device__ float2 comp_float_mult (float2, float2) ;
__device__  float2 comp_float_div (float2, float2) ;

/* ---------------------------------------------------------------------------------*/


/* ------------------------------ Global variables --------------------------------- */
extern float    *gpV1 ;
extern float    *gpV2 ;
extern float    *gpV3 ;

extern float    *pfDevTract1 ;
extern float    *pfDevTract2 ;
extern float    *pfDevTract3 ;

extern int      ihSx1 ;
extern int      ihSx2 ;
extern int      ihSx3 ;

hipfftHandle     ghPlan ;		    /* This holds the current FFT plan */
hipfftComplex   	*gpCompData ;		/* Pointer to the device data */
hipfftHandle 	ghThreeFftPlan ;	/* Plan which is used for 3 continuous FFT's */
hipfftComplex 	*gpCompData1 ;		/* Pointer to the device data */
hipfftComplex 	*gpCompData2 ;		/* Pointer to the device data */
hipfftComplex 	*gpCompData3 ;		/* Pointer to the device data */
int 			giCurDirection ;	/* The current direction of FFT - > Forward or Inverse */
int 			giDim ;			    /* The dimensions of the current FFT -> 1 for 1D,2 for 2D and 3 for 3D */
hipStream_t  	cuStreamMem ;		/* Stream in which the memory copy should take place */
//hipStream_t  cuStreamExec ;		/* Stream in which the execution of FFT's should take place */
hipfftComplex    *gpComp2dData1 ; 
hipfftComplex 	*gpComp2dData2 ; 
hipfftComplex 	*gpComp2dData3 ;
hipfftHandle     ghTwoFftPlan ;
hipfftHandle 	ghPlanOne ; 
float 			*pfB1 ;
float 			*pfB2 ;
float 			*pfB3 ;
hipfftHandle     cuThreeFftPlan ;
hipfftHandle     cuTwoFftPlan ;
hipfftHandle     hInversePlan ;
hipfftHandle     hInvTwoPlan ;
/* -----------------------------------------------------------------------------------*/

/* ------------------------------  Main Functions ----------------------------------- */
int createPlanForFFT(int iSx1, 
                     int iSx2, 
                     int iSx3)
{
    hipfftResult     cuRet ;

    cuRet = hipfftPlan3d(&cuThreeFftPlan, iSx3, iSx2, iSx1, HIPFFT_R2C) ;
    if (HIPFFT_SUCCESS != cuRet)
    {
	    printf ("createPlanForFFT : the plan creation failed 1\n") ;
        return 1 ;
    }

    cuRet = hipfftPlan2d(&cuTwoFftPlan, iSx2, iSx1, HIPFFT_R2C) ;
    if (HIPFFT_SUCCESS != cuRet)
    {
        printf ("createPlanForFFT : the plan creation failed 2\n") ;
        hipfftDestroy (cuThreeFftPlan) ;
        return 1 ;
    }
    cuRet = hipfftPlan3d(&hInversePlan, iSx3, iSx2, iSx1, HIPFFT_C2R) ;
    if (HIPFFT_SUCCESS != cuRet)
    {
        printf ("createPlanForFFT : the plan creation failed 3\n") ;
        hipfftDestroy (cuThreeFftPlan) ;
        hipfftDestroy (cuTwoFftPlan) ;
        return 1 ;
    }

    cuRet = hipfftPlan2d(&hInvTwoPlan, iSx2, iSx1, HIPFFT_C2R) ;
    if (HIPFFT_SUCCESS != cuRet)
    {
        printf ("createPlanForFFT : the plan creation failed 4\n") ;
        hipfftDestroy (cuThreeFftPlan) ;
        hipfftDestroy (cuTwoFftPlan) ;
        hipfftDestroy (hInversePlan) ;
        return 1 ;
    }

    return 0 ;
}

int destroyPlanForFFT ()
{
    if (HIPFFT_SUCCESS != hipfftDestroy (cuThreeFftPlan))
    {
        printf ("destroyPlanForFFT: Destroying the plan failed 1 \n") ;
    } 
	
    if (HIPFFT_SUCCESS != hipfftDestroy (cuTwoFftPlan))
    {
        printf ("destroyPlanForFFT: Destroying the plan failed 2\n") ;
    }
    
    if (HIPFFT_SUCCESS != hipfftDestroy (hInversePlan))
    {
        printf ("destroyPlanForFFT: Destroying the plan failed 3\n") ;
    }

    if (HIPFFT_SUCCESS != hipfftDestroy (hInvTwoPlan))
    {
        printf ("destroyPlanForFFT: Destroying the plan failed 4 \n") ;
    }
	
    return 0 ;
}

/**
 * @breif   This function is used to create a FFT plan and if the plan creation was 
 *		    successul then allocate memory to copy the input for the FFT. 
 *
 * @param 	iSx1[in]        The size of array in x direction.
 * @param 	iSx2[in]        The size of array in y direction.
 * @param 	iSx3[in]        The size of array in z direction.
 * @param 	iDim[in]        The number of dimensions i.e., 1D 2D or 3D.
 * @param	iDirection[in]  The direction of the FFT i.e., Forward or Inverse.
 * @return 	iRet[in,out] 	Returns 1 if successful and 0 if unsuccessful.
 *		 
 */

extern "C" void initialize_cufft_ (int iSx1, 
                                   int iSx2, 
                                   int iSx3,
                                   int iDim,
                                   int iDirection,
                                   int *iRet)
{
    hipfftResult cuRet ;
    hipError_t cuError = hipSuccess;
    *iRet = 1 ;

    switch (iDim)
    {
        case E_ONE_DIMENSION:
        break ;

        case E_TWO_DIMENSION :
        {
            if (iDirection == FFT_FORWARD)
            {
                cuRet = hipfftPlan2d(&ghPlan, iSx2, iSx1, HIPFFT_R2C) ;
                if (HIPFFT_SUCCESS == cuRet)
                {
                    cuError = hipMalloc((void**)&gpCompData, 
                              sizeof(hipfftComplex) * ((iSx1/2) + 1) * iSx2) ;
                }
                else
                {
                    *iRet = 0 ;
                }
            }
            else
            {
                cuRet = hipfftPlan2d(&ghPlan, iSx2, iSx1, HIPFFT_C2R) ;
                if (HIPFFT_SUCCESS == cuRet)
                {
                    cuError = hipMalloc((void**)&gpCompData, 
                              sizeof(float) * (iSx1+2) * iSx2) ;
                }
                else
                {
                    *iRet = 0 ;
                }
            }
            if (hipSuccess != cuError)
            {
                *iRet = 0 ;
                fprintf(stderr, "Cuda error: Failed because :  %s\n",
                hipGetErrorString(cuError)) ;
            }
            else
            {
                giCurDirection = iDirection ;
                giDim = iDim ;
                *iRet = 1 ;
            }
        }
        break ;
                    
        case E_THREE_DIMENSION :
        {
#ifdef GPU_MEMORY_LOG
            size_t iFreeMem = 0 ;
            size_t iTotalMem = 0 ;
            hipMemGetInfo(&iFreeMem, &iTotalMem);  
            fprintf(stdout, "Memory avaliable: Free: %lu MB, Total: %lu MB \n", 
            iFreeMem/(1024 * 1024), iTotalMem/(1024 * 1024));
#endif
                
            if (iDirection == FFT_FORWARD)
            {
                cuRet = hipfftPlan3d(&ghPlan, iSx3, iSx2, iSx1, HIPFFT_R2C) ;
                if (HIPFFT_SUCCESS == cuRet)
                {
#ifdef GPU_MEMORY_LOG
                    hipMemGetInfo(&iFreeMem, &iTotalMem);
                    fprintf(stdout, "Memory avaliable: Free: %lu MB, Total: %lu MB \n", 
                    iFreeMem/(1024 * 1024), iTotalMem/(1024 * 1024));
                    fprintf (stdout, "Going to allocate %lu MB\n", 
                    (sizeof(hipfftComplex) * ((iSx1/2) + 1) * iSx2 * iSx3) /(1024 * 1024)) ; 
#endif
                    cuError = hipMalloc((void**)&gpCompData, 
                              sizeof(hipfftComplex) * ((iSx1/2) + 1) * iSx2 * iSx3) ;
                }
                else
                {
                    fprintf(stderr, "Plan failed\n"); 
                    *iRet = 0 ;
                }
            }
            else
            {
                cuRet = hipfftPlan3d(&ghPlan, iSx3, iSx2, iSx1, HIPFFT_C2R) ;
                if (HIPFFT_SUCCESS == cuRet)
                {
                    cuError = hipMalloc((void**)&gpCompData, sizeof(float) * (iSx1+2) * iSx2 * iSx3) ;
                }
                else
                {
                    fprintf(stderr, "Plan failed 2\n") ;
                    *iRet = 0 ;
                }	
            }
            if (cuError != hipSuccess)
            {
                *iRet = 0 ;
                fprintf(stderr, "Cuda error: Failed because :  %s\n",hipGetErrorString(cuError)) ;
            }
            else
            {
                giCurDirection = iDirection ;
                giDim = iDim ;
                *iRet = 1 ;
            }
        }
        break ;
    }
		
	return ;
}

/**
 * @breif   This function is used to execute the FFT using the plan already created.  
 *          After the execution of the FFT it also uses the scaling kernel to scale FFT output. 
 *
 * @param	fData[in,out]	The array which contains the input and we do in-place transform 
 *				            so it contains the FFT output as well.
 * @param 	fScale[in]	    This is scaling factor. 	
 * @param   iSx1[in]        The size of array in x direction.
 * @param   iSx2[in]        The size of array in y direction.
 * @param   iSx3[in]        The size of array in z direction.
 *               
 */

extern "C" void calculatefft_(float fData[],
                              float fScale,
                              int   iSx1, 
                              int   iSx2, 
                              int   iSx3) 
{
 	hipfftResult_t   cuRet ;
	hipError_t     cuError ;

    switch (giDim)
    {
        case E_ONE_DIMENSION:
        break ;
                
        case E_TWO_DIMENSION :
        {	
            if (FFT_FORWARD == giCurDirection)
            {
                cuError = hipMemcpy (gpCompData, fData, 
                          sizeof (float) * (iSx1 + 2) * iSx2, hipMemcpyHostToDevice) ;
                CHECK_CUDA_ERROR("calculatefft", EXIT_FUNCTION)

                cuRet = hipfftExecR2C(ghPlan, (hipfftReal *)gpCompData, gpCompData) ;
                if (HIPFFT_SUCCESS != cuRet)
                {
                    fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
                    return ;
                }
                if (hipDeviceSynchronize() != hipSuccess)
                {
                    fprintf(stderr, "Cuda error: Failed to synchronize\n") ;
                    return ;
                }
                else
                {
                    /*Success case : Everything went well*/
                    dim3 dimGrid(iSx2, 1, 1) ;
                    dim3 dimBlock((iSx1 + 2), 1, 1) ;
                    scaling <float> <<<dimGrid, dimBlock>>> ((float *)gpCompData, fScale, 
                                                             (iSx1 + 2), iSx2, 2) ; 
                    cuError = hipMemcpy (fData, gpCompData, 
                    sizeof(hipfftComplex) * ((iSx1/2) + 1) * iSx2, hipMemcpyDeviceToHost) ;
                    CHECK_CUDA_ERROR("calculatefft", EXIT_FUNCTION)
                }
            }
            else
            {
                cuError = hipMemcpy (gpCompData, fData, 
                          sizeof (hipfftComplex) * (iSx1/2 + 1) * iSx2, hipMemcpyHostToDevice) ;
                CHECK_CUDA_ERROR("calculatefft", EXIT_FUNCTION)
                                   
                cuRet = hipfftExecC2R(ghPlan, gpCompData, (hipfftReal *)gpCompData) ; 
                if (HIPFFT_SUCCESS != cuRet)
                {
                    fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
                    return ;
                }

                if (hipDeviceSynchronize() != hipSuccess)
                {
                    fprintf(stderr, "Cuda error: Failed to synchronize\n") ;
                    return ;
                }
                else
                {
                    /*Success case : Everything went well*/
                    dim3 dimGrid(iSx2, 1, 1) ;
                    dim3 dimBlock((iSx1 + 2), 1, 1) ;
                    scaling <float> <<<dimGrid, dimBlock>>> ((float *)gpCompData, fScale, 
                                                             (iSx1 + 2), iSx2, 2) ;
                    cuError = hipMemcpy (fData, gpCompData, 
                    sizeof(float) * (iSx1+ 2) * iSx2, hipMemcpyDeviceToHost) ;
                    CHECK_CUDA_ERROR("calculatefft", EXIT_FUNCTION)
                }
            }
        }
        break ;
                    
        case E_THREE_DIMENSION :
        {
            if (FFT_FORWARD == giCurDirection)
            {
                cuError = hipMemcpy (gpCompData, fData, 
                          sizeof (float) * (iSx1 + 2) * iSx2 * iSx3, hipMemcpyHostToDevice) ;
                CHECK_CUDA_ERROR("calculatefft", EXIT_FUNCTION)

                cuRet = hipfftExecR2C(ghPlan, (hipfftReal *)gpCompData, gpCompData) ; 
                if (HIPFFT_SUCCESS != cuRet)
                {
                    fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
                    return ;
                }

                if (hipDeviceSynchronize() != hipSuccess)
                {
                    fprintf(stderr, "Cuda error: Failed to synchronize\n") ;
                    return ;
                }
                else
                {
                    dim3 dimGrid(iSx3, iSx2, 1) ;
                    dim3 dimBlock((iSx1 + 2), 1, 1) ;
                    /*Success case : Everything went well*/
                    scaling <float> <<<dimGrid, dimBlock>>> ((float *)gpCompData, fScale, (iSx1 + 2), 
                                                             iSx2, iSx3) ;
                    cuError = hipMemcpy (fData, gpCompData, 
                    sizeof(hipfftComplex) * ((iSx1/2) + 1) * iSx2 * iSx3, hipMemcpyDeviceToHost) ;
                    CHECK_CUDA_ERROR("calculatefft", EXIT_FUNCTION)
                }
            }
            else
            {
                cuError = hipMemcpy (gpCompData, fData, 
                sizeof (hipfftComplex) * (iSx1/2 + 1) * iSx2 * iSx3, hipMemcpyHostToDevice) ;
                CHECK_CUDA_ERROR("calculatefft", EXIT_FUNCTION)

                cuRet = hipfftExecC2R(ghPlan, gpCompData, (hipfftReal *)gpCompData) ;
                if (HIPFFT_SUCCESS != cuRet)
                {
                    fprintf(stderr, "CUFFT error: ExecR2C Forward failed");
                    return ;
                }

                if (hipDeviceSynchronize() != hipSuccess)
                {
                    fprintf(stderr, "Cuda error: Failed to synchronize\n") ;
                    return ;
                }
                else
                {
                    dim3 dimGrid(iSx3, iSx2, 1) ;
                    dim3 dimBlock((iSx1 + 2), 1, 1) ;
                    /*Success case : Everything went well*/
                    scaling <float> <<<dimGrid, dimBlock>>> ((float *)gpCompData, fScale, (iSx1 + 2), 
                                                             iSx2, iSx3) ;
                    cuError = hipMemcpy (fData, gpCompData, 
                    sizeof(float) * (iSx1+ 2) * iSx2 * iSx3, hipMemcpyDeviceToHost) ;
                    CHECK_CUDA_ERROR("calculatefft", EXIT_FUNCTION)
                }
            }
        }
        break ;
    }

EXIT_FUNCTION:	
	return ;
}

/**
 * @breif   This function is used to free the memory allocated and 
 *			destroys the FFT plan created. 
 *                               
 */

extern "C" void deinitialize_cufft_ ()
{
	hipfftDestroy (ghPlan) ;
	hipFree (gpCompData) ; 

#ifdef GPU_MEMORY_LOG
    size_t iFreeMem = 0 ; 
    size_t iTotalMem = 0 ;
    hipMemGetInfo(&iFreeMem, &iTotalMem) ;
    fprintf(stdout, "deinitialize_cufft : Memory avaliable : Free: %lu MB, Total: %lu MB \n", 
    iFreeMem/(1024 * 1024), iTotalMem/(1024 * 1024)) ;
#endif 
    return ;
}


/* -------------------------------------------------------------------------------------------- */


/* ----------------------------------- Cuda Kernels ------------------------------------------- */

/**
 * @breif       This kernel function is used to scale the FFT output. 
 *
 * @param  pCompData[in,out] The array which contains the input to be scaled.
 * @param  fScale[in]      	 This is scaling factor.         
 * @param  iNx[in]        	 The size of array in x direction.
 * @param  iNy[in]        	 The size of array in y direction.
 * @param  iNz[in]        	 The size of array in z direction.
 *               
 
template <class T>
__global__ void scaling (T 	*pCompData,
			            T   fScale,
                        int iNx, 
                        int iNy, 
                        int iNz)
{
	int iX ; 
	int iY ;
	int iZ ; 
	unsigned int 	iIdx ;

 	iX = threadIdx.x ;
    iY = blockIdx.x ;
    iZ = blockIdx.y ;
	
	if ((iX < iNx) && (iY < iNy) && (iZ < iNz))
	{
        iIdx = (((iZ * iNy) + iY) * iNx) + iX ;
        pCompData[iIdx] = pCompData[iIdx] * fScale ;
	}
}*/

/**
 * @breif  This is a device function which calculates the wavenumber.
 *
 * @param  iInd1[in] Indice1.
 * @param  iInd2[in] Indice2.
 * @param  iInd3[in] Indice3.
 * @param  iSx1[in]  The size of array in x direction.
 * @param  iSx2[in]  The size of array in y direction.
 * @param  iSx3[in]  The size of array in z direction.
 * @param  dDx1[in]  Sampling size.
 * @param  dDx2[in]  Sampling size.
 * @param  dDx3[in]  Sampling size.
 * @param  *dK1[out] The wavenumber.
 * @param  *dK2[out] The wavenumber. 
 * @param  *dK3[out] The wavenumber.
 *               
 */

__device__ void cuWaveNumber (int  	  iInd1, 
                              int 	  iInd2, 
                              int	  iInd3,
                              int 	  iSx1, 
                              int 	  iSx2,
                              int 	  iSx3,
                              double  dDx1, 
                              double  dDx2, 
                              double  dDx3, 
                              double  *dK1, 
                              double  *dK2, 
                              double  *dK3)
{
    if (iInd3 < ((iSx3/2)+1))
    {
        *dK3 = ((double)iInd3) / (iSx3 * dDx3) ;
    }
    else
    {
        *dK3 = -((double) (iSx3 - iInd3)) / (iSx3 * dDx3) ;
    } 

    if (iInd2 < ((iSx2 /2) + 1))
    {
        *dK2 = ((double)iInd2) / (iSx2 * dDx2) ;
    }
    else
    {
        *dK2 = -((double) (iSx2 - iInd2)) / (iSx2 * dDx2) ;
    }

    *dK1 = ((double) iInd1) / (iSx1 * dDx1) ; 
	return ;
}

/**
 * @breif  This function can be called from the fortran interface, it copies the 
 *		   data onto the GPU and calculates the elastic response. 
 *
 * @param  fInData1[in,out] Pointer to the data array
 * @param  fInData2[in,out] Pointer to the data array
 * @param  fInData3[in,out] Pointer to the data array.
 * @param  iSx1[in]         The size of array in x direction.
 * @param  iSx2[in]         The size of array in y direction.
 * @param  iSx3[in]         The size of array in z direction.
 * @param  dDx1[in]         Sampling size.
 * @param  dDx2[in]         Sampling size.
 * @param  dDx3[in]         Sampling size.
 *               
 */

extern "C" void cuelasticresp_ (float   fInData1[],
                                float   fInData2[],
                                float   fInData3[],
                                int     iSx1,
                                int     iSx2,
                                int     iSx3,
                                double  dDx1,
                                double  dDx2,
                                double  dDx3,
                                double  dR1,
                                double  dRatio2)
{
	float 		 *fData1 ;
	float 		 *fData2 ;
	float 		 *fData3 ;
	hipError_t  cuError ; 
	unsigned int iArrSize = 0 ; 

	/*Initialize the array size*/
    iArrSize = sizeof (float) * (iSx1 + 2) * iSx2 * iSx3 ; 

#ifdef PRINT_DEBUG_INFO
    fprintf (stdout, "Entered the cuelasticresp function \n") ; 
#endif

#ifdef GPU_MEMORY_LOG
    size_t iFreeMem = 0 ;
    size_t iTotalMem = 0 ;
    hipMemGetInfo(&iFreeMem, &iTotalMem) ;
    fprintf(stdout, "Memory avaliable: Free: %lu MB, Total: %lu MB \n", iFreeMem/(1024 * 1024), iTotalMem/(1024 * 1024));

    unsigned int t ;
    t = sizeof (float) * (iSx1 + 2) * iSx2 * iSx3 ;
    fprintf (stdout, "Going to allocate:  %u MB \n ",t / (1024 * 1024)) ;
#endif	

    dim3 dimGrid(iSx3, iSx2, 1) ;
    dim3 dimBlock((iSx1/2 + 1), 1, 1) ;	
	
/*Test code */
/*   	float           *fIn ;
        float           sumBef = 0 ;
        float           sumAft = 0 ;
	int i, j, k = 0 ; 
	
	fIn = (float *)malloc (sizeof (float) * (iSx1 + 2) * iSx2 * iSx3) ;
	if (NULL != fIn)
	{ 	
		for (i = 0; i < (iSx3); i++)
		{
			for (j = 0 ; j < (iSx2); j++)
			{
				for (k = 0; k < (iSx1 + 2); k++)
				{
					fIn[(i * iSx2 + j) * (iSx1 + 2)  + k] = 1 ;
					sumBef += 1 ;
				}	
			}
		}
	}
	else 
	{
		printf ("Memory allocation failed\n") ;
	}
*/

    cuError = hipMalloc ((void**)&fData1, iArrSize) ;
    if (hipSuccess != cuError)
    {
        printf ("failed in memory allocation. 1 : %s\n", hipGetErrorString(cuError)) ;
        return ;
    }

    cuError = hipMalloc ((void**)&fData2, iArrSize) ;
    if (hipSuccess != cuError)
    {
        printf ("failed in memory allocation. 2 :  %s\n", hipGetErrorString (cuError)) ;
        hipFree (fData1) ;
        return ; 
    }

    cuError = hipMalloc ((void**)&fData3, iArrSize) ;
    if (hipSuccess != cuError)
    {
        printf ("failed in memory allocation. 3 : %s\n", hipGetErrorString (cuError)) ;
        hipFree (fData1) ;
        hipFree (fData2) ;
        return ;
    }
	
    cuError = hipMemcpy (fData1, fInData1, iArrSize, hipMemcpyHostToDevice) ;
	
	/* -- Test code starts -- */
	/* ---- cuError = hipMemcpy (fData1, fIn, iArrSize, hipMemcpyHostToDevice) ; ---  */
	/* -- Test code ends -- */
	
    CHECK_CUDA_ERROR ("cudaMemcpy1 of cuelasticresp_", EXIT_WITH_FREE)
	/* -- Test code starts -- */
	/* -- memset (fIn, 0, iArrSize) ; -- */
	/* -- Test code ends -- */

    cuError = hipMemcpy (fData2, fInData2, iArrSize, hipMemcpyHostToDevice) ;
    CHECK_CUDA_ERROR ("cudaMemcpy2 of cuelasticresp_", EXIT_WITH_FREE)

    cuError = hipMemcpy (fData3, fInData3, iArrSize, hipMemcpyHostToDevice) ;
    CHECK_CUDA_ERROR ("cudaMemcpy3 of cuelasticresp_", EXIT_WITH_FREE)


#ifdef PRINT_DEBUG_INFO 
    printf ("Memcpy done\n") ;
#endif
 
    cuElasticResKernel <<<dimGrid, dimBlock>>> (fData1, fData2, fData3, (iSx1/2 + 1), iSx2, iSx3, 
                                                dDx1, dDx2, dDx3, dR1, dRatio2) ;
    cuError = hipGetLastError () ;
    CHECK_CUDA_ERROR ("cuElasticResKernel of cuelasticresp_", EXIT_WITH_FREE)

#ifdef PRINT_DEBUG_INFO
    printf ("Launched the kernel\n") ;	
#endif 
        
    if (hipDeviceSynchronize() != hipSuccess)
    {
        fprintf(stderr, "Cuda error: Failed to synchronize\n") ;
        goto EXIT_WITH_FREE ;		
    }
	
    cuError = hipMemcpy (fInData1, fData1, iArrSize, hipMemcpyDeviceToHost) ;
    CHECK_CUDA_ERROR ("cudaMemcpy4 of cuelasticresp_", EXIT_WITH_FREE)
	
	/* -- Test code starts -- */
	/* -- cuError = hipMemcpy (fIn, fData1, iArrSize, hipMemcpyDeviceToHost) ; -- */
	/* -- Test code ends -- */

	/*for (i = 0; i < (iSx3); i++)
        {
        	for (j = 0 ; j < (iSx2); j++)
                {
                	for (k = 0; k < (iSx1 + 2); k++)
                        {
                        	sumAft += fIn[i * (iSx1 + 2)  * iSx2 + j * (iSx1 + 2) + k];
                        }
                }       
        }

	if (2 * sumBef != sumAft) 
	{
		printf ("Something is wrong, The value before is %f and after is %f\n", sumBef, sumAft) ;
	}
	else 
	{
		printf ("They are equal %f and %f", sumBef, sumAft) ; 
	}
	free (fIn) ;
*/

    cuError = hipMemcpy (fInData2, fData2, iArrSize, hipMemcpyDeviceToHost) ;
    CHECK_CUDA_ERROR ("cudaMemcpy5 of cuelasticresp_", EXIT_WITH_FREE)
	
    cuError = hipMemcpy (fInData3, fData3, iArrSize, hipMemcpyDeviceToHost) ;
    CHECK_CUDA_ERROR ("cudaMemcpy6 of cuelasticresp_", EXIT_WITH_FREE)
        

EXIT_WITH_FREE :

    hipFree (fData1) ;
    hipFree (fData2) ;
    hipFree (fData3) ;

    return ;
}

/**
 * @breif  This is the kernel which calculates the elastic response. 
 *
 * @param  fInData1[in,out] Pointer to equivalent body-forces in the Fourier domain
 * @param  fInData2[in,out] Pointer to equivalent body-forces in the Fourier domain
 * @param  fInData3[in,out] Pointer to equivalent body-forces in the Fourier domain
 * @param  iSx1[in]         The size of array in x direction.
 * @param  iSx2[in]         The size of array in y direction.
 * @param  iSx3[in]         The size of array in z direction.
 * @param  dDx1[in]         Sampling size.
 * @param  dDx2[in]         Sampling size.
 * @param  dDx3[in]         Sampling size.
 *               
 */
__global__ void cuElasticResKernel (float  *fData1, 
                                    float  *fData2, 
                                    float  *fData3, 
                                    int    iSx1,
                                    int    iSx2,
                                    int    iSx3,
                                    double dDx1,
                                    double dDx2,
                                    double dDx3,
                                    double dR1, 
                                    double dRatio2)
{
	/* Variable declaration */
    double 	     dR2 ; 
    double 	     dDenom ; 
    double 	     dK1 ;
    double 	     dK2 ;
    double 	     dK3 ;

    double2      cuC1 ;
    double2      cuC2 ;
    double2      cuC3 ;
    double2      cuBuf1 ; 
    double2      cuBuf2 ;
    double2      cuBuf3 ;
    unsigned int iIdx1 ; 
    unsigned int iIdx2 ; 
    unsigned int iIdx3 ; 
    unsigned int iIdx ;	
#ifdef SHARED_MEMORY_IMPL_ELASTIC	
    unsigned int i ; 
    unsigned int index ; 
    unsigned int index2 ; 

    __shared__ float pfshBuf1[258] ;
    __shared__ float pfshBuf2[258] ;
    __shared__ float pfshBuf3[258] ;
#else
    float2       *pC1 ;
    float2       *pC2 ;
    float2       *pC3 ;
#endif

	/* Get the proper index */
    iIdx1 = threadIdx.x ;
    iIdx2 = blockIdx.y ;
    iIdx3 = blockIdx.x ;

#ifdef PRINT_DEBUG_INFO
    printf ("Entered the kernel\n") ;
#endif
	
	/*Make sure the indices are within the range */
    if ((iIdx1 < iSx1) && (iIdx2 < iSx2) && (iIdx3 < iSx3))
    {
#ifdef SHARED_MEMORY_IMPL_ELASTIC
        if (iIdx1 < (2 * warpSize))
        {
            for (i = 0; i < (iSx1/warpSize); i++)
            {
                index = (iIdx1 * (iSx1 / warpSize)) + i ;
                index2 = (((iIdx3 * iSx2) + iIdx2) * 2 * iSx1) + index ;
                pfshBuf1[index] = fData1[index2] ;
                pfshBuf2[index] = fData2[index2] ;
                pfshBuf3[index] = fData3[index2] ;
            }
        }
        if (iIdx1 == (iSx1 - 1))
        {
            index = 2 * iIdx1  ;
            index2 = (((iIdx3 * iSx2) + iIdx2) * 2 * iSx1) + index ;
            pfshBuf1[index] = fData1[index2] ;
            pfshBuf1[index+1] = fData1[index2+1] ;
            pfshBuf2[index] = fData2[index2] ;
            pfshBuf2[index+1] = fData2[index2+1] ;
            pfshBuf3[index] = fData3[index2] ;
            pfshBuf3[index+1] = fData3[index2+1] ;
        }
        __syncthreads () ;	
#endif
 
#ifdef PRINT_DEBUG_INFO
        printf ("The values of indexes are iIdx1 : %u, iIdx2 : %u, iIdx3 : %u\n", iIdx1, iIdx2, iIdx3) ;
#endif 
        cuWaveNumber (iIdx1, iIdx2, iIdx3, (2 * iSx1) - 2, iSx2, iSx3, dDx1, dDx2, dDx3, &dK1, &dK2, &dK3) ;

        dR2 = dK1 * dK1 + dK2 * dK2 + dK3 * dK3 ; 
        dDenom = (dR1 / (dR2 * dR2)) ;
		
        iIdx = (((iIdx3 * iSx2) + iIdx2) * 2 * iSx1) + 2 * iIdx1 ;

#ifdef PRINT_DEBUG_INFO		
        printf ("The Idx is %u\n", iIdx) ; 	
#endif
/*		float temp = 0 ; 
		temp = fData1[iIdx] ;
		fData1[iIdx] = 2 *  temp;
		temp = fData1[iIdx + 1] ;
		fData1[iIdx + 1] = 2 *  temp;
*/		
		
/*
		//Change for 64 bit access pattern. Need to check if 128 bit is possible ?
		cuC1 = make_comp_double ((double) fData1[iIdx], (double) fData1[iIdx + 1]) ;
        cuC2 = make_comp_double ((double) fData2[iIdx], (double) fData2[iIdx + 1]) ;
        cuC3 = make_comp_double ((double) fData3[iIdx], (double) fData3[iIdx + 1]) ;
*/ 

#ifdef SHARED_MEMORY_IMPL_ELASTIC
        iIdx = 2 * iIdx1 ;
        cuC1 = make_comp_double ((double) pfshBuf1[iIdx], (double) pfshBuf1[iIdx+1]) ;
        cuC2 = make_comp_double ((double) pfshBuf2[iIdx], (double) pfshBuf2[iIdx+1]) ;
        cuC3 = make_comp_double ((double) pfshBuf3[iIdx], (double) pfshBuf3[iIdx+1]) ;
#else
        pC1 = (float2 *) &fData1[iIdx] ;
        pC2 = (float2 *) &fData2[iIdx] ;
        pC3 = (float2 *) &fData3[iIdx] ;

        cuC1 = make_comp_double ((double) ((*pC1).x), (double) ((*pC1).y)) ;
        cuC2 = make_comp_double ((double) ((*pC2).x), (double) ((*pC2).y)) ;
        cuC3 = make_comp_double ((double) ((*pC3).x), (double) ((*pC3).y)) ;
#endif		
		cuBuf1 = comp_double_mult (comp_double_sub(comp_double_mult
                 (make_comp_double(dK2 * dK2 + dK3 * dK3 + dRatio2 * dR2, 0),  cuC1) ,
                 (comp_double_mult(make_comp_double(dK1,0), 
                 comp_double_add(comp_double_mult(make_comp_double(dK2,0),  cuC2), 
                 comp_double_mult (make_comp_double(dK3,0), cuC3))))), (make_comp_double (dDenom, 0)));

        cuBuf2 = comp_double_mult (comp_double_sub(comp_double_mult	
                 (make_comp_double(dK1 * dK1 + dK3 * dK3 + dRatio2 * dR2, 0),  cuC2) ,
                 (comp_double_mult(make_comp_double(dK2,0), 
                 comp_double_add(comp_double_mult(make_comp_double(dK1,0),  cuC1), 
                 comp_double_mult (make_comp_double(dK3,0), cuC3))))), (make_comp_double(dDenom, 0)));

        cuBuf3 = comp_double_mult (comp_double_sub(comp_double_mult
                 (make_comp_double(dK1 * dK1 + dK2 * dK2 + dRatio2 * dR2, 0),  cuC3) ,
                 (comp_double_mult(make_comp_double(dK3,0), 
                 comp_double_add(comp_double_mult(make_comp_double(dK1,0),  cuC1), 
                 comp_double_mult (make_comp_double(dK2,0), cuC2))))), (make_comp_double (dDenom,0))) ;

#ifdef SHARED_MEMORY_IMPL_ELASTIC
        pfshBuf1[iIdx] = (float) cuBuf1.x ;
        pfshBuf1[iIdx+1] = (float) cuBuf1.y ;
		
        pfshBuf2[iIdx] = (float) cuBuf2.x ;
        pfshBuf2[iIdx+1] = (float) cuBuf2.y ;

        pfshBuf3[iIdx] = (float) cuBuf3.x ;
        pfshBuf3[iIdx+1] = (float) cuBuf3.y ;

        __syncthreads () ;

        if (iIdx1 < (2 * warpSize)) 
        {
            for (i = 0; i < (iSx1/warpSize); i++)
            {
                index = (iIdx1 * (iSx1 / warpSize)) + i ;
                index2 = (((iIdx3 * iSx2) + iIdx2) * 2 * iSx1) + index ;
                fData1[index2] = pfshBuf1 [index];
                fData2[index2] = pfshBuf2 [index] ;
                fData3[index2] = pfshBuf3 [index] ;
            }
        }
        if (iIdx1 == (iSx1 - 1))
        {
            index = 2 * iIdx1  ;
            index2 = (((iIdx3 * iSx2) + iIdx2) * 2 * iSx1) + index ;
            fData1[index2] = pfshBuf1 [index];
            fData1[index2+1] = pfshBuf1 [index+1] ;
            fData2[index2] = pfshBuf2 [index] ;
            fData2[index2+1] = pfshBuf2 [index+1] ;
            fData3[index2] = pfshBuf3 [index] ;
            fData3[index2+1] = pfshBuf3 [index+1] ;		
        }	
#else
        (*pC1).x = (float) cuBuf1.x ;
        (*pC1).y = (float) cuBuf1.y ;

        (*pC2).x = (float) cuBuf2.x ;
        (*pC2).y = (float) cuBuf2.y ;

        (*pC3).x = (float) cuBuf3.x ;
        (*pC3).y = (float) cuBuf3.y ;
	
#endif
		/*fData1[iIdx] = (float) cuBuf1.x ;
		fData1[iIdx + 1] = (float) cuBuf1.y ;
 
		fData2[iIdx] = (float) cuBuf2.x ;
        	fData2[iIdx + 1] = (float) cuBuf2.y ;

		fData3[iIdx] = (float) cuBuf3.x ;
        	fData3[iIdx + 1] = (float) cuBuf3.y ; */
    }	
}

/* ------------------------------- Utility functions ------------------------------------------- */

__device__ double2 comp_double_mult(double2 ab, double2 cd)
{
    return make_comp_double (ab.x * cd.x - ab.y * cd.y, ab.x * cd.y + ab.y * cd.x);
}

__device__ double2 comp_double_add(double2 a, double2 b)
{
    return make_comp_double (a.x + b.x, a.y + b.y);
}

__device__ double2 comp_double_sub(double2 a, double2 b)
{
    return make_comp_double (a.x - b.x, a.y - b.y);
}


__device__ double2 make_comp_double (double a, double b)
{
    double2 c ;
    c.x = a  ;
    c.y = b ;
    return c ;
}

__device__ double2 comp_double_div (double2 x, double2 y)
{
    double2 quot;
    double s = (fabs(y.x)) + (fabs(y.y));
    double oos = 1.0 / s;
    double ars = x.x * oos;
    double ais = x.y * oos;
    double brs = y.x * oos;
    double bis = y.y * oos;
    s = (brs * brs) + (bis * bis);
    oos = 1.0 / s;
    quot = make_comp_double (((ars * brs) + (ais * bis)) * oos,
                                 ((ais * brs) - (ars * bis)) * oos);
    return quot;
}

__device__ float comp_float_real (float2 x)
{
    return x.x;
}

__device__ float comp_float_img (float2 x)
{
    return x.y;
}

__device__ float2 make_comp_float (float r, float i)
{
    float2 res;
    res.x = r;
    res.y = i;
    return res;
}

__device__ float2 comp_float_add (float2 x, float2 y)
{
    return make_comp_float (comp_float_real(x) + comp_float_real(y),
                            comp_float_img(x) + comp_float_img(y));
}

__device__ float2 comp_float_sub (float2 x, float2 y)
{
    return make_comp_float (comp_float_real(x) - comp_float_real(y),
                            comp_float_img(x) - comp_float_img(y));
}

__device__ float2 comp_float_mult (float2 x, float2 y)
{
    float2 prod;
    prod = make_comp_float  ((comp_float_real(x) * comp_float_real(y)) -
                             (comp_float_img(x) * comp_float_img(y)),
                             (comp_float_real(x) * comp_float_img(y)) +
                             (comp_float_img(x) * comp_float_real(y)));
    return prod;
}

__device__  float2 comp_float_div (float2 x, float2 y)
{
    float2 quot;
    float s = fabsf(comp_float_real(y)) + fabsf(comp_float_img(y));
    float oos = 1.0f / s;
    float ars = comp_float_real(x) * oos;
    float ais = comp_float_img(x) * oos;
    float brs = comp_float_real(y) * oos;
    float bis = comp_float_img(y) * oos;
    s = (brs * brs) + (bis * bis);
    oos = 1.0f / s;
    quot = make_comp_float (((ars * brs) + (ais * bis)) * oos,
                                ((ais * brs) - (ars * bis)) * oos);
    return quot;
}


/* -------------------------------------------------------------------------------------------------- */



/**
 * @breif  This function allocates the device memory to hold the data
 *		   and copies the host data into it. 
 *
 * @param  fData1[in] Pointer to equivalent body-forces in the Fourier domain
 * @param  fData2[in] Pointer to equivalent body-forces in the Fourier domain
 * @param  fData3[in] Pointer to equivalent body-forces in the Fourier domain
 * @param  iSize[in]  Number of bytes to be allocated.
 *               
 */

static int allocAndCopy (float  *fData1,
                         float  *fData2,
                         float  *fData3,
                         int    iSize,
                         float  *fData4,
                         float  *fData5,
                         float  *fData6,
                         int    iSize2,
                         float  fScale,
                         float  fScale2,
                         int    iSx1,
                         int    iSx2,
                         int    iSx3)
{
	hipfftResult cuRet = HIPFFT_SUCCESS;
	hipError_t cuError = hipSuccess;

    dim3  dimGrid(iSx3, iSx2, 1) ;
    dim3  dimBlock((iSx1 + 2), 1, 1) ;

    dim3  dimGrid1(iSx2, 1, 1) ;
    dim3  dimBlock1((iSx1 + 2), 1, 1) ;

//	hipfftHandle     cuThreeFftPlan ;
//        hipfftHandle     cuTwoFftPlan ;
//	hipStream_t 	cuStreamFFT ;	

/*	hipStreamCreate (&cuStreamFFT) ;
	
	cuRet = hipfftPlan3d(&cuThreeFftPlan, iSx3, iSx2, iSx1, HIPFFT_R2C) ;
        if (HIPFFT_SUCCESS != cuRet)
        {
                return 0 ;
        }

        cuRet = hipfftPlan2d(&cuTwoFftPlan, iSx2, iSx1, HIPFFT_R2C) ;
        if (HIPFFT_SUCCESS != cuRet)
        {
                hipfftDestroy (cuThreeFftPlan) ;
                return 0 ;
        }

	cuRet = hipfftSetStream (cuThreeFftPlan, cuStreamFFT) ;
        if (HIPFFT_SUCCESS != cuRet)
        {
		hipfftDestroy (cuThreeFftPlan) ;
	        hipfftDestroy (cuTwoFftPlan) ;	
                return 0 ;
        }
*/

	// DO NOT DELETE THIS COMMENT
	// Strange behaviour : When we create the plan here we get wrong results.
	/*cuRet = hipfftPlan3d(&cuThreeFftPlan, iSx3, iSx2, iSx1, HIPFFT_R2C) ;
        if (HIPFFT_SUCCESS != cuRet)
        {
		goto FREE_ALL ;
        }

        cuRet = hipfftPlan2d(&cuTwoFftPlan, iSx2, iSx1, HIPFFT_R2C) ;
        if (HIPFFT_SUCCESS != cuRet)
        {
		hipfftDestroy (cuThreeFftPlan) ;
                goto FREE_ALL ;
        }*/

    cuRet = hipfftExecR2C(cuThreeFftPlan, (hipfftReal *)gpCompData1, gpCompData1) ;
    if (HIPFFT_SUCCESS != cuRet)
    {
        printf("CUFFT error: ExecR2C Forward failed 1");
        goto FREE_ALL ;
    }

    cuRet = hipfftExecR2C(cuThreeFftPlan, (hipfftReal *)gpCompData2, gpCompData2) ;
    if (HIPFFT_SUCCESS != cuRet)
    {
        printf("CUFFT error: ExecR2C Forward failed 2");
        goto FREE_ALL ;
    }

	// Not to remove the commented code below : 
	// Just to remember that this is already tried and doesnt improve the performance.
/*	scaling <float> <<<dimGrid, dimBlock>>> ((float *)gpCompData1, fScale, (iSx1 + 2), iSx2, iSx3) ;
	cuError = hipGetLastError () ;	
	CHECK_CUDA_ERROR ("AllocAndCopy : scale1", FREE_ALL)
*/
        
    cuRet = hipfftExecR2C(cuThreeFftPlan, (hipfftReal *)gpCompData3, gpCompData3) ;
    if (HIPFFT_SUCCESS != cuRet)
    {
        printf("CUFFT error: ExecR2C Forward failed 3");
        goto FREE_ALL ;
    }
	
    cuRet = hipfftExecR2C(cuTwoFftPlan, (hipfftReal *)gpComp2dData1, gpComp2dData1) ;
    if (HIPFFT_SUCCESS != cuRet)
    {
        printf("CUFFT error: ExecR2C Forward failed 4");
        goto FREE_ALL ;
    }

    cuRet = hipfftExecR2C(cuTwoFftPlan, (hipfftReal *)gpComp2dData2, gpComp2dData2) ;
    if (HIPFFT_SUCCESS != cuRet)
    {
        printf("CUFFT error: ExecR2C Forward failed 5");
        goto FREE_ALL ;
    }

    cuRet = hipfftExecR2C(cuTwoFftPlan, (hipfftReal *)gpComp2dData3, gpComp2dData3) ;
    if (HIPFFT_SUCCESS != cuRet)
    {
        printf("CUFFT error: ExecR2C Forward failed 6");
        goto FREE_ALL;
    }
    if (hipSuccess != hipDeviceSynchronize ())
    {
        printf ("Sync failed \n") ;
    }
    cuError = scaleAll (fScale, fScale2, iSx1, iSx2, iSx3) ;
    CHECK_CUDA_ERROR ("scaleAll of allocAndCopy", FREE_ALL)
    if (hipSuccess != hipDeviceSynchronize ())
    {
        printf ("Sync failed \n") ;
    }

    //hipfftDestroy (cuThreeFftPlan) ;
    //hipfftDestroy (cuTwoFftPlan) ;
	
    return 1 ;

FREE_ALL: 
    hipFree (gpComp2dData1) ;
    hipFree (gpComp2dData2) ;
    hipFree (gpComp2dData3) ;
    hipFree (gpCompData1) ;
    hipFree (gpCompData2) ;
    hipFree (gpCompData3) ;
	
    return 0 ;
}


hipError_t memcpyUsingStreams (float 		   *fDest,
                                float 		   *fSrc,
                                int 		   iBytes,
                                hipMemcpyKind eDirection, 
                                hipStream_t   *pCuStream) 
{
    int 		iIndex = 0 ;
    hipError_t	cuError = hipSuccess;
    int 		iOffset = 0 ;

    iOffset = (iBytes / NUM_STREAMS) ;

    /*Creating streams if not present */
    if (NULL == pCuStream)
    {
        // Need to destroy the streams at the end of fftnelastic function.
        pCuStream = (hipStream_t *) malloc(NUM_STREAMS * sizeof(hipStream_t));
        for (iIndex = 0 ; iIndex < NUM_STREAMS; iIndex++)
        {
            cuError = hipStreamCreate (&pCuStream[iIndex]) ;
        }
    }	

    if (cuError != hipSuccess)
    {
        cuError = hipMemcpy (fDest, fSrc, iBytes, eDirection) ; 
    }
    else
    {
        for (iIndex = 0 ; iIndex < NUM_STREAMS; iIndex++)
        {
            iOffset = iIndex * iOffset ;
            cuError = hipMemcpyAsync (fDest +  iOffset , fSrc + iOffset, iBytes / NUM_STREAMS , eDirection, pCuStream[iIndex]) ;
        }
    }

    if (NULL != pCuStream)
    {	
        for (iIndex = 0 ; iIndex < NUM_STREAMS; iIndex++)
        {
            cuError = hipStreamDestroy (pCuStream[iIndex]) ;
        }
        free (pCuStream) ;
    }
	
    return cuError ;
}

/**
 * @breif	This function executes the scaling kernel for all the data.
 *  
 */

static hipError_t scaleAll (float 	fScale,
                           float 	fScale2,
                           int 		iSx1,
                           int 		iSx2,
                           int 		iSx3)
{
    hipError_t cuError = hipSuccess; 
    dim3        dimGrid(iSx3, iSx2, 1) ;
    dim3        dimBlock((iSx1 + 2), 1, 1) ;
    dim3        dimGrid1(1, iSx2, 1) ;
    dim3        dimBlock1((iSx1 + 2), 1, 1) ;


    scaling <float> <<<dimGrid, dimBlock>>> ((float *)gpCompData1, fScale, (iSx1 + 2), iSx2, iSx3) ;
    CHECK_ERROR("scaleAll") 

    scaling <float> <<<dimGrid, dimBlock>>> ((float *)gpCompData2, fScale, (iSx1 + 2), iSx2, iSx3) ;
    CHECK_ERROR("scaleAll") 

    scaling <float> <<<dimGrid, dimBlock>>> ((float *)gpCompData3, fScale, (iSx1 + 2), iSx2, iSx3) ;
    CHECK_ERROR("scaleAll") 

    scaling <float> <<<dimGrid1, dimBlock1>>> ((float *)gpComp2dData1, fScale2, (iSx1 + 2), iSx2, 2) ;
    CHECK_ERROR("scaleAll")

    scaling <float> <<<dimGrid1, dimBlock1>>> ((float *)gpComp2dData2, fScale2, (iSx1 + 2), iSx2, 2) ;
    CHECK_ERROR("scaleAll")

    scaling <float> <<<dimGrid1, dimBlock1>>> ((float *)gpComp2dData3, fScale2, (iSx1 + 2), iSx2, 2) ;
    CHECK_ERROR("scaleAll")

    return cuError ;
}



void copyfftmemory (float *fData1,
                    float *fData2,
                    float *fData3,
                    float *fData4,
                    float *fData5,
                    float *fData6,
                    int   iSize,
                    int   iSize2,
                    int   iDirection) 
{
#ifdef ENABLE_FFTW3
hipError_t cuError = hipSuccess ;

    if (1 == iDirection)
    {
        cuError = hipMemcpy (gpV1, fData1, iSize, hipMemcpyHostToDevice) ;
        CHECK_CUDA_ERROR("calfft : memcpy error 1\n", COPY_EXIT)

        cuError = hipMemcpy (gpV2, fData2, iSize, hipMemcpyHostToDevice) ;
        CHECK_CUDA_ERROR("calfft : memcpy error 1\n", COPY_EXIT)

        cuError = hipMemcpy (gpV3, fData3, iSize, hipMemcpyHostToDevice) ;
        CHECK_CUDA_ERROR("calfft : memcpy error 1\n", COPY_EXIT)

        cuError = hipMemcpy (pfDevTract1, fData4, iSize2, hipMemcpyHostToDevice) ;
        CHECK_CUDA_ERROR("calfft : memcpy error 1\n", COPY_EXIT)

        cuError = hipMemcpy (pfDevTract2, fData5, iSize2, hipMemcpyHostToDevice) ;
        CHECK_CUDA_ERROR("calfft : memcpy error 1\n", COPY_EXIT)

        cuError = hipMemcpy (pfDevTract3, fData6, iSize2, hipMemcpyHostToDevice) ;
        CHECK_CUDA_ERROR("calfft : memcpy error 1\n", COPY_EXIT)

    }
    //	else
    if (0 == iDirection)
    {

        cuError = hipMemcpy (fData1, gpV1, iSize, hipMemcpyDeviceToHost) ;
        CHECK_CUDA_ERROR("calfft : memcpy error 1\n", COPY_EXIT)

        cuError = hipMemcpy (fData2, gpV2, iSize, hipMemcpyDeviceToHost) ;
        CHECK_CUDA_ERROR("calfft : memcpy error 2\n", COPY_EXIT )

        cuError = hipMemcpy (fData3, gpV3, iSize, hipMemcpyDeviceToHost) ;
        CHECK_CUDA_ERROR("calfft : memcpy error 3\n", COPY_EXIT)

        cuError = hipMemcpy (fData4, pfDevTract1, iSize2, hipMemcpyDeviceToHost) ;
        CHECK_CUDA_ERROR("calfft : memcpy error 1\n", COPY_EXIT)

        cuError = hipMemcpy (fData5, pfDevTract2, iSize2, hipMemcpyDeviceToHost) ;
        CHECK_CUDA_ERROR("calfft : memcpy error 1\n", COPY_EXIT)
                        
        cuError = hipMemcpy (fData6, pfDevTract3, iSize2, hipMemcpyDeviceToHost) ;
        CHECK_CUDA_ERROR("calfft : memcpy error 1\n", COPY_EXIT)

    }
return ;

COPY_EXIT: 
    printf ("Error\n") ;
#endif
}
/**
 * @breif       This function executes fft and elastic response in asynchronous manner with different streams.
 *  
 */


extern "C" void calfftnelasic_ (float   *fData1,
                                float   *fData2,
                                float   *fData3,
                                float   *fData4,
                                float   *fData5,
                                float   *fData6,
                                int     iSx1,
                                int     iSx2,
                                int     iSx3,
                                double  dDx1,
                                double  dDx2,
                                double  dDx3,
                                double  dLambda,
                                double  dMu,
                                double  dGamma, 
                                int     *iRet)
{
    hipError_t	cuError ;
    size_t 		iFreeMem = 0 ;
    size_t 		iTotalMem = 0 ; 
    //	size_t 		iReqSize = 0 ;
    size_t	 	iSize = 0 ;
    size_t	 	iSize2 = 0 ;
    dim3 		dimGrid(ihSx3, ihSx2, 1) ;
    dim3 		dimBlock((ihSx1 + 2), 1, 1) ;
    dim3 		dimGrid1(ihSx3, ihSx2, 1) ;
    dim3 		dimBlock1(((ihSx1/2)+1), 1, 1) ;
    float 		fScale ;
    float 		fScale2 ;
    double 		dRatio1 ;
    double 		dRatio2 ;
#ifdef PAPI_PROF	
    char   		cTimerName[17] = "FFT             " ;
#endif

    *iRet = 1 ;
        
    hipMemGetInfo(&iFreeMem, &iTotalMem) ;
#ifdef PAPI_PROF	
    papistartprofiling_(cTimerName) ;
#endif
    iSx1=ihSx1 ;
    iSx2=ihSx2 ;
    iSx3=ihSx3 ;

    /* Lets check the available memory before we allocate */
    iSize = (sizeof (float) * (iSx1 + 2) * (iSx2) * (iSx3)) ;
    iSize2 = (sizeof (float) * (iSx1 + 2) * (iSx2)) ;

//	copyfftmemory (fData1, fData2, fData3, fData4, fData5, fData6, iSize, iSize2, 1) ;	
//	iReqSize = iSize * 3 + (3 * iSize2) ;
	
//        printf ("Free : %lu\n", iFreeMem) ;
       /* if (iFreeMem < iReqSize)
        {
                printf ("Free : %lu, Required: %lu\n", iFreeMem, iReqSize) ;
                return ;
        }*/
    gpCompData1 = (hipfftComplex *) gpV1 ;
    gpCompData2 = (hipfftComplex *) gpV2 ;
    gpCompData3 = (hipfftComplex *) gpV3 ;

    gpComp2dData1 = (hipfftComplex *) pfDevTract1;
    gpComp2dData2 = (hipfftComplex *) pfDevTract2;
    gpComp2dData3 = (hipfftComplex *) pfDevTract3;

    fScale = (float) (dDx1 * dDx2 * dDx3) ;
    fScale2 = (float) (dDx1 * dDx2) ;

#ifndef ENABLE_FFTW3
	/* Allocation copying and fft */
    *iRet = allocAndCopy (fData1, fData2, fData3, iSize, fData4, fData5, fData6, 
                          iSize2, fScale, fScale2, iSx1, iSx2, iSx3) ;
    if (0 == *iRet)
    {
        printf ("Memory allocation failure 1\n") ;
        return ;        
    }
#endif

#ifdef ENABLE_FFTW3
    copyfftmemory (fData1, fData2, fData3, fData4, fData5, fData6, iSize, iSize2, 0) ;
    int iDirection ; 

#ifdef FFTW3
    iDirection = -1 ;
    __fourier_MOD_fft3 (fData1, &iSx1, &iSx2, &iSx3, &dDx1, &dDx2, &dDx3, &iDirection) ;
    __fourier_MOD_fft3 (fData2, &iSx1, &iSx2, &iSx3, &dDx1, &dDx2, &dDx3, &iDirection) ;
    __fourier_MOD_fft3 (fData3, &iSx1, &iSx2, &iSx3, &dDx1, &dDx2, &dDx3, &iDirection) ;
    iDirection = -1 ;
    __fourier_MOD_fft2 (fData4, &iSx1, &iSx2, &dDx1, &dDx2, &iDirection) ;
    __fourier_MOD_fft2 (fData5, &iSx1, &iSx2, &dDx1, &dDx2, &iDirection) ;
    __fourier_MOD_fft2 (fData6, &iSx1, &iSx2, &dDx1, &dDx2, &iDirection) ;
#endif
    copyfftmemory (fData1, fData2, fData3, fData4, fData5, fData6, iSize, iSize2, 1) ;
#endif

#ifdef PAPI_PROF
    if (hipDeviceSynchronize() != hipSuccess)
    {
        printf("Cuda error: Failed to synchronize\n") ;
        goto FREE_AND_EXIT ;
    }	
    papiendprofiling_(cTimerName) ;
        
    strcpy (cTimerName, "Elastic         ") ;
    papistartprofiling_(cTimerName) ;
#endif
    dRatio1 = (dLambda + dMu) / (dLambda + (2 * dMu)) / dMu / (PI2 * PI2) ;
    dRatio2 =  dMu/ (dLambda + dMu) ;
	
	/* So far so good.. lets calculate elastic response */ 
    cuElasticResKernel <<<dimGrid1, dimBlock1>>> ((float *)gpCompData1, (float *)gpCompData2, 
                        (float *)gpCompData3, (iSx1/2 + 1), iSx2, iSx3, dDx1, dDx2, dDx3, dRatio1, 
                    dRatio2) ;
    cuError = hipGetLastError () ;
    CHECK_CUDA_ERROR ("cuElasticResKernel of calfftnelasic_", FREE_AND_EXIT)
	
    /* This is to turn the NAN to ZERO */
    cuError = hipMemset (gpCompData1, 0,(2 * sizeof (float))) ;
    CHECK_CUDA_ERROR ("cudaMemset1 of calfftnelasic_", FREE_AND_EXIT)

    cuError = hipMemset (gpCompData2, 0,(2 * sizeof (float))) ;
    CHECK_CUDA_ERROR ("cudaMemset1 of calfftnelasic_", FREE_AND_EXIT)

    cuError = hipMemset (gpCompData3, 0,(2 * sizeof (float))) ;
    CHECK_CUDA_ERROR ("cudaMemset1 of calfftnelasic_", FREE_AND_EXIT)

#ifdef PAPI_PROF
    if (hipDeviceSynchronize() != hipSuccess)
    {
        printf("Cuda error: Failed to synchronize\n") ;
        goto FREE_AND_EXIT ;
    }	
    papiendprofiling_(cTimerName) ;

    strcpy (cTimerName, "Surface         ") ;
    papistartprofiling_(cTimerName) ;
#endif

    cuError = doSurfaceTraction (dLambda, dMu, dGamma, iSx1, iSx2, iSx3, dDx1, dDx2, dDx3) ;

	// No need to sync here, its there in doSurfacetraction function. 
    if (cuError != hipSuccess)
    {
        printf ("Something went wrong in doSurfaceTraction check logs\n") ;
        goto FREE_AND_EXIT ;
    }

#ifdef PAPI_PROF
    strcpy (cTimerName, "cerruti         ") ;
    papiendprofiling_(cTimerName) ;
	
    strcpy (cTimerName, "inversefft      ") ;
    papistartprofiling_ (cTimerName) ;
#endif

#ifndef ENABLE_FFTW3
    *iRet = calfft ((float *)gpComp2dData1, (float *)gpComp2dData2, (float *)gpComp2dData3, 
                                             iSx1, iSx2, dDx1, dDx2, fData4, fData5, fData6)  ;
    if (0 != *iRet)
    {
        printf ("calfft failed \n") ;
        goto FREE_AND_EXIT ;
    }

    *iRet = inverseFFT ((float *)gpCompData1, (float *)gpCompData2, (float *)gpCompData3, 
                        iSx1, iSx2, iSx3, dDx1, dDx2, dDx3, fData1, fData2, fData3) ;
    if (0 != *iRet)	
    {
        printf ("inverseFFT failed \n") ;
        goto FREE_AND_EXIT ;
    }
	//copyfftmemory (fData1, fData2, fData3, fData4, fData5, fData6, iSize, iSize2, 0) ;

#endif


#ifdef ENABLE_FFTW3
    copyfftmemory (fData1, fData2, fData3, fData4, fData5, fData6, iSize, iSize2, 0) ;

#ifdef FFTW3
    iDirection = 1 ;
    __fourier_MOD_fft3 (fData1, &iSx1, &iSx2, &iSx3, &dDx1, &dDx2, &dDx3, &iDirection) ;
    __fourier_MOD_fft3 (fData2, &iSx1, &iSx2, &iSx3, &dDx1, &dDx2, &dDx3, &iDirection) ;
    __fourier_MOD_fft3 (fData3, &iSx1, &iSx2, &iSx3, &dDx1, &dDx2, &dDx3, &iDirection) ;
    iDirection = 1 ;
    __fourier_MOD_fft2 (fData4, &iSx1, &iSx2, &dDx1, &dDx2, &iDirection) ; 
    __fourier_MOD_fft2 (fData5, &iSx1, &iSx2, &dDx1, &dDx2, &iDirection) ;
    __fourier_MOD_fft2 (fData6, &iSx1, &iSx2, &dDx1, &dDx2, &iDirection) ;
#endif
    copyfftmemory (fData1, fData2, fData3, fData4, fData5, fData6, iSize, iSize2, 1) ;
#endif //ENABLE_FFTW3

#ifdef PAPI_PROF
    if (hipDeviceSynchronize() != hipSuccess)
    {
        printf("Cuda error: Failed to synchronize\n") ;
        goto FREE_AND_EXIT ;
    }
    papiendprofiling_(cTimerName) ;	
#endif	

    *iRet = 0 ;	
    return ;
	
FREE_AND_EXIT :
    hipFree (gpCompData1) ;
    hipFree (gpCompData2) ;
    hipFree (gpCompData3) ;

    hipFree (gpComp2dData1) ;
    hipFree (gpComp2dData2) ;
    hipFree (gpComp2dData3) ;
    return ;
}

static int calfft (float   *pSrcData1,
                   float   *pSrcData2,
                   float   *pSrcData3,
                   int     iSx1,
                   int     iSx2,
                   double  dDx1,
                   double  dDx2,
                   float   *pDestData1,
                   float   *pDestData2,
                   float   *pDestData3)
{
    hipfftResult_t   cuRet ;
    //	hipfftHandle     hInversePlan ;
    dim3            dimGrid(1, iSx2, 1) ;              
    dim3            dimBlock((iSx1 + 2), 1, 1) ; 
    hipError_t     cuError = hipSuccess ;
    int             iRet = 0 ;
    float           fScale = 0 ;
	
	/*cuRet = hipfftPlan2d(&hInversePlan, iSx2, iSx1, HIPFFT_C2R) ;
        if (HIPFFT_SUCCESS != cuRet)
	{
		printf ("calfft : Failed in plan creation\n") ;
		iRet = 1 ;
              	return iRet ; 
	}*/
    cuRet = hipfftExecC2R(hInvTwoPlan, (hipfftComplex *)pSrcData1, (hipfftReal *)pSrcData1) ;
    if (HIPFFT_SUCCESS != cuRet)
    {
        printf("calfft : execC2R failed 1\n") ;
        goto CALFFT_EXIT ;
    }

    cuRet = hipfftExecC2R(hInvTwoPlan, (hipfftComplex *)pSrcData2, (hipfftReal *)pSrcData2) ;
    if (HIPFFT_SUCCESS != cuRet)
    {                       
        printf("calfft : execC2R failed 2\n") ;
        goto CALFFT_EXIT ;
    }

    cuRet = hipfftExecC2R(hInvTwoPlan, (hipfftComplex *)pSrcData3, (hipfftReal *)pSrcData3) ;
    if (HIPFFT_SUCCESS != cuRet)
    {                       
        printf("calfft : execC2R failed 3\n") ;
        goto CALFFT_EXIT ;
    }

    fScale = (float) (1.0 / (iSx1 * dDx1 * iSx2 * dDx2)) ;

    scaling <float> <<<dimGrid, dimBlock>>> ((float *)pSrcData1, fScale, (iSx1 + 2), iSx2, 2) ;
    cuError = hipGetLastError () ;
    CHECK_CUDA_ERROR ("calfft: scaling ", CALFFT_EXIT)

    scaling <float> <<<dimGrid, dimBlock>>> ((float *)pSrcData2, fScale, (iSx1 + 2), iSx2, 2) ;
    cuError = hipGetLastError () ;
    CHECK_CUDA_ERROR ("calfft: scaling ", CALFFT_EXIT)

    scaling <float> <<<dimGrid, dimBlock>>> ((float *)pSrcData3, fScale, (iSx1 + 2), iSx2, 2) ;
    cuError = hipGetLastError () ;
    CHECK_CUDA_ERROR ("calfft: scaling ", CALFFT_EXIT)
        
    // hipfftDestroy (hInversePlan) ;
	
    return iRet ;	
	
CALFFT_EXIT:
    //hipfftDestroy (hInversePlan) ;
    printf ("Something went wrong in calfft function\n") ;
    iRet = 1 ;
    return iRet ;	
}

static int  inverseFFT (float 	*pSrcData1, 
                        float 	*pSrcData2, 
                        float 	*pSrcData3, 
                        int 	iSx1, 
                        int 	iSx2, 
                        int 	iSx3, 
                        double  dDx1, 
                        double  dDx2, 
                        double  dDx3, 
                        float   *pDestData1,
                        float   *pDestData2,
                        float   *pDestData3)
{
    hipfftResult_t   cuRet ;
        //hipfftHandle 	hInversePlan ; 
    hipError_t	cuError = hipSuccess ;
    int 		iRet = 0 ;
    dim3 		dimGrid(iSx3, iSx2, 1) ;
    dim3 		dimBlock((iSx1 + 2), 1, 1) ; 
    float 		fScale = 0 ;
	
	/*cuRet = hipfftPlan3d(&hInversePlan, iSx3, iSx2, iSx1, HIPFFT_C2R) ;
        if (HIPFFT_SUCCESS != cuRet)
	{
		printf ("inverseFFT : Failed in plan creation\n") ;
		iRet = 1 ;
		return iRet ;
	}*/


    cuRet = hipfftExecC2R (hInversePlan, (hipfftComplex *)pSrcData1, (hipfftReal *)pSrcData1) ;
    if (HIPFFT_SUCCESS != cuRet)
    {
        printf("inverseFFT: ExecC2R failed1\n");
        goto INVERSE_EXIT ;
    }
    cuRet = hipfftExecC2R (hInversePlan, (hipfftComplex *)pSrcData2, (hipfftReal *)pSrcData2) ;
    if (HIPFFT_SUCCESS != cuRet)
    {
        printf("inverseFFT: ExecC2R failed2\n");
        goto INVERSE_EXIT ;
    }
    cuRet = hipfftExecC2R (hInversePlan, (hipfftComplex *)pSrcData3, (hipfftReal *)pSrcData3) ;
    if (HIPFFT_SUCCESS != cuRet)
    {
        printf("inverseFFT: ExecC2R failed3\n");
        goto INVERSE_EXIT ;
    }

    fScale = (float)(1.0/(iSx1*dDx1*iSx2*dDx2*iSx3*dDx3)) ;
 
    scaling <float> <<<dimGrid, dimBlock>>> ((float *)pSrcData1, fScale, (iSx1 + 2), iSx2, iSx3) ;
    cuError = hipGetLastError () ;
    CHECK_CUDA_ERROR ("inverseFFT: scaling ", INVERSE_EXIT)

	scaling <float> <<<dimGrid, dimBlock>>> ((float *)pSrcData2, fScale, (iSx1 + 2), iSx2, iSx3) ;
    cuError = hipGetLastError () ;
    CHECK_CUDA_ERROR ("inverseFFT: scaling ", INVERSE_EXIT)
 
    scaling <float> <<<dimGrid, dimBlock>>> ((float *)pSrcData3, fScale, (iSx1 + 2), iSx2, iSx3) ;
    cuError = hipGetLastError () ;
    CHECK_CUDA_ERROR ("inverseFFT: scaling ", INVERSE_EXIT)
        
//        hipfftDestroy (hInversePlan) ;
	
    return iRet ;	

INVERSE_EXIT :
  //      hipfftDestroy (hInversePlan) ;
    printf ("Something went wrong in inverseFFT function\n") ;
    iRet = 1 ;
    return iRet ;
}


static void cuCerrutiModified (double  dGamma, 
                               double  dLambda,
                               double  dMu,
                               double  dDx1,
                               double  dDx2,
                               double  dDx3,
                               float   pfP1[],
                               float   pfP2[],
                               float   pfP3[],
                               int     iSx1,
                               int     iSx2,
                               int     iSx3,
                               float   *fData1,
                               float   *fData2,
                               float   *fData3)         
{
    dim3  dimGrid(iSx2, iSx1/2+1, 1) ;
    dim3  dimBlock(iSx3, 1, 1) ;       

	double  dAlpha ;
	
    dAlpha = (dLambda + dMu)/(dLambda + 2 * dMu) ;

    cuCerrutiModKernel <<<dimGrid, dimBlock>>>(dAlpha, dMu, dGamma, pfP1, pfP2, pfP3,
                       iSx1/2+1, iSx2, iSx3, dDx1, dDx2, dDx3, fData1, fData2, fData3) ;

    if( hipSuccess != hipGetLastError ())
    {
        printf ("icuCerrutiModified : Cerruti kernel failed\n");	
    }
}

__global__ void cuCerrutiModKernel (double 	dAlpha,
				    double      dMu,
				    double 	dGamma, 
				    float       pfP1[],
                        	    float       pfP2[],
                        	    float       pfP3[],
				    int         iSx1,
                        	    int         iSx2,
                        	    int         iSx3,
				    double      dDx1,
                        	    double      dDx2,
                        	    double      dDx3,
				    float       *fData1,
                        	    float       *fData2,
                        	    float       *fData3)
{
    unsigned int  iIdx1 ;
    unsigned int  iIdx2 ;
    unsigned int  iIdx3 ;
    unsigned int  iIdxSec ;
    unsigned int  iIdx ;
    double        dK1 ;
    double        dK2 ;
    double        dK3 ;
    double2       cuC1 ;
    double2       cuC2 ;
    double2       cuC3 ;
        
    float2 		  *pC1 ;
    float2 		  *pC2 ;
    float2 		  *pC3 ;
        
    float2        compSum1 ;
    float2        compSum2 ;
    float2        compSum3 ;

    iIdx3 = threadIdx.x ;
    iIdx2 = blockIdx.x ;
    iIdx1 = blockIdx.y ;

    if ((iIdx1 < iSx1) && (iIdx2 < iSx2) && (iIdx3 < iSx3))
    {
        cuWaveNumber (iIdx1, iIdx2, iIdx3, (2 * iSx1) - 2, iSx2, iSx3, dDx1, dDx2, dDx3, 
                      &dK1, &dK2, &dK3) ;

        iIdx = (iIdx2 * 2 * iSx1) + 2 * iIdx1 ; 
        iIdxSec = (((iIdx3 * iSx2) + iIdx2) * 2 * iSx1) + 2 * iIdx1 ;

        pC1 = (float2 *) &pfP1[iIdx] ;
        pC2 = (float2 *) &pfP2[iIdx] ;
        pC3 = (float2 *) &pfP3[iIdx] ;

        cuC1 = make_comp_double ((double) (*pC1).x, (double) (*pC1).y) ;
        cuC2 = make_comp_double ((double) (*pC2).x, (double) (*pC2).y) ;
        cuC3 = make_comp_double ((double) (*pC3).x, (double) (*pC3).y) ;
        /*
            cuC1 = make_comp_double ((double) pfP1[iIdx], (double) pfP1[iIdx + 1]) ;
            cuC2 = make_comp_double ((double) pfP2[iIdx], (double) pfP2[iIdx + 1]) ;
            cuC3 = make_comp_double ((double) pfP3[iIdx], (double) pfP3[iIdx + 1]) ;
        */

        cuCerrutiModDevice (dMu, cuC1, cuC2, cuC3, dAlpha, dGamma, &compSum1, &compSum2, &compSum3, 
                            dK1, dK2, dK3) ;

        atomicAdd (&fData1[iIdxSec], (float) compSum1.x) ;
        atomicAdd (&fData1[iIdxSec + 1], (float) compSum1.y) ;

        atomicAdd (&fData2[iIdxSec], (float) compSum2.x) ;
        atomicAdd (&fData2[iIdxSec + 1], (float) compSum2.y) ;

        atomicAdd (&fData3[iIdxSec], (float) compSum3.x) ;
        atomicAdd (&fData3[iIdxSec + 1], (float) compSum3.y) ;
    }
}

__device__ void cuCerrutiModDevice (double   dMu,
                                    double2  cuC1,
                                    double2  cuC2,
                                    double2  cuC3,
                                    double   dAlpha,
                                    double   dGamma,
                                    float2   *ppB1,
                                    float2   *ppB2,
                                    float2   *ppB3,
                                    double   dK1,
                                    double   dK2,
                                    double   dK3)
{
    double   dBeta ;
    double   dH ;
    double 	 dFi ;

    double2  i ;
    double2  cuB1 ;
    double2  cuB2 ;
    double2  cuB3 ;
    double2  cuTemp ;
    double2  cuTemp1 ;
    double2  cuV1 ;
    double2  cuV2 ;
    double2  cuV3 ;

    if ((0 == dK1) && (0 == dK2))
    {
        *ppB1 = make_comp_float (0, 0) ;
        *ppB2 = make_comp_float (0, 0) ;
        *ppB3 = make_comp_float (0, 0) ;
    }
    else
    {
        dBeta = PI2 * sqrt ((dK1 * dK1) + (dK2 * dK2)) ;
        i = make_comp_double (0, 1.0) ;
        dH = dGamma / dBeta ;
        dFi = (2 * dBeta) / (((PI2 * PI2) * (dK3 * dK3)) + (dBeta * dBeta)) ;

        cuTemp = make_comp_double ((1.0 / (2.0 * dMu * dBeta * dBeta * dBeta)) * dFi, 0) ;
        cuB1 = comp_double_mult (cuTemp, make_comp_double ((double)cuC1.x, (double)cuC1.y)) ;
        cuB2 = comp_double_mult (cuTemp, make_comp_double ((double)cuC2.x, (double)cuC2.y)) ; 

        //b3=(beta*p3+i*(1._8-alpha)*(pi2*k1*p1+pi2*k2*p2))/(2._8*alpha*mu*beta**4*(1+h))

        cuB3 = comp_double_div (comp_double_sub (comp_double_mult (
               make_comp_double ((double)cuC3.x, (double)cuC3.y), 
               make_comp_double (dBeta, 0)), comp_double_mult( comp_double_mult (comp_double_add (
               comp_double_mult (make_comp_double (PI2 * dK1, 0), make_comp_double (cuC1.x, cuC1.y)),   
               comp_double_mult (make_comp_double (PI2 * dK2, 0), make_comp_double (cuC2.x, cuC2.y))), 
               make_comp_double (1.0 - dAlpha, 0)), i)), 
               make_comp_double (2.0 * dAlpha * dMu * dBeta * dBeta * dBeta * dBeta * (1.0 + dH),0)) ;

    //tmp=alpha*i*beta*pi2*b3*(1._8-1._8/alpha-i*pi2*k3*fi)
    
        cuTemp = comp_double_mult (comp_double_mult (comp_double_mult (
                 make_comp_double (dAlpha * dBeta * PI2, 0), i), cuB3), 
                 comp_double_sub (make_comp_double (1.0 - (1.0 / dAlpha), 0),
                 comp_double_mult (i, make_comp_double (PI2 * dK3 * dFi, 0)))) ;
 
        cuV1 = comp_double_mult(make_comp_double(dFi, 0), 
               comp_double_mult (cuTemp, make_comp_double (dK1, 0))) ;
        cuV2 = comp_double_mult(make_comp_double(dFi, 0), 
               comp_double_mult (cuTemp, make_comp_double (dK2, 0))) ;
        cuV3 = comp_double_mult (make_comp_double(-1.0, 0), comp_double_mult (comp_double_mult (
               make_comp_double (dAlpha * dBeta * dBeta, 0), cuB3), 
               comp_double_sub (make_comp_double ((1.0 / dAlpha), 0), 
               comp_double_mult (i, make_comp_double (PI2 * dK3 * dFi, 0))))) ;

    /* cuTemp = alpha*(pi2**2)*(b1*k1+b2*k2)*(1._8-(i*pi2*k3*fi)))) */
    /* u1=CMPLX(v1+(-(2._8*beta**2*b1)+ cuTemp *k1)) */
     
        cuTemp = comp_double_mult (comp_double_mult (make_comp_double (dAlpha * PI2 * PI2, 0), 
                 comp_double_add (comp_double_mult (cuB1, make_comp_double (dK1, 0)), 
                 comp_double_mult (cuB2, make_comp_double (dK2, 0)))), 
                 comp_double_sub (make_comp_double (1.0, 0), 
                 comp_double_mult (i, make_comp_double (PI2 * dK3 * dFi, 0)))) ;
    
        cuTemp1 = comp_double_add (cuV1, comp_double_add (comp_double_mult (
                  make_comp_double (-2.0 * dBeta * dBeta, 0), cuB1), 
                  comp_double_mult (cuTemp, make_comp_double(dK1, 0)))) ;
        (*ppB1).x = (float) cuTemp1.x ;
        (*ppB1).y = (float) cuTemp1.y ;

    
        cuTemp1 = comp_double_add (cuV2, comp_double_add (comp_double_mult (
                  make_comp_double (-2.0 * dBeta * dBeta, 0), cuB2), 
                  comp_double_mult (cuTemp, make_comp_double(dK2, 0)))) ;
        (*ppB2).x = (float) cuTemp1.x ;
        (*ppB2).y = (float) cuTemp1.y ;


        cuTemp1 = comp_double_add ( comp_double_mult (cuV3, make_comp_double (dFi, 0)), 
                  comp_double_mult (make_comp_double(dAlpha * dBeta * PI2 * PI2 * dK3 * dFi, 0), 
                  comp_double_add (comp_double_mult (cuB1, make_comp_double (dK1, 0)),
                  comp_double_mult (cuB2, make_comp_double (dK2, 0))))) ;
    
        (*ppB3).x = (float) cuTemp1.x ;
        (*ppB3).y = (float) cuTemp1.y ;
    }
}


/*
 * @brief : This function to calculate the surface traction is called after the fft and elastic response.
 *          There is version of this function which is called from the surfacetractioncowling function, 
 * 	        this is to make sure that when the memory is insufficient the software just works fine.
*/

static hipError_t doSurfaceTraction (double        dLambda,
                                      double        dMu,
                                      double        dGamma,
                                      int           iSx1,
                                      int           iSx2,
                                      int           iSx3,
                                      double        dDx1,
                                      double        dDx2,
                                      double        dDx3)
{
    hipError_t cuError ;
    float       *pfP1 ;
    float       *pfP2 ;
    float       *pfP3 ;
    float       fScale ;

    dim3 		dimGrid(iSx2, iSx3, 1) ;
    dim3 		dimBlock((iSx1/2 + 1), 1, 1) ;

    dim3 		dimGrid1(iSx2, 1, 1) ;
    dim3 		dimBlock1((iSx1 + 2), 1, 1) ;

    double      dModulus ;
    double      dAlpha ;
    double      dGravity ;

#ifdef PAPI_PROF        
    char        cTimerName[17] = "Surface         " ;
#endif

#ifdef REDUCTION 
    int         blocks ;
    int         threads ; 

    blocks = iSx1 ;
    threads = iSx2 ;
#endif

    dModulus = dLambda + 2 * dMu ;
    dAlpha = (dLambda + dMu) / (dLambda + 2 * dMu) ;
    dGravity = 2 * dMu * dAlpha * dGamma ;

    fScale = (float)(1.0 / (iSx3 * dDx3)) ;

    cuError = hipMalloc((void**)&pfP1, ((iSx1+2) * iSx2 * sizeof (float))) ;
    if (hipSuccess != cuError)
    {
        printf ("doSurfaceTraction: Memory allocation failure 1\n") ;
        return cuError ;
    }

    cuError = hipMalloc((void**)&pfP2, ((iSx1+2) * iSx2 * sizeof (float))) ;
    if (hipSuccess != cuError)
    {
        hipFree (pfP1) ;
        printf ("doSurfaceTraction: Memory allocation failure 2\n") ;
        return cuError ;
    }

    cuError = hipMalloc((void**)&pfP3, ((iSx1+2) * iSx2 * sizeof (float))) ;
    if (hipSuccess != cuError)
    {
        hipFree (pfP1) ;
        hipFree (pfP2) ;
        printf ("doSurfaceTraction: Memory allocation failure 3\n") ;
        return cuError ;
    }

    cuError = hipMemset (pfP1, 0,((iSx1+2) * iSx2 * sizeof (float))) ;
    CHECK_CUDA_ERROR ("doSurfaceTraction: cudaMemset1", FREE_DATA_DO_SURFACE)

    cuError = hipMemset (pfP2, 0,((iSx1+2) * iSx2 * sizeof (float))) ;
    CHECK_CUDA_ERROR ("doSurfaceTraction: cudaMemset2", FREE_DATA_DO_SURFACE)

    cuError = hipMemset (pfP3, 0,((iSx1+2) * iSx2 * sizeof (float))) ;
    CHECK_CUDA_ERROR ("doSurfaceTraction: cudaMemset3", FREE_DATA_DO_SURFACE)
#ifdef REDUCTION
	
    cuSurfaceReduction <<<blocks, threads>>> (dModulus, dLambda, dMu, dGravity, dDx1, dDx2, dDx3,
                                              pfP1, pfP2, pfP3, (iSx1/2 + 1), iSx2, iSx3,
                                              (float *)gpCompData1, (float *)gpCompData2, 
                                              (float *)gpCompData3) ;
#else 
    cuSurfaceKernel <<<dimGrid, dimBlock>>> (dModulus, dLambda, dMu, dGravity, dDx1, dDx2, dDx3,
                                             pfP1, pfP2, pfP3, (iSx1/2 + 1), iSx2, iSx3,
                                             (float *)gpCompData1, (float *)gpCompData2, 
                                             (float *)gpCompData3) ;

#endif
    cuError = hipGetLastError () ;
    CHECK_CUDA_ERROR ("doSurfaceTraction : cuSurfaceKernel ", FREE_DATA_DO_SURFACE)

    scaleAndSub <<<dimGrid1, dimBlock1>>> (pfP1, (float *) gpComp2dData1, fScale, 
                                           (iSx1 + 2), iSx2, 2) ;
    cuError = hipGetLastError () ;
    CHECK_CUDA_ERROR ("doSurfaceTraction : scalingKernel ", FREE_DATA_DO_SURFACE)

    scaleAndSub <<<dimGrid1, dimBlock1>>> (pfP2, (float *) gpComp2dData2, fScale, 
                                           (iSx1 + 2), iSx2, 2) ;
    cuError = hipGetLastError () ;
    CHECK_CUDA_ERROR ("doSurfaceTraction : scalingKernel ", FREE_DATA_DO_SURFACE)

    scaleAndSub <<<dimGrid1, dimBlock1>>> (pfP3, (float *) gpComp2dData3, fScale, 
                                           (iSx1 + 2), iSx2, 2) ;
    cuError = hipGetLastError () ;
    CHECK_CUDA_ERROR ("doSurfaceTraction :	scalingKernel ", FREE_DATA_DO_SURFACE)

#ifdef PAPI_PROF        
    if (hipDeviceSynchronize() != hipSuccess)
    {
        printf("Cuda error: Failed to synchronize\n") ;
        goto FREE_DATA_DO_SURFACE;
    }

    papiendprofiling_(cTimerName) ;

    strcpy (cTimerName, "cerruti         ") ;
    papistartprofiling_(cTimerName) ;

#endif

    cuCerrutiModified (dGamma, dLambda, dMu, dDx1, dDx2, dDx3, (float *)pfP1, (float *)pfP2,       
                       (float *)pfP3, iSx1, iSx2, iSx3, (float *)gpCompData1, (float *)gpCompData2,
                       (float *)gpCompData3);


    if (hipDeviceSynchronize() != hipSuccess)
    {
        printf("Cuda error: Failed to synchronize after cuCerrutiModified call\n") ;
    }
        

FREE_DATA_DO_SURFACE :
    hipFree (pfP1) ;
    hipFree (pfP2) ;
    hipFree (pfP3) ;
        
    return cuError ;	
} 

#ifdef REDUCTION

__global__ void cuSurfaceReduction(double  dModulus,
                                   double  dLambda,
                                   double  dMu,
                                   double  dGravity,
                                   double  dDx1,
                                   double  dDx2,
                                   double  dDx3,
                                   float   pfP1[],
                                   float   pfP2[],
                                   float   pfP3[],
                                   int     iSx1,
                                   int     iSx2,
                                   int     iSx3,
                                   float   *fData1,
                                   float   *fData2,
                                   float   *fData3)
{
	unsigned int iIdx ;
	unsigned int iStride ;
	unsigned int iElements ;
	unsigned int iIdxSec ;
	unsigned int iIdx1 ;
	unsigned int iIdx2 ;
	unsigned int iIdx3 ;

    iStride = 2  ;

    if ((threadIdx.x < blockDim.x) && (blockIdx.x < gridDim.x))
    {	
        iIdx = (blockIdx.x * (iSx1 * 2))  +  (threadIdx.x * (iSx2 * iSx1 * 2)) ;
        if (iIdx < (iSx2 * iSx3 * iSx1 * 2))
        {
            iElements = iIdx + (iSx1 * 2) ; 
            iIdxSec = blockIdx.x * (iSx1 * 2)  ;
        
            iIdx1 = 0 ;
            iIdx2 = blockIdx.x ;
            iIdx3 = threadIdx.x ;

            while (iIdx < iElements)
            {
                surfaceCal (dModulus, dLambda, dMu, dGravity, dDx1, dDx2, dDx3,
                            pfP1, pfP2, pfP3, iSx1, iSx2, iSx3,
                            fData1, fData2, fData3,
                            iIdx, iIdxSec, iIdx1, iIdx2, iIdx3) ;
                    
                iIdx += iStride ;
                iIdxSec += iStride ;
                iIdx1 += 1 ;	
            }
        }
    }
}

__device__ void surfaceCal (double        dModulus,
                            double        dLambda,
                            double        dMu,
                            double        dGravity,
                            double        dDx1,
                            double        dDx2,
                            double        dDx3,
                            float         pfP1[],
                            float         pfP2[],
                            float         pfP3[],
                            int           iSx1,
                            int           iSx2,
                            int           iSx3,
                            float         *fData1,
                            float         *fData2,
                            float         *fData3, 
				            unsigned int  iIdx, 
				            unsigned int  iIdxSec, 
				            unsigned int  iIdx1,
				            unsigned int  iIdx2,
				            unsigned int  iIdx3) 
{
    double   dK1 ;
    double   dK2 ;
    double   dK3 ;

    double2  cuC1 ;
    double2  cuC2 ;
    double2  cuC3 ;

    double2  compSum1 ;
    double2  compSum2 ;
    double2  compSum3 ;
        
    cuWaveNumber (iIdx1, iIdx2, iIdx3, (2 * iSx1) - 2, iSx2, iSx3, dDx1, dDx2, dDx3, 
                  &dK1, &dK2, &dK3) ;

    cuC1 = make_comp_double ((double) fData1[iIdx], (double) fData1[iIdx + 1]) ;
    cuC2 = make_comp_double ((double) fData2[iIdx], (double) fData2[iIdx + 1]) ;
    cuC3 = make_comp_double ((double) fData3[iIdx], (double) fData3[iIdx + 1]) ;

    compSum1 = comp_double_mult (comp_double_mult (make_comp_double (0, PI2), 
               make_comp_double (dMu, 0)), comp_double_add (comp_double_mult (
               make_comp_double (dK3, 0), cuC1), comp_double_mult (make_comp_double (dK1, 0), 
               cuC3))) ;

    compSum2 = comp_double_mult (comp_double_mult (make_comp_double (0, PI2), 
               make_comp_double (dMu, 0)), comp_double_add (comp_double_mult (
               make_comp_double (dK3, 0), cuC2), comp_double_mult (make_comp_double (dK2, 0), 
               cuC3))) ;

    compSum3 = comp_double_sub (comp_double_mult (make_comp_double (0, PI2),
               comp_double_add (comp_double_mult (comp_double_mult (make_comp_double 
               (dModulus, 0), make_comp_double(dK3, 0)), cuC3),
               comp_double_mult (make_comp_double (dLambda, 0),
               comp_double_add (comp_double_mult (make_comp_double (dK1, 0), cuC1),
               comp_double_mult (make_comp_double (dK2, 0), cuC2))))),
               comp_double_mult (make_comp_double (dGravity, 0), cuC3)) ;

    atomicAdd (&pfP1[iIdxSec], (float) compSum1.x) ;
    atomicAdd (&pfP1[iIdxSec + 1], (float) compSum1.y) ;

    atomicAdd (&pfP2[iIdxSec], (float) compSum2.x) ;
    atomicAdd (&pfP2[iIdxSec + 1], (float) compSum2.y) ;

    atomicAdd (&pfP3[iIdxSec], (float) compSum3.x) ;
    atomicAdd (&pfP3[iIdxSec + 1], (float) compSum3.y) ;

}
#endif 


__global__ void scaleAndSub (float  *pCompData1,
                             float 	*pCompData2,
                             float  fScale,
                             int    iNx,
                             int    iNy,
                             int    iNz)
{
    int           iX ;
    int           iY ;
    int           iZ ;
    unsigned int  iIdx ;

    iX = threadIdx.x ;
    iY = blockIdx.x ;
    iZ = blockIdx.y ;

    if ((iX < iNx) && (iY < iNy) && (iZ < iNz))
    {
        iIdx = (((iZ * iNy) + iY) * iNx) + iX ;
        pCompData1[iIdx] = pCompData2[iIdx] - (pCompData1[iIdx] * fScale) ;
    }
}

/**
 * @brief 	This function is called from the surfacetractioncowling of fortran code in case there 
 *          was insufficient memory and couldnt process all the things at once.
 */

extern "C" void cusurfacetraction_ (double 	dModulus, 
                                    double 	dAlpha, 
                                    double 	dLambda, 
                                    double 	dMu, 
                                    double 	dGravity, 
                                    float 	*fData1,
                                    float 	*fData2,
                                    float 	*fData3,
                                    double 	dDx1, 
                                    double 	dDx2, 
                                    double 	dDx3,
                                    float 	*fOutData1,
                                    float 	*fOutData2,
                                    float 	*fOutData3,
                                    int 	iSx1,
                                    int 	iSx2,
                                    int 	iSx3)
{
    hipError_t cuError ;
    int 	    iSize ;
    float       *pfP1 ;
    float       *pfP2 ;
    float       *pfP3 ;
    float       fScale ;

    dim3 dimGrid(iSx2, iSx3, 1) ;
    dim3 dimBlock((iSx1/2 + 1), 1, 1) ;

    dim3 dimGrid1(iSx2, 1, 1) ;
    dim3 dimBlock1((iSx1 + 2), 1, 1) ;

    iSize = (sizeof (float) * (iSx1 + 2) * (iSx2) * (iSx3)) ;

    cuError = hipMalloc((void**)&gpCompData1, iSize) ;
    if (hipSuccess != cuError)
    {
        printf ("Memory allocation failure 1\n") ;
        return ;
    }

    cuError = hipMalloc((void**)&gpCompData2, iSize) ;
    if (hipSuccess != cuError)
    {
        hipFree (gpCompData1) ;
        printf ("Memory allocation failure 2\n") ;
        return ;
    }
    cuError = hipMalloc((void**)&gpCompData3, iSize) ;
    if (hipSuccess != cuError)
    {
        hipFree (gpCompData1) ;
        hipFree (gpCompData2) ;
        printf ("Memory allocation failure 3\n") ;
        return ;
    }

    cuError = hipMalloc((void**)&pfP1, ((iSx1+2) * iSx2 * sizeof (float))) ;
    if (hipSuccess != cuError)
    {
        printf ("Memory allocation failure 4\n") ;
        goto FREE_DATA_SURFACE ;
    }
        
    cuError = hipMalloc((void**)&pfP2, ((iSx1+2) * iSx2 * sizeof (float))) ;
    if (hipSuccess != cuError)
    {
        hipFree (pfP1) ;
        printf ("Memory allocation failure 5\n") ;
        goto FREE_DATA_SURFACE ;
    }
        
    cuError = hipMalloc((void**)&pfP3, ((iSx1+2) * iSx2 * sizeof (float))) ;
    if (hipSuccess != cuError)
    {
        hipFree (pfP1) ;
        hipFree (pfP2) ;
        printf ("Memory allocation failure 6\n") ;
        goto FREE_DATA_SURFACE ;
    }
        
    /* To make sure all are 0 since we add in the kernel */
    hipMemset (pfP1, 0,((iSx1+2) * iSx2 * sizeof (float))) ;
    CHECK_CUDA_ERROR ("SurfaceTraction: cudaMemset1", FREE_DATA_SURFACE)

    hipMemset (pfP2, 0,((iSx1+2) * iSx2 * sizeof (float))) ;
    CHECK_CUDA_ERROR ("SurfaceTraction: cudaMemset2", FREE_DATA_SURFACE)

    hipMemset (pfP3, 0,((iSx1+2) * iSx2 * sizeof (float))) ;
    CHECK_CUDA_ERROR ("SurfaceTraction: cudaMemset3", FREE_DATA_SURFACE)

    cuError = hipMemcpy (gpCompData1, fData1, iSize, hipMemcpyHostToDevice) ;
    CHECK_CUDA_ERROR ("SurfaceTraction: cudaMemcpy1", FREE_DATA_SURFACE)

    cuError = hipMemcpy (gpCompData2, fData2, iSize, hipMemcpyHostToDevice) ;
    CHECK_CUDA_ERROR ("SurfaceTraction: cudaMemcpy2", FREE_DATA_SURFACE)

    cuError = hipMemcpy (gpCompData3, fData3, iSize, hipMemcpyHostToDevice) ;
    CHECK_CUDA_ERROR ("SurfaceTraction: cudaMemcpy3", FREE_DATA_SURFACE)

    fScale = (float)(1.0 / (iSx3 * dDx3)) ;
        
    cuSurfaceKernel <<<dimGrid, dimBlock>>> (dModulus, dLambda, dMu, dGravity, dDx1, dDx2, dDx3, 
                                             pfP1, pfP2, pfP3, (iSx1/2 + 1), iSx2, iSx3, 
                                             (float *)gpCompData1, (float *)gpCompData2, 
                                             (float *)gpCompData3) ;
    cuError = hipGetLastError () ;
    CHECK_CUDA_ERROR ("Surfacetraction : cuSurfaceKernel ", FREE_DATA_SURFACE)
        
    scaling <float> <<<dimGrid1, dimBlock1>>> (pfP1, fScale, (iSx1 + 2), iSx2, 2) ;
    cuError = hipGetLastError () ;
    CHECK_CUDA_ERROR ("SurfaceTraction : scalingKernel 1", FREE_DATA_SURFACE)

    scaling <float> <<<dimGrid1, dimBlock1>>> (pfP2, fScale, (iSx1 + 2), iSx2, 2) ;
    cuError = hipGetLastError () ;
    CHECK_CUDA_ERROR ("SurfaceTraction : scalingKernel 2", FREE_DATA_SURFACE)

    scaling <float> <<<dimGrid1, dimBlock1>>> (pfP3, fScale, (iSx1 + 2), iSx2, 2) ;
    cuError = hipGetLastError () ;
    CHECK_CUDA_ERROR ("SurfaceTraction : scalingKernel 3", FREE_DATA_SURFACE)

        
    cuError = hipMemcpy (fOutData1, pfP1, ((iSx1+2) * iSx2 * sizeof (float)), 
                          hipMemcpyDeviceToHost) ;
    CHECK_CUDA_ERROR ("SurfaceTraction: cudaMemcpy4", FREE_DATA_SURFACE)	

    cuError = hipMemcpy (fOutData2, pfP2, ((iSx1+2) * iSx2 * sizeof (float)), 
                          hipMemcpyDeviceToHost) ;
    CHECK_CUDA_ERROR ("SurfaceTraction: cudaMemcpy5", FREE_DATA_SURFACE)

    cuError = hipMemcpy (fOutData3, pfP3, ((iSx1+2) * iSx2 * sizeof (float)), 
                          hipMemcpyDeviceToHost) ;
    CHECK_CUDA_ERROR ("SurfaceTraction: cudaMemcpy6", FREE_DATA_SURFACE)


FREE_DATA_SURFACE :
    hipFree (gpCompData1) ;
    hipFree (gpCompData2) ;
    hipFree (gpCompData3) ;
    hipFree (pfP1) ;
    hipFree (pfP2) ;
    hipFree (pfP3) ;

    return ;
}
	
	
/**
 * brief :	This is kernel  funtion for surfacetraction calculations.
 *
 */

__global__ void cuSurfaceKernel (double  dModulus,
                                 double  dLambda,
                                 double  dMu,
                                 double  dGravity,
                                 double  dDx1,
                                 double  dDx2,
                                 double  dDx3,
                                 float   pfP1[],
                                 float   pfP2[],
                                 float   pfP3[],
                                 int     iSx1,
                                 int     iSx2,
                                 int     iSx3, 
                                 float 	 *fData1, 
                                 float 	 *fData2, 
                                 float 	 *fData3)
{
    unsigned int iIdx1 ;
    unsigned int iIdx2 ;
    unsigned int iIdx3 ;
    unsigned int iIdxSec ;
    unsigned int iIdx ;
        
    double       dK1 ;
    double       dK2 ;
    double       dK3 ;

    float2 		 *pC1 ;
    float2 		 *pC2 ;
    float2 		 *pC3 ;
        
    double2 	 cuC1 ;
    double2 	 cuC2 ;
    double2 	 cuC3 ;

    double2 	 compSum1 ;
    double2 	 compSum2 ;
    double2 	 compSum3 ;

    iIdx1 = threadIdx.x ;
    iIdx2 = blockIdx.x ;
    iIdx3 = blockIdx.y ;

    if ((iIdx1 < iSx1) && (iIdx2 < iSx2) && (iIdx3 < iSx3))
    {
                    
        cuWaveNumber (iIdx1, iIdx2, iIdx3, (2 * iSx1) - 2, iSx2, iSx3, dDx1, dDx2, dDx3, 
                      &dK1, &dK2, &dK3) ;

        iIdx = (((iIdx3 * iSx2) + iIdx2) * 2 * iSx1) + 2 * iIdx1 ;
                
        //printf ("iIdx = %u\n", iIdx) ;	
        pC1 = (float2 *) &fData1[iIdx] ;
        pC2 = (float2 *) &fData2[iIdx] ;
        pC3 = (float2 *) &fData3[iIdx] ;

        cuC1 = make_comp_double ((double) ((*pC1).x), (double) ((*pC1).y)) ;
        cuC2 = make_comp_double ((double) ((*pC2).x), (double) ((*pC2).y)) ;
        cuC3 = make_comp_double ((double) ((*pC3).x), (double) ((*pC3).y)) ;
            
        /*cuC1 = make_comp_double ((double) fData1[iIdx], (double) fData1[iIdx + 1]) ;
        cuC2 = make_comp_double ((double) fData2[iIdx], (double) fData2[iIdx + 1]) ;
        cuC3 = make_comp_double ((double) fData3[iIdx], (double) fData3[iIdx + 1]) ;	
        */
        compSum1 = comp_double_mult (comp_double_mult (make_comp_double (0, PI2), 
                   make_comp_double (dMu, 0)), comp_double_add (comp_double_mult (
                   make_comp_double (dK3, 0), cuC1), comp_double_mult (make_comp_double (dK1, 0), 
                   cuC3))) ;
          
        compSum2 = comp_double_mult (comp_double_mult (make_comp_double (0, PI2), 
                   make_comp_double (dMu, 0)), comp_double_add (comp_double_mult (
                   make_comp_double (dK3, 0), cuC2), comp_double_mult (make_comp_double (dK2, 0), 
                   cuC3))) ; 

        compSum3 = comp_double_sub (comp_double_mult (make_comp_double (0, PI2), 
                   comp_double_add (comp_double_mult (comp_double_mult (
                   make_comp_double (dModulus, 0), make_comp_double(dK3, 0)), cuC3), 
                   comp_double_mult (make_comp_double (dLambda, 0), 
                   comp_double_add (comp_double_mult (make_comp_double (dK1, 0), cuC1), 
                   comp_double_mult (make_comp_double (dK2, 0), cuC2))))), 
                   comp_double_mult (make_comp_double (dGravity, 0), cuC3)) ;

        iIdxSec = (iIdx2 * 2 * iSx1) + 2 * iIdx1 ;		

        atomicAdd (&pfP1[iIdxSec], (float) compSum1.x) ;
        atomicAdd (&pfP1[iIdxSec + 1], (float) compSum1.y) ;
                
        atomicAdd (&pfP2[iIdxSec], (float) compSum2.x) ;
        atomicAdd (&pfP2[iIdxSec + 1], (float) compSum2.y) ;
                
        atomicAdd (&pfP3[iIdxSec], (float) compSum3.x) ;
        atomicAdd (&pfP3[iIdxSec + 1], (float) compSum3.y) ;
        /*	
            pfP1[iIdxSec] += (float) compSum1.x ; 
            pfP1[iIdxSec + 1] += (float) compSum1.y ; 
            
            pfP2[iIdxSec] += (float) compSum2.x ;
                    pfP2[iIdxSec + 1] += (float) compSum2.y ;

            pfP3[iIdxSec] += (float) compSum3.x ;
                    pfP3[iIdxSec + 1] += (float) compSum3.y ;
     */
    }	
}

static void cuAllocate (int iSize)
{
    hipError_t     cuError ;

    cuError = hipMalloc((void**)&pfB1, iSize) ;
    if (hipSuccess != cuError)
    {
        printf ("cuAllocate : Memory allocation failure 1\n") ;
        return ;
    }

    cuError = hipMalloc((void**)&pfB2, iSize) ;
    if (hipSuccess != cuError)
    {
        hipFree (pfB1) ;
        printf ("cuAllocate : Memory allocation failure 2\n") ;
        return ;
    }

    cuError = hipMalloc((void**)&pfB3, iSize) ;
    if (hipSuccess != cuError)
    {
        hipFree (pfB1) ;
        hipFree (pfB2) ;
        printf ("cuAllocate : Memory allocation failure 3\n") ;
        return ;
    }
}


extern "C" void cucerruti_ (float   *p1,
                            float   *p2,
                            float   *p3,
                            float   *fData1,
                            float   *fData2,
                            float   *fData3,
                            double  dLambda,
                            double  dMu,
                            double  dGamma,
                            double  dDx1,
                            double  dDx2,
                            double  dDx3,
                            int     iSx1,
                            int     iSx2,
                            int     iSx3)
{
    double      dAlpha ;
    hipError_t cuError ;
    int         iSize ;


    iSize = (sizeof (float) * (iSx1 + 2) * (iSx2) * (iSx3)) ;

    cuError = hipMalloc((void**)&gpCompData1, iSize) ;
    if (hipSuccess != cuError)
    {
        printf ("Memory allocation failure 1\n") ;
        return  ;
    }

    cuError = hipMalloc((void**)&gpCompData2, iSize) ;
    if (hipSuccess != cuError)
    {
        hipFree (gpCompData1) ;
        printf ("Memory allocation failure 2\n") ;
        return ;
    }

    cuError = hipMalloc((void**)&gpCompData3, iSize) ;
    if (hipSuccess != cuError)
    {
        hipFree (gpCompData1) ;
        hipFree (gpCompData2) ;
        printf ("Memory allocation failure 3\n") ;
        return ;
    }

    cuError = hipMemcpy (gpCompData1, fData1, iSize, hipMemcpyHostToDevice) ;
    CHECK_CUDA_ERROR ("cucerruti_: cudaMemcpy1", FREE_DATA_CERRUTI)

    cuError = hipMemcpy (gpCompData2, fData2, iSize, hipMemcpyHostToDevice) ;
    CHECK_CUDA_ERROR ("cucerruti_: cudaMemcpy2", FREE_DATA_CERRUTI)

    cuError = hipMemcpy (gpCompData3, fData3, iSize, hipMemcpyHostToDevice) ;
    CHECK_CUDA_ERROR ("cucerruti_: cudaMemcpy3", FREE_DATA_CERRUTI)

    dAlpha = (dLambda + dMu) / (dLambda + (2 * dMu)) ;


    cerruti (p1, p2, p3, iSx1, iSx2, iSx3, dDx1, dDx2, dDx3, dMu,
             dLambda, dAlpha, dGamma, (float *)gpCompData1, 
             (float *)gpCompData2, (float *)gpCompData3) ;

    cuError = hipMemcpy (fData1, gpCompData1, iSize, hipMemcpyDeviceToHost) ;
    CHECK_CUDA_ERROR ("cucerruti_: cudaMemcpy1", FREE_DATA_CERRUTI)

    cuError = hipMemcpy (fData2, gpCompData2, iSize, hipMemcpyDeviceToHost) ;
    CHECK_CUDA_ERROR ("cucerruti_: cudaMemcpy2", FREE_DATA_CERRUTI)

    cuError = hipMemcpy (fData3, gpCompData3, iSize, hipMemcpyDeviceToHost) ;
    CHECK_CUDA_ERROR ("cucerruti_: cudaMemcpy3", FREE_DATA_CERRUTI)

FREE_DATA_CERRUTI:
    hipFree (gpCompData1) ;
    hipFree (gpCompData2) ;
    hipFree (gpCompData3) ;

}

static void cerruti (float   *p1,
                     float   *p2,
                     float   *p3,
                     int     iSx1,
                     int     iSx2,
                     int     iSx3,
                     double  dDx1,
                     double  dDx2,
                     double  dDx3,
                     double  dMu,
                     double  dLambda,
                     double  dAlpha,
                     double  dGamma,
                     float   *pfU1,
                     float   *pfU2,
                     float   *pfU3)
{
    int          iIn1 ;
    int          iIn2 ;
    dim3         dimGrid(1, 1, 1) ;
    dim3         dimBlock (iSx3, 1, 1) ;
    int          iIdxSec ;

    hipfftResult  cuRet ;
    double       dK1 ;
    double       dK2 ;
    double       dK3 ;

    double2      cuC1 ;
    double2      cuC2 ;
    double2      cuC3 ;


    cuAllocate (iSx3 * sizeof (hipfftComplex)) ;

    cuRet = hipfftPlan1d(&ghPlanOne, iSx3, HIPFFT_C2C, 1) ;
    if (HIPFFT_SUCCESS != cuRet)
    {
        printf ("Plan creation is failed\n") ;
        return ;
    }
            
    for (iIn2 = 0; iIn2 < iSx2 ; iIn2++)
    {
        for (iIn1 = 0; iIn1 < ((iSx1/2)+1) ; iIn1++)
        {
            cuWaveNumber (iIn1, iIn2, 1, iSx1, iSx2, 1, dDx1, dDx2, 1, &dK1, &dK2, &dK3) ;

            iIdxSec = (iIn2 * ((iSx1/2)+1) * 2) + 2 * iIn1 ;

            cuC1 = make_comp_double ((double)p1[iIdxSec], (double)p1[iIdxSec + 1]) ;
            cuC2 = make_comp_double ((double)p2[iIdxSec], (double)p2[iIdxSec + 1]) ;
            cuC3 = make_comp_double ((double)p3[iIdxSec], (double)p3[iIdxSec + 1]) ;


            cerrutiKernel <<<dimGrid, dimBlock>>> (cuC1, cuC2, cuC3, (float2 *)pfB1, (float2 *)pfB2,
                                                   (float2 *)pfB3, iSx3, dK1, dK2, dDx3, dMu, 
                                                   dAlpha, dGamma) ;

            if (hipDeviceSynchronize() != hipSuccess)
            {
                fprintf(stderr, "Cuda error: Failed to synchronize\n") ;
                return ;
            }
            OneDfft (ghPlanOne, pfB1, pfB2, pfB3, iSx3, dDx3) ;

            cuAddKernel <<<dimGrid, dimBlock>>> (pfU1, pfU2, pfU3, (float2 *)pfB1, (float2 *)pfB2, 
                                                 (float2 *)pfB3, iIn1, iIn2, iSx1, iSx2, iSx3) ;

            if (hipDeviceSynchronize() != hipSuccess)
            {
                fprintf(stderr, "Cuda error: Failed to synchronize\n") ;
                return ;
            }
        }
    }
    hipfftDestroy (ghPlanOne) ;
    hipFree (pfB1) ;
    hipFree (pfB2) ;
    hipFree (pfB3) ;

}

__global__ void cuAddKernel (float   *pfU1,
                             float   *pfU2,
                             float   *pfU3,
                             float2  *pfB1,
                             float2  *pfB2,
                             float2  *pfB3,
                             int     iIn1,
                             int     iIn2,
                             int     iSx1,
                             int     iSx2,
                             int     iSx3)
{
    int     iIdx ;
    int     iIdx1 ;

    iIdx1 = threadIdx.x ;

    if (iIdx1 < iSx3)
    {
        iIdx = ((iIdx1 * iSx2) + iIn2) * (iSx1 + 2) + 2 * iIn1 ;

        pfU1[iIdx] += pfB1[iIdx1].x ;
        pfU1[iIdx + 1] += pfB1[iIdx1].y ;

        pfU2[iIdx] += pfB2[iIdx1].x ;
        pfU2[iIdx + 1] += pfB2[iIdx1].y ;

        pfU3[iIdx] += pfB3[iIdx1].x ;
        pfU3[iIdx + 1] += pfB3[iIdx1].y ;
    }
}

__global__ void cerrutiKernel (double2  cuC1,
                               double2  cuC2,
                               double2  cuC3,
                               float2   *pfB1,
                               float2   *pfB2,
                               float2   *pfB3,
                               int      iSx3,
                               double   dK1,
                               double   dK2,
                               double   dDx3,
                               double   dMu,
                               double   dAlpha,
                               double   dGamma)
{
    unsigned int iIdx ;
    double       dX3 ;
    float2       dB1 ;
    float2       dB2 ;
    float2       dB3 ;

    iIdx = threadIdx.x ;

    if (iSx3 > iIdx)
    {
        if (iIdx < iSx3/2)
        {
            dX3 = iIdx * dDx3 ;
        }
        else
        {
            dX3 = ((iSx3 - iIdx) * dDx3) ;
        }

        cerrutiSolution (dMu, cuC1, cuC2, cuC3, dAlpha, dGamma,  &dB1,
        &dB2, &dB3, dK1, dK2, dX3, ((double) (iSx3/2) * dDx3), iIdx) ;

        pfB1[iIdx].x = dB1.x ;
        pfB1[iIdx].y = dB1.y ;

        pfB2[iIdx].x = dB2.x ;
        pfB2[iIdx].y = dB2.y ;

        pfB3[iIdx].x = dB3.x ;
        pfB3[iIdx].y = dB3.y ;
    }
}

static void OneDfft (hipfftHandle  ghPlanOne,
                     float        *pB1,
                     float        *pB2,
                     float        *pB3,
                     int          iSx3,
                     double       dDx3)
{
    hipfftResult     cuRet ;
    dim3 dimGrid(1,1,1) ;
    dim3 dimBlock(iSx3,1,1) ;

    cuRet = hipfftExecC2C(ghPlanOne, (hipfftComplex *)pB1, (hipfftComplex *)pB1, HIPFFT_FORWARD) ;
    if (HIPFFT_SUCCESS != cuRet)
    {
        printf ("OneDfft : Failed 1\n") ;
        return ;
    }

    cuRet = hipfftExecC2C(ghPlanOne, (hipfftComplex *) pB2, (hipfftComplex *) pB2, HIPFFT_FORWARD) ;
    if (HIPFFT_SUCCESS != cuRet)
    {
        printf ("OneDfft : Failed 2\n") ;
        return ;
    }
    cuRet = hipfftExecC2C(ghPlanOne, (hipfftComplex *)pB3, (hipfftComplex *)pB3, HIPFFT_FORWARD) ;
    if (HIPFFT_SUCCESS != cuRet)
    {
        printf ("OneDfft : Failed 3\n") ;
    }
    scale1D <<<dimGrid, dimBlock>>> (pB1, pB2, pB3, (float)dDx3) ;

}

__global__ void scale1D (float *pB1,
                         float *pB2,
                         float *pB3,
                         float fScale)
{
    int iIdx = threadIdx.x ;

    if (iIdx < blockDim.x)
    {
        iIdx = 2 * iIdx ;

        pB1[iIdx] = pB1[iIdx] * fScale ;
        pB1[iIdx + 1] = pB1[iIdx + 1] * fScale ;
        pB2[iIdx] = pB2[iIdx] * fScale ;
        pB2[iIdx + 1] = pB2[iIdx + 1] * fScale ;
        pB3[iIdx] = pB3[iIdx] * fScale ;
        pB3[iIdx + 1] = pB3[iIdx + 1] * fScale ;
    }

}

__device__ void cerrutiSolution (double   dMu,
                                 double2  cuC1,
                                 double2  cuC2,
                                 double2  cuC3,
                                 double   dAlpha,
                                 double   dGamma,
                                 float2   *ppB1,
                                 float2   *ppB2,
                                 float2   *ppB3,
                                 double   dK1,
                                 double   dK2,
                                 double   dX3,
                                 double   dL,
				                 int 	  iIdx)
{
    double   dBeta ;
    double   dDepthDecay ;
    double   dH ;
    double   dTemp ;

    double2  i ;
    double2  cuB1 ;
    double2  cuB2 ;
    double2  cuB3 ;
    double2  cuTemp ;
    double2  cuV1 ;
    double2  cuV2 ;
    double2  cuV3 ;
            
    if ((0 == dK1) && (0 == dK2))
    {
        *ppB1 = make_comp_float ((cuC1.x / dMu) * (dX3 - dL), 0) ;
        *ppB2 = make_comp_float ((cuC2.x / dMu) * (dX3 - dL), 0) ;
        *ppB3 = make_comp_float (((cuC3.x / dMu) * (dX3 - dL) * (1.0 - dAlpha)) /
                                 (1.0 + 2.0 * (dL * dAlpha * dGamma * (1.0 - dAlpha))), 0) ;
    }
    else
    {
        dBeta = PI2 * sqrt ((dK1 * dK1) + (dK2 * dK2)) ;
        dDepthDecay = exp (-dBeta * abs (dX3)) ;
        dH = dGamma / dBeta ;
        i = make_comp_double (0, PI2) ;


        dTemp = (1.0 / (2.0 * dMu * (dBeta * dBeta * dBeta))) * dDepthDecay ;
        cuB1 = comp_double_mult (make_comp_double(dTemp, 0),
               make_comp_double ((double)cuC1.x, (double)cuC1.y)) ;
        cuB2 = comp_double_mult (make_comp_double(dTemp, 0),
               make_comp_double ((double)cuC2.x, (double)cuC2.y)) ;

        cuB3 = comp_double_div (comp_double_mult (make_comp_double (dTemp, 0),
               make_comp_double ((double) cuC3.x, (double) cuC3.y)),
               make_comp_double (1.0+dH, 0)) ;


        cuTemp = comp_double_mult (comp_double_mult (i, cuB3), make_comp_double (dBeta * (1.0 -
                                  (1.0 / dAlpha) + (dBeta * dX3)), 0)) ;

        cuV1 = comp_double_mult (cuTemp, make_comp_double (dK1, 0)) ;
        cuV2 = comp_double_mult (cuTemp, make_comp_double (dK2, 0)) ;
        cuV3 = comp_double_mult (make_comp_double ((dBeta * dBeta) *
                                 ((1.0 / dAlpha) + (dBeta * dX3)), 0), cuB3) ;
        cuV3 = comp_double_mult (cuV3, make_comp_double(-1.0,0)) ;
		

        dTemp = (PI2 * PI2) * (2.0  - (1.0/dAlpha) + (dBeta * dX3)) / (1+dH) ;
                
		cuV1 = comp_double_add (cuV1, comp_double_mult (cuB1, make_comp_double
                               (-2.0 * (dBeta * dBeta) + ((dK1 * dK1) * dTemp), 0))) ;
        cuV2 = comp_double_add (cuV2, comp_double_mult (cuB1, make_comp_double
                                (dK1 * dK2 * dTemp, 0))) ;
        cuV3 = comp_double_add (cuV3, comp_double_mult (comp_double_mult (cuB1, i), make_comp_double
                                ((dK1 * dBeta) * ((1.0/dAlpha) - 1.0 + (dBeta * dX3))/(1+dH), 0))) ;


        cuTemp = comp_double_add (cuV1, comp_double_mult (cuB2, make_comp_double
                                 ((dK1 * dK2 * dTemp), 0))) ;
        (*ppB1).x = (float) cuTemp.x ;
        (*ppB1).y = (float) cuTemp.y ;

        cuTemp = comp_double_add (cuV2, comp_double_mult (cuB2, make_comp_double
                                 ((-2.0 * (dBeta * dBeta) + (dK2 * dK2) * dTemp), 0))) ;
        (*ppB2).x = (float) cuTemp.x ;
        (*ppB2).y = (float) cuTemp.y ;
		

        cuTemp = comp_double_add (cuV3, comp_double_mult
                 (comp_double_mult (cuB2, i), make_comp_double
                 (dBeta * dK2 * ((1.0/dAlpha) - 1.0 + (dBeta * dX3)) / (1+dH), 0))) ;
        (*ppB3).x = (float) cuTemp.x ;
        (*ppB3).y = (float) cuTemp.y ;
 
    }
}

#endif

/* EOF */
